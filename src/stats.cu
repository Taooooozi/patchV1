#include "hip/hip_runtime.h"
#include "stats.cuh"
using namespace std;

__global__
void pixelizeOutput(
        Float* __restrict__ fr,
        Float* __restrict__ output,
        PosInt* __restrict__ pid, 
		Size* __restrict__ m, // within one pixel
		Size trainDepth, PosInt currentTimeSlot, Size nPerPixel_I, Size nPerPixel_C, Size nPixel_I, nPixel)
{
	PosInt tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < nPixel) {
		Size m_local = m[tid];
		Float value = 0;
		if (m_local > 0) {
			Size nPerPixel = tid < nPixel_I? nPerPixel_I: nPerPixel_C;
			for (PosInt i=0; i<m_local; i++) {
				PosInt id = pid[tid*nPerPixel + i];
				PosInt sInfo = fr[trainDepth*id + currentTimeSlot];
				if (sInfo > 0) {
					value += ceiling(sInfo);
				}
			}
			value = value/m_local;
		}
		__syncwarp();
		output[tid] += value
	}
}

// From nChunks of [chunkSize, ngTypeE+ngTypeI, blockSize] -> [ngTypeE+ngTypeI, nV1], where nV1 = nChunk*chunkSize*blockSize
void reshape_chunk_and_write(Float chunk[], ofstream &fRawData, Size maxChunkSize, Size remainChunkSize, PosInt iSizeSplit, Size nChunk, Size nE, Size nI, Size nV1)
{
    PosIntL offset = 0;
    size_t gSize = nV1*(nE+nI);
    Float *flatten = new Float[gSize];
    Size chunkSize = maxChunkSize;
    for (PosInt i=0; i<nChunk; i++) {
        PosIntL offset_f;
        if (i > iSizeSplit - 1) {
            chunkSize = remainChunkSize;
            offset_f = iSizeSplit*maxChunkSize + (i-iSizeSplit)*chunkSize;
        } else {
            offset_f = i*maxChunkSize;
        }
        for (PosInt j=0; j<nE; j++) {
            for (PosInt k=0; k<chunkSize*blockSize; k++) {
                flatten[j*nV1 + f_offset + k] = chunk[offset];
                offset++;
            }
        }
        for (PosInt j=0; j<nI; j++) {
            for (PosInt k=0; k<chunkSize*blockSize; k++) {
                flatten[(nE+j)*nV1 + f_offset + k] = chunk[offset];
                offset++;
            }
        }
    }
    assert(offset == (iSizeSplit*maxChunkSize + (nChunk - iSizeSplit)*remainChunkSize)*blockSize*(nE+nI));
    fRawData.write((char*) flatten, gSize*sizeof(Float));
    delete []flatten;
}

void getLGN_V1_surface(vector<PosInt> &xy, vector<vector<PosInt>> &LGN_V1_ID, PosInt* surface_xy, Size* nLGNperV1, Size max_LGNperV1, Size nLGN)
{
    Size nV1 = LGN_V1_ID.size();
    for (PosInt i=0; i<nV1; i++) {
        nLGNperV1[i] = LGN_V1_ID[i].size();
        for (PosInt j=0; j<nLGNperV1[i]; j++) {
            PosInt xid = i*max_LGNperV1 + j;
            surface_xy[xid] = xy[LGN_V1_ID[i][j]]; // x
            PosInt yid = nV1*max_LGNperV1 + xid;
            surface_xy[yid] = xy[nLGN + LGN_V1_ID[i][j]];
        }
    }
}
