#include "hip/hip_runtime.h"
#include "stats.cuh"
using namespace std;

__global__
void pixelize(
        Float* __restrict__ array,
        double* __restrict__ x,
        double* __restrict__ y,
        Float* __restrict__ frame,
        Size width, Size height) 
{
}

// From nChunks of [chunkSize, ngTypeE+ngTypeI, blockSize] -> [ngTypeE+ngTypeI, nV1], where nV1 = nChunk*chunkSize*blockSize
void reshape_chunk_and_write(Float chunk[], ofstream &fRawData, Size maxChunkSize, Size remainChunkSize, PosInt iSizeSplit, Size nChunk, Size nE, Size nI, Size nV1) {
    PosIntL offset = 0;
    size_t gSize = nV1*(nE+nI);
    Float *flatten = new Float[gSize];
    Size chunkSize = maxChunkSize;
    for (PosInt i=0; i<nChunk; i++) {
        PosIntL offset_f;
        if (i > iSizeSplit - 1) {
            chunkSize = remainChunkSize;
            offset_f = iSizeSplit*maxChunkSize + (i-iSizeSplit)*chunkSize;
        } else {
            offset_f = i*maxChunkSize;
        }
        for (PosInt j=0; j<nE; j++) {
            for (PosInt k=0; k<chunkSize*blockSize; k++) {
                flatten[j*nV1 + f_offset + k] = chunk[offset];
                offset++;
            }
        }
        for (PosInt j=0; j<nI; j++) {
            for (PosInt k=0; k<chunkSize*blockSize; k++) {
                flatten[(nE+j)*nV1 + f_offset + k] = chunk[offset];
                offset++;
            }
        }
    }
    assert(offset == (iSizeSplit*maxChunkSize + (nChunk - iSizeSplit)*remainChunkSize)*blockSize*(nE+nI));
    fRawData.write((char*) flatten, gSize*sizeof(Float));
    delete []flatten;
}

void getLGN_V1_surface(vector<PosInt> &xy, vector<vector<PosInt>> &LGN_V1_ID, PosInt* surface_xy, Size* nLGNperV1, Size max_LGNperV1, Size nLGN) {
    Size nV1 = LGN_V1_ID.size();
    for (PosInt i=0; i<nV1; i++) {
        nLGNperV1[i] = LGN_V1_ID[i].size();
        for (PosInt j=0; j<nLGNperV1[i]; j++) {
            PosInt xid = i*max_LGNperV1 + j;
            surface_xy[xid] = xy[LGN_V1_ID[i][j]]; // x
            PosInt yid = nV1*max_LGNperV1 + xid;
            surface_xy[yid] = xy[nLGN + LGN_V1_ID[i][j]];
        }
    }
}
