#include "hip/hip_runtime.h"
#include "coredynamics.cuh"
//TODO: gap junction and learning in cortex, synaptic depression
//TODO: synaptic failure, noise
extern surface<void, cudaSurfaceType2DLayered> LGNspikeSurface;

__launch_bounds__(1024,2)
__global__ 
void rand_spInit(Float* __restrict__ tBack,
                 Float* __restrict__ spikeTrain,
                 Float* __restrict__ v,
                 Float* __restrict__ w,
                 Size* __restrict__ nLGNperV1,
                 Float* __restrict__ sp0,
                 Size* __restrict__ typeAcc,
                 Float* __restrict__ vR,
                 Float* __restrict__ gL,
                 Float* __restrict__ tRef_type,
                 Float* __restrict__ tau_w,
                 Float* __restrict__ a,
                 Float* __restrict__ b,
                 hiprandStateMRG32k3a* __restrict__ rGenCond,
                 PosIntL seed, Size networkSize, Size nType, Size SCsplit, Size trainDepth, Float dt, bool iModel) 
{
    PosIntL id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < networkSize) {
        hiprandStateMRG32k3a localState = rGenCond[id];
        Size iLGN = nLGNperV1[id];
        Size type;
        for (PosInt i=0; i<nType; i++) {
            if (id < typeAcc[i]) {
                type = typeAcc[i];
                break;
            }
        }
        hiprand_init(seed + id, 0, 0, &localState);
        Float rand = uniform(&localState);
        Float chance;
        Float ref = 0.0;
        if (iLGN > SCsplit) {
            chance = sp0[type*2 + 0]; 
        } else {
            chance = sp0[type*2 + 1]; 
        }
        Float tRef = tRef_type[type];
        if (chance > 0) {
            if (rand < chance) {
                Float tsp = uniform(&localState);
                spikeTrain[id + 0*networkSize] = 1.0 + tsp;
				Float tb = tRef - (1-tsp)*dt;
				if (tb > 0) {
					printf("%u: tb = %.3f\n", id, tb);
                	tBack[id] = tb;
				}
				Float v0 = vR[type];
                v[id] = v0;
				if (iModel == 1) {
					Float A = a[type]*(v0-vL) * tau_w[type];
					Float w0 = w[id] + b[type];
					w[id] = (w0 - A) * exponential(-dt*(1-tsp)/tau_w[type]) + A;
				}

                ref = (1-tsp)*dt + tRef - dt;
            }
        }
        for (PosInt i=trainDepth-1; i>0; i--) {
            if (ref < dt) {
                if (ref < 0) ref = 0;
                if (uniform(&localState) < chance*(dt-ref)/dt) {
                    Float tsp = uniform(&localState)*(dt-ref)/dt;
                    spikeTrain[i*networkSize + id] = 1.0  + tsp;
                    ref = tRef + (1-tsp)*dt;
                } 
            }
            ref -= dt;
        }
        rGenCond[id] = localState;
    }
}

__launch_bounds__(1024,2)
__global__
void logRand_init(Float* __restrict__ logRand,
                  Float* __restrict__ lTR,
                  int* __restrict__ LGN_idx,
                  int* __restrict__ LGN_idy,
                  hiprandStateMRG32k3a *state,
                  PosIntL seed, Size n, Size nFF)
{
    Size id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		hiprandStateMRG32k3a localState = state[id];
		hiprand_init(seed + id, 0, 0, &localState);
		Float rand = uniform(&localState);
		logRand[id] = -log(uniform(&localState));
		state[id] = localState;
		lTR[id] = 0;
        int x = LGN_idx[id];
        int y = LGN_idy[id];
        Float value = 0; // this is needed, otherwise surf2DLayeredwrite will raise runtime error
        surf2DLayeredwrite(value, LGNspikeSurface, 4*x, y, 0);
        #pragma unroll sum_nLearnTypeFF
        for (int i=0; i<nFF; i++) {
            surf2DLayeredwrite(value, LGNspikeSurface, 4*x, y, 1+3*i+0);
            surf2DLayeredwrite(value, LGNspikeSurface, 4*x, y, 1+3*i+1);
            surf2DLayeredwrite(value, LGNspikeSurface, 4*x, y, 1+3*i+2);
        }
	}
}

__device__
__forceinline__
Float step(IF* model, Float dt, PosInt id, Float gE, Float gI) {
    model->spikeCount = 0;
    Float sInfo = 0.0;
    // not in refractory period
    if (model->tBack < dt) {
        // return from refractory period
        if (model->tBack > 0.0f) {
            model->recompute_v0(dt);
            #ifdef DEBUG
                if (id == 0 || id == 768) {
                    printf("backed\n");
                }
            #endif
        }
        model->rk2(dt);
        while (model->v > model->vThres && model->tBack < dt) { // forbids firing exactly at the end of the timestep, 
            // crossed threshold
            model->compute_spike_time(dt); 
            sInfo += model->tsp;
            model->spikeCount++;
            model->tBack = model->tsp + model->tRef;
            #ifdef DEBUG
                if (id == 0 || id == 768) {
                    printf("#%u spiked at %f, to come back at %f\n", id, model->tsp, model->tBack);
                }
            #endif
            if (model->tBack < dt) {
                // refractory period ended during dt
                model->recompute(dt);
            }
        }
    }
    if (model->tBack >= dt) {
        // during refractory period
        model->reset1();
    }
    model->tBack -= dt;
    #ifdef DEBUG
        if (model->v < vI) {
    		printf("#%i implicit rk2 is A-Stable! something is off gE1 = %f, gI1 = %f, v = %f, v0 = %f, a0 = %f, b0 = %f, a1 = %f, b1 = %f\n", id, gE, gI, model->v, model->v0, model->a0, model->b0, model->a1, model->b1);
        }   
    #endif
    if (model->spikeCount > 0) sInfo /= model->spikeCount*dt; //decimal part: tsp (normalize by dt)
    sInfo += model->spikeCount; // integer part: nsp
    #ifdef DEBUG
        if ((sInfo > 0 && sInfo < 1) || model->spikeCount >= 2) {
            printf("sInfo = %.3f, gE = %.3e, gI = %.3e, spikeCount = %u\n", sInfo, gE, gI, model->spikeCount);
            assert(sInfo == 0 || (sInfo >= 1 && sInfo < 2 && model->spikeCount < 2));
        }
    #endif
    __syncwarp();
    return sInfo;
}

__device__ 
__forceinline__
void IF::reset1() {
    v = vR;
}

__device__
__forceinline__
void IF::compute_spike_time(Float dt, Float t0) {
    tsp = comp_spike_time(v, v0, vThres, dt, t0);
}

__device__ 
__forceinline__
void IF::set_p0(Float gE, Float gI) {
    a0 = get_a(gE, gI, gL);
    b0 = get_b(gE, gI, gL); 
}

__device__ 
__forceinline__
void IF::set_p1(Float gE, Float gI) {
    a1 = get_a(gE, gI, gL);
    b1 = get_b(gE, gI, gL); 
}

__device__
__forceinline__
void LIF::rk2(Float dt) {
    v = impl_rk2(dt, a0, b0, a1, b1, v0);
}

__device__
__forceinline__
void LIF::recompute(Float dt, Float t0) {
    Float rB = dt/(tBack-t0) - 1; 
    Float denorm = 2 + a1*dt;
    Float A = (2 - a0*dt)/denorm;
    Float B = (b0 + b1)*dt/denorm;
    v0 = recomp_v0(A, B, rB);
    v = A*v0 + B;
}

__device__ 
__forceinline__
void LIF::recompute_v(Float dt, Float t0) {
    Float rB = dt/(tBack-t0) - 1; 
    Float denorm = 2 + a1*dt;
    Float A = (2 - a0*dt)/denorm;
    Float B = (b0 + b1)*dt/denorm;
    v = recomp_v(A, B, rB);
}

__device__ 
__forceinline__
void LIF::recompute_v0(Float dt, Float t0) {
    Float rB = dt/(tBack-t0) - 1; 
    Float denorm = 2 + a1*dt;
    Float A = (2 - a0*dt)/denorm;
    Float B = (b0 + b1)*dt/denorm;
    v0 = recomp_v0(A, B, rB);
}

__device__ 
__forceinline__
void AdEx::rk2(Float dt) {
	Float fk1 = -a0*v0 + b0 + deltaT*gL*exponential((v0-vT)/deltaT) - w0;
	Float gk1 = a*(v0-vL) - w0/tau_w;
	Float v1 = v0 + fk1*dt;
	Float w1 = w0 + gk1*dt;
	Float fk2 = -a1*v1 + b1 + deltaT*gL*exponential((v1-vT)/deltaT) - w1;
	Float gk2 = a*(v1-vL) - w1/tau_w;
	v = v0 + (fk1 + fk2)/2 * dt;
	w = w0 + (gk1 + gk2)/2 * dt;
}

//TODO: distant connection learning
void recal_G_vec(
        std::vector<std::vector<std::vector<Float>>> &spikeTrain, std::vector<std::vector<Size>> &trainDepth, std::vector<std::vector<PosInt>> &currentTimeSlot,
        std::vector<Size> &nVec,  std::vector<std::vector<PosInt>> &vecID, std::vector<std::vector<Float>> &conVec, std::vector<std::vector<Float>> &delayVec,
        Float gE[], Float gI[], Float hE[], Float hI[], Float pE[], Float pI[], Size typeAcc[],
        std::default_random_engine *rGenCond, Float noisyCondE[], Float noisyCondI[], Float synFailE[], Float synFailI[],
        Float dt, ConductanceShape condE, ConductanceShape condI, Size ngTypeE, Size ngTypeI, PosInt block_offset, Size nType, Size nE, Size nV1, Float speedOfThought, Size chunkSize, bool noisyH) 
{
    Float ipE[max_ngTypeE];
    Float ipI[max_ngTypeI];
    Float local_gE[max_ngTypeE];
    Float local_hE[max_ngTypeE];
    Float local_gI[max_ngTypeI];
    Float local_hI[max_ngTypeI];
    PosInt i0 = block_offset*blockSize;
    std::normal_distribution<Float> normal_dist(0.0, 1.0);
    std::uniform_real_distribution<Float> uniform_dist(0.0, 1.0);
    for (PosInt i=0; i<chunkSize*blockSize; i++) {
        // initialize
        PosInt itype;
        #pragma unroll max_nType
        for (PosInt j=0; j<nType; j++) {
            if (i%blockSize < typeAcc[j]) {
                itype = j;
                break;
            }
        }
        if (nVec[i] == 0) continue;
        #pragma unroll max_ngTypeE
        for (PosInt ig=0; ig<ngTypeE; ig++) {
            local_gE[ig] = 0.0f;
            local_hE[ig] = 0.0f;
            ipE[ig] = pE[itype*ngTypeE + ig];
        }
        #pragma unroll max_ngTypeI
        for (PosInt ig=0; ig<ngTypeI; ig++) {
            local_gI[ig] = 0.0f;
            local_hI[ig] = 0.0f;
            ipI[ig] = pI[itype*ngTypeI + ig];
        }
        #pragma unroll 4
        for (PosInt j = 0; j < nVec[i0+i]; j++) {
            PosInt ipre = vecID[i0+i][j];
            PosInt tid = ipre%blockSize; 
            Float strength = conVec[i0+i][j];

            Float time2post = delayVec[i0+i][j]/speedOfThought;
            Float *local_g;
            Float *local_h;
            Float *ip;
            Float *noisyCond;
            Float synFail;
            Size ngType;
            ConductanceShape *cond;
            // TODO direct output to g and h (local memory vs register)
            if (tid < nE) {
                local_g = local_gE;
                local_h = local_hE;
                ngType = ngTypeE;
                cond = &condE;
                ip = ipE;
                noisyCond = noisyCondE;
                synFail = synFailE[itype];
            } else {
                local_g = local_gI;
                local_h = local_hI;
                ngType = ngTypeI;
                cond = &condI;
                ip = ipI;
                noisyCond = noisyCondI;
                synFail = synFailI[itype];
            }
            PosInt it2post = static_cast<PosInt>(ceiling(time2post/dt));
            time2post = it2post*dt - time2post;
            assert(time2post>=0);
            assert(time2post<dt);
            PosInt k0 = currentTimeSlot[i0+i][j] - it2post + trainDepth[i0+i][j];
            currentTimeSlot[i0+i][j] = (currentTimeSlot[i0+i][j]+1)%trainDepth[i0+i][j];
            #pragma unroll max_ngType
            for (PosInt ig=0; ig<ngType; ig++) {
                Float g0 = 0;
				Float h0 = 0;
                #pragma unroll 2
                for (PosInt k = 0; k < 2; k++) {
                    Float sInfo = spikeTrain[i0+i][j][k0+k];
                    if (sInfo > 0) {
                        Float nsp = flooring(sInfo);
                        Float tsp = (sInfo - nsp + k)*dt - time2post;
                        if (tsp < dt && tsp >= 0) {
							if (uniform_dist(rGenCond[i0+i]) > synFail){
                            	cond->compute_single_input_conductance(g0, h0, strength*nsp*ip[ig], dt-tsp, ig);
							}
                        }
                    }
                }
                if (noisyCond[ig] > 0) {
                    Float noise = noisyCond[ig]*strength*ip[ig]*normal_dist(rGenCond[i0+i]);
					if (noisyH) {
						h0 += noise;
                    	if (h0<0) h0 = 0;
					} else {
						g0 += noise;
                    	if (g0<0) g0 = 0;
					}
                }
                local_g[ig] += g0;
                local_h[ig] += h0;
            }
        }
        // output
        #pragma unroll max_ngTypeE
        for (PosInt ig=0; ig<ngTypeE; ig++) {
            PosInt gid = ig*chunkSize*blockSize + i;
            gE[gid] = local_gE[ig];
            hE[gid] = local_hE[ig];
        }
        #pragma unroll max_ngTypeI
        for (PosInt ig=0; ig<ngTypeI; ig++) {
            PosInt gid = ig*chunkSize*blockSize + i;
            gI[gid] = local_gI[ig];
            hI[gid] = local_hI[ig];
        }
    }
}

//template<int ntimesFF, int ntimesE, int ntimesI>
__launch_bounds__(1024,1)
__global__ 
void compute_V_collect_spike_learnFF(
        Float* __restrict__ v,
        Float* __restrict__ w,
        Float* __restrict__ gFF, // not in chunks
        Float* __restrict__ hFF,
        Float** __restrict__ gE, // in chunks
        Float** __restrict__ gI,
        Float** __restrict__ hE,
        Float** __restrict__ hI,
        Size* __restrict__ nLGN,
        Float* __restrict__ sLGN,
        int* __restrict__ LGN_idx,
        int* __restrict__ LGN_idy,
        Float* __restrict__ tBack,
        Float* __restrict__ spikeTrain, //         [                depth, nblock, blockSize  ]
        Float* __restrict__ vLTD_FF_E, //    post, [nLearnTypeFF_E,        nblock, nE         ]
        Float* __restrict__ vTrip_FF_E, //   post, [nLearnTypeFF_E,        nblock, nE         ]
        Float* __restrict__ vLTD_FF_I, //    post, [nLearnTypeFF_I,        nblock, nI         ]
        Float* __restrict__ vTrip_FF_I, //   post, [nLearnTypeFF_I,        nblock, nI         ]
        Float* __restrict__ vAvgE, //        post, [                       nblock, nE,       2]
        Float* __restrict__ vAvgI, //        post, [                       nblock, nI         ]
        Float* __restrict__ vLTP_E, //        pre, [nLearnTypeE,    depth, nblock, nE,       2]
        Float* __restrict__ vLTD_E, //       post, [nLearnTypeE,           nblock, nE,       2]
        Float* __restrict__ vTripE, //       post, [nLearnTypeE,           nblock, nE,       2]
        Float* __restrict__ vSTDP_QE,  //  E post, [nLearnTypeQ,           nblock, nE        2]
        Float* __restrict__ vSTDP_QI,  //   I pre, [nLearnTypeQ,    depth, nblock, nI,       2]
        Float* __restrict__ pFF,
        Float* __restrict__ vR,
        Float* __restrict__ vThres,
        Float* __restrict__ gL,
        Float* __restrict__ tRef,
        Float* __restrict__ vT,
        Float* __restrict__ deltaT,
        Float* __restrict__ tau_w,
        Float* __restrict__ a,
        Float* __restrict__ b,
        Size* __restrict__ typeAcc,
        hiprandStateMRG32k3a* __restrict__ rGenCond,
        Float* __restrict__ noisyCondFF,
        Float* __restrict__ synFailFF,
        PosInt currentTimeSlot, Size trainDepth, Size max_nLGN, Size ngTypeFF, Size ngTypeE, Size ngTypeI, ConductanceShape condFF, ConductanceShape condE, ConductanceShape condI, Float dt, Size maxChunkSize, Size remainChunkSize, PosInt iSizeSplit, Size nChunk, Size nE, Size nI, Size nV1, int learning, int varSlot, Size nType,
        LearnVarShapeFF_E_pre  learnE_pre,  LearnVarShapeFF_I_pre  learnI_pre, 
        LearnVarShapeFF_E_post learnE_post, LearnVarShapeFF_I_post learnI_post, 
        LearnVarShapeE learnE, LearnVarShapeQ learnQ, int iModel, bool noisyH)
{
	//assert(blockDim.x == blockSize);
    PosInt tid = blockIdx.x * blockDim.x + threadIdx.x;
    PosInt iChunk;
    Size chunkSize;
    PosInt cid;
    if (blockIdx.x >= iSizeSplit*maxChunkSize) {
        iChunk = iSizeSplit + (blockIdx.x-iSizeSplit*maxChunkSize)/remainChunkSize;
        chunkSize = remainChunkSize*blockDim.x;
        cid = tid - (iSizeSplit*maxChunkSize + (iChunk-iSizeSplit)*remainChunkSize)*blockDim.x;
    } else {
        iChunk = blockIdx.x/maxChunkSize;
        chunkSize = maxChunkSize*blockDim.x;
        cid = tid - iChunk*maxChunkSize*blockDim.x;
    }

    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    // TODO: load individual gl, tref
    PosInt itype;
    #pragma unroll max_nType
    for (PosInt j=0; j<nType; j++) {
        if (threadIdx.x < typeAcc[j]) {
            itype = j;
            break;
        }
    }
    IF* model;
    if (iModel == 0) {
        model = new LIF(v[tid], tBack[tid], vR[itype], vThres[itype], gL[itype], tRef[itype]);
    } else {
        model = new AdEx(w[tid], tau_w[itype], a[itype], b[itype], v[tid], tBack[tid], vR[itype], vThres[itype], gL[itype], tRef[itype], vT[itype], deltaT[itype]);
	}
    /* set a0 b0 and a1 b1 */
    // cond FF
    //#pragma unroll (MAX_NGTYPE_FF)
    Size m = nLGN[tid];
    //Size nsp_FFt = 0;
    hiprandStateMRG32k3a localState = rGenCond[tid];
	
	Float ge[max_ngTypeE];
	Float he[max_ngTypeE];
	Float gi[max_ngTypeI];
	Float hi[max_ngTypeI];

	Float gE_t1 = 0.0;
	Float gI_t1 = 0.0;
	if (model->spikeCount == 0) {
		//	cond E, decay only
		//		g1, initialize g0	
    	#pragma unroll (max_ngTypeE) 
    	for (PosInt ig=0; ig<ngTypeE; ig++) {
    	    PosInt gid = chunkSize*ig + cid;
    	    Float g = gE[iChunk][gid];
    	    Float h = hE[iChunk][gid];
			ge[ig] = g;
			he[ig] = h;

    	    condE.decay_conductance(g, h, dt, ig); 
    	    gE[iChunk][gid] = g;
    	    hE[iChunk][gid] = h;
			gE_t1 += g;
    	}
    	//	cond I, decay only
		//		g1, initialize g0	
    	#pragma unroll (max_ngTypeI)
    	for (PosInt ig=0; ig<ngTypeI; ig++) {
    	    PosInt gid = chunkSize*ig + cid;
    	    Float g = gI[iChunk][gid];
    	    Float h = hI[iChunk][gid];
			gi[ig] = g;
			hi[ig] = h;

    	    condI.decay_conductance(g, h, dt, ig); 
    		gI[iChunk][gid] = g;
    		hI[iChunk][gid] = h;
			gI_t1 += g;
    	}
	}


	Float g0[max_ngTypeFF];
	Float h0[max_ngTypeFF];
	Float g1[max_ngTypeFF];
	Float h1[max_ngTypeFF];
	Float noisyCond[max_ngTypeFF];
    #pragma unroll (max_ngTypeFF) //(ntimesFF)
    for (PosInt ig=0; ig<ngTypeFF; ig++) {
        PosInt gid = nV1*ig + tid; // not in chunks
        g1[ig] = gFF[gid]; // g1, all input and decay
        h1[ig] = hFF[gid];
        g0[ig] = g1[ig]; // only before tBack
        h0[ig] = h1[ig];

		//	decay
    	condFF.decay_conductance(g1[ig], h1[ig], dt, ig); //  decayed from new_t0 to tBack
	}

	Float* rand = new Float[m];
	Float synFail = synFailFF[itype];

	bool backingUpFromRef = model->tBack < dt && model->tBack > 0;
	Float new_t0 = 0;
	Float sInfo = 0;
	do {
		Float dtBack;
		if (backingUpFromRef) {
			dtBack = model->tBack - new_t0;
		}
		//	condFF
		//		decay part
    	#pragma unroll (max_ngTypeFF) //(ntimesFF)
    	for (PosInt ig=0; ig<ngTypeFF; ig++) {
			if (backingUpFromRef) { // collect conductance at tBack
    		    condFF.decay_conductance(g0[ig], h0[ig], dtBack, ig); //  decayed from new_t0 to tBack
			}
		}
		//		input part
    	#pragma unroll (4)
    	for (PosInt i = 0; i<m; i++) {
    	    PosInt lid = tid*max_nLGN + i;
    	    Float f = sLGN[lid];
    	    int x = LGN_idx[lid];
    	    int y = LGN_idy[lid];
    	    Float sInfo;
    	    surf2DLayeredread(&sInfo, LGNspikeSurface, 4*x, y, 0);
    	    Float nsp = flooring(sInfo); // integer part: #spikes
    	    Float tsp = (sInfo - nsp)*dt; // decimal part: normalized mean tsp
			Float synapse[max_ngTypeFF];
			if (model->spikeCount == 0) {
    			#pragma unroll (max_ngTypeFF) //(ntimesFF)
    			for (PosInt ig=0; ig<ngTypeFF; ig++) {
					synapse[ig] = 0.0;
				}
			}
			
			if (nsp > 0) {
				if (model->spikeCount == 0) {
					rand[i] = uniform(&localState);
				}
				if (rand[i] > synFail && tsp >= new_t0) {
					Float ddt;
					if (backingUpFromRef) {
						ddt = model->tBack - tsp;
					}
    				#pragma unroll (max_ngTypeFF) //(ntimesFF)
    				for (PosInt ig=0; ig<ngTypeFF; ig++) {
    		    		Float str = f * pFF[itype*ngTypeFF + ig];
						if (backingUpFromRef) {
							if (ddt > 0) { // before tBack
    		    				condFF.compute_single_input_conductance(g0[ig], h0[ig], str*nsp, ddt, ig);
    		    			}
						}
						if (model->spikeCount == 0) { // all inputs
							Float gS = 0;
							Float hS = 0;
    		    			condFF.compute_single_input_conductance(gS, hS, str*nsp, dt*(1-tsp), ig);
							g1[ig] += gS;
							h1[ig] += hS;
							if (noisyH) {
								synapse[ig] = hS;
							} else {
								synapse[ig] = gS;
							}
						}
					}
				}
			}
			// noisyCond
			if (model->spikeCount == 0) {
    			#pragma unroll (max_ngTypeFF) //(ntimesFF)
    			for (PosInt ig=0; ig<ngTypeFF; ig++) {
    	    		if (noisyCond[ig] > 0) {
    	    		    Float rand0 = normal(&localState);
    		    		Float str = f * pFF[itype*ngTypeFF + ig];
    	    		    Float noise = noisyCond[ig]*str*pFF[itype*ngTypeFF + ig]*rand0;
						synapse[ig] += noise;
    	    			if (synapse[ig]<0) synapse[ig] = 0;
						if (noisyH) {
							h1[ig] += synapse[ig];
						} else {
							g1[ig] += synapse[ig];
						}
    	    		}
				}
			}
    	}
		if (model->spikeCount == 0) {
    		rGenCond[tid] = localState;
		}
		//	collect
		Float gE_t0 = 0.0;
    	#pragma unroll (max_ngTypeFF) //(ntimesFF)
    	for (PosInt ig=0; ig<ngTypeFF; ig++) {
        	PosInt gid = nV1*ig + tid;
    	    gE_t0 += g0[ig];
			if (model->spikeCount == 0) {
    	    	gE_t1 += g1[ig];
    			gFF[gid] = g1[ig];
    			hFF[gid] = h1[ig];
			}
    	}
    	
		//	condE, g0
    	#pragma unroll (max_ngTypeE)
    	for (PosInt ig=0; ig<ngTypeE; ig++) {
			if (backingUpFromRef) {
    	    	condE.decay_conductance(ge[ig], he[ig], dtBack, ig); 
			}
    	    gE_t0 += ge[ig];
		}

		//	condI, g0
    	Float gI_t0 = 0.0;
    	#pragma unroll (max_ngTypeI)
    	for (PosInt ig=0; ig<ngTypeI; ig++) {
			if (backingUpFromRef) {
    	    	condI.decay_conductance(gi[ig], hi[ig], dtBack, ig); 
			}
    	    gI_t0 += gi[ig];
		}

    	// stepping
		model->tsp = 0;
		if (model->tBack < dt) {
    		model->set_p0(gE_t0, gI_t0);
			if (model->spikeCount == 0) {
    			model->set_p1(gE_t1, gI_t1);
			}

			Float new_dt = dt - model->tBack;
			if (backingUpFromRef) { //	stepping other variable before tBack
				model->rk2_vFixedBefore(dtBack);
			} 
			model->rk2(new_dt);

			// check spiking
    	    if (model->v > model->vThres) { // forbids firing exactly at the end of the timestep, 
    	        // crossed threshold
				new_t0 = model->tBack;
    	        model->compute_spike_time(new_dt, new_t0); 
    	        sInfo += model->tsp;
    	        model->spikeCount++;
    	        model->tBack = model->tsp + model->tRef;
				backingUpFromRef = model->tBack < dt && model->tBack > 0;
				if (backingUpFromRef) {
					model->reset0();
				}
    	    } else {
				if (model->tBack > 0) model->tBack -= dt;
				backingUpFromRef = false;
			}
		} 
		if (model->tBack >= dt) { // tRef till end
			model->reset1();
			model->rk2_vFixedAfter(dt-model->tsp);
			model->tBack -= dt;
		}
    	/* evolve g to t+dt with ff input only */
	} while (backingUpFromRef);
	delete []rand;
	
    if (model->spikeCount > 0) {
		sInfo /= model->spikeCount*dt; //decimal part: tsp (normalize by dt)
		model->tBack -= dt;
	}
	if (model->tBack < 0) model->tBack = 0;
    sInfo += model->spikeCount; // integer part: nsp
    spikeTrain[nV1*currentTimeSlot + tid] = sInfo;

	/* debug
    	if (isnan(sInfo) || tid == 16737) {
    	    Size nsp = flooring(sInfo);
    	    printf("%u(%u): spiked at sInfo: %f, %u + %f, gFF[0] = %f, gFF[1] = %f, gE[0] = %f, gE[1] = %f, gE_t = %f, gI_t = %f\n", tid, cid, sInfo, nsp, sInfo - nsp, gFF[tid], gFF[tid+nV1], gE[iChunk][cid], gE[iChunk][cid + chunkSize], gE_t1, gI_t1);
    	    assert(!isnan(sInfo));
    	}
    	if (sInfo > 0 && (threadIdx.x == 0 || threadIdx.x == 768)) {
    	if (sInfo > 0 && (gI_t0 > 0 || threadIdx.x >= nE)) {
    	    Size nsp = flooring(sInfo);
    	    printf("%u(%u): spiked at sInfo: %u + %f, gF = %e(%u), gE = %e, gI = %e\n", tid, cid, nsp, sInfo - nsp, gFF[tid], m, gE[iChunk][cid], gI_t0);
    	}
	*/
	v[tid] = model->v;
	if (iModel == 1) {
		Float** var = new Float*[1];
		var[0] = w+tid;
		model->update(var);
		delete []var;
	}
    tBack[tid] = model->tBack;
    delete []model;

    if (learning) {
        Float nsp = flooring(sInfo);
        Float tsp = sInfo>0? sInfo - nsp: 1;
        // will compute ff learning, first row at start of time step, second row at tsp
        Float lFF[2*2*max_nLearnTypeFF]; // row 0: start, row 1: sp
        Float lAvg[2];
        // only temporary store
        Float lE[3*max_nLearnTypeE];
        Float lQ[max_nLearnTypeQ];
        // read ff (post) lVar
        PosInt eid = nE*blockIdx.x+threadIdx.x;
        if (learning < 4) { // read regardless of cortical spike 
            if (threadIdx.x < nE) {
                #pragma unroll max_nLearnTypeFF_E
                for (PosInt i=0; i<learnE_post.n; i++) {
                    lFF[2*i+0] =  vLTD_FF_E[nE*gridDim.x*i + eid];
                    lFF[2*i+1] = vTrip_FF_E[nE*gridDim.x*i + eid];
                }
                lAvg[0] = vAvgE[eid*2];
            } else {
                if (learnI_post.n) {
                    PosInt iid = nI*blockIdx.x+threadIdx.x-nE;
                    #pragma unroll max_nLearnTypeFF_I
                    for (PosInt i=0; i<learnI_post.n; i++) {
                        lFF[2*i+0] =  vLTD_FF_I[nI*gridDim.x*i + iid];
                        lFF[2*i+1] = vTrip_FF_I[nI*gridDim.x*i + iid];
                    }
                    lAvg[0] = vAvgI[iid];
                }
            }
        }
        if (nsp > 0) {
            if (learning !=3) { // E and Q are active, read cortical lVar and AvgE if previouly not read
                if (threadIdx.x < nE) {
                    // E
                    #pragma unroll max_nLearnTypeE
                    for (PosInt i=0; i<learnE.n; i++) {
                        lE[3*i+0] = vLTP_E[(nE*gridDim.x*trainDepth*i + nE*gridDim.x*currentTimeSlot + eid)*2];
                        lE[3*i+1] = vLTD_E[(nE*gridDim.x*i + eid)*2];
                        lE[3*i+2] = vTripE[(nE*gridDim.x*i + eid)*2];
                    }
                    // Q_E
                    #pragma unroll max_nLearnTypeQ
                    for (PosInt i=0; i<learnQ.n; i++) {
                        lQ[i] = vSTDP_QE[(nE*gridDim.x*i + eid)*2];
                    }
                    if (learning == 4) { // otherwise already read
                        lAvg[0] = vAvgE[eid*2];
                    }
                } else {
                    // Q_I
                    PosInt iid = nI*(gridDim.x*currentTimeSlot + blockIdx.x) + threadIdx.x-nE;
                    #pragma unroll max_nLearnTypeQ
                    for (PosInt i=0; i<learnQ.n; i++) {
                        lQ[i] = vSTDP_QI[(nI*gridDim.x*trainDepth*i + iid)*2];
                    }
                }
            }
            if (learning < 4) { // compute ff post vars' decay till tsp
                if (threadIdx.x < nE) {
                    #pragma unroll max_nLearnTypeFF_E
                    for (PosInt i=0; i<learnE_post.n; i++) {
                        lFF[2*max_nLearnTypeFF + 2*i+0] = lFF[2*i+0];
                        lFF[2*max_nLearnTypeFF + 2*i+1] = lFF[2*i+1];
                    }
                    #pragma unroll max_nLearnTypeFF_E
                    for (PosInt i=0; i<learnE_post.n; i++) {
                        decay(lFF[2*max_nLearnTypeFF + 2*i+0], learnE_post.tau[2*i+0], tsp);
                        decay(lFF[2*max_nLearnTypeFF + 2*i+1], learnE_post.tau[2*i+1], tsp);
                    }
                } else {
                    if (learnI_post.n) {
                        #pragma unroll max_nLearnTypeFF_I
                        for (PosInt i=0; i<learnI_post.n; i++) {
                            lFF[2*max_nLearnTypeFF + 2*i+0] = lFF[2*i+0];
                            lFF[2*max_nLearnTypeFF + 2*i+1] = lFF[2*i+1];
                        }
                        #pragma unroll max_nLearnTypeFF_I
                        for (PosInt i=0; i<learnI_post.n; i++) {
                            decay(lFF[2*max_nLearnTypeFF + 2*i+0], learnI_post.tau[2*i+0], tsp);
                            decay(lFF[2*max_nLearnTypeFF + 2*i+1], learnI_post.tau[2*i+1], tsp);
                        }
                        lAvg[1] = lAvg[0];
                        decay(lAvg[1], learnI_post.tau[2*learnI_post.n], tsp);
                    }
                }
            }
            if (threadIdx.x < nE) { // compute AvgE
                lAvg[1] = lAvg[0];
                decay(lAvg[1], learnE_post.tau[2*learnE_post.n], tsp);
            }
            if (learning !=3) { // compute and store lVars of E, Q and AvgE
                // compute
                if (threadIdx.x < nE) {
                    #pragma unroll max_nLearnTypeE
                    for (PosInt i=0; i<learnE.n; i++) {
                        decay(lE[3*i+0], learnE.tau[3*i+0], tsp);
                        decay(lE[3*i+1], learnE.tau[3*i+1], tsp);
                        decay(lE[3*i+2], learnE.tau[3*i+2], tsp);
                    }
                    #pragma unroll max_nLearnTypeQ
                    for (PosInt i=0; i<learnQ.n; i++) {
                        decay(lQ[i], learnQ.tau[2*i+0], tsp); // Q_E
                    }
                } else {
                    #pragma unroll max_nLearnTypeQ
                    for (PosInt i=0; i<learnQ.n; i++) {
                        decay(lQ[i], learnQ.tau[2*i+1], tsp); // Q_I
                    }
                }
                // store
                if (threadIdx.x < nE) {
                    #pragma unroll max_nLearnTypeE
                    for (PosInt i=0; i<learnE.n; i++) {
                         vLTP_E[(nE*gridDim.x*trainDepth*i + nE*gridDim.x*currentTimeSlot + eid)*2 + 1] = lE[3*i+0];
                         vLTD_E[(nE*gridDim.x*i + eid)*2 + 1] = lE[3*i+1];
                         vTripE[(nE*gridDim.x*i + eid)*2 + 1] = lE[3*i+2];
                    }
                    vAvgE[2*eid+1] = lAvg[1];
                    #pragma unroll max_nLearnTypeQ
                    for (PosInt i=0; i<learnQ.n; i++) { // store to the second slot of the array
                        vSTDP_QE[(nE*gridDim.x*i + eid)*2 + 1] =  lQ[i];
                    }
                } else {
                    PosInt iid = nI*(gridDim.x*currentTimeSlot + blockIdx.x) + threadIdx.x-nE;
                    #pragma unroll max_nLearnTypeQ
                    for (PosInt i=0; i<learnQ.n; i++) { // store to the second slot of the array
                        vSTDP_QI[(nI*gridDim.x*trainDepth*i + iid)*2 + 1] =  lQ[i];
                    }
                }
            }
        }
        // learn LGN connection and update LGN lVars
        if (learning < 4 && (threadIdx.x < nE || learnI_pre.n)) { 
            // learn
            for (PosInt i = 0; i<m; i++) {
                PosInt lid = tid*max_nLGN + i;
                Float f = sLGN[lid];
                int x = LGN_idx[lid];
                int y = LGN_idy[lid];
                Float sInfo_FF;
                surf2DLayeredread(&sInfo_FF, LGNspikeSurface, 4*x, y, 0);
                Float nsp_FF = flooring(sInfo_FF);
                Float tsp_FF = sInfo_FF > 0? sInfo_FF - nsp_FF: 1;
                if (nsp_FF > 0) { // LTD, regarless of post spike
                    PosInt cPick;
                    Float delta_t;
                    if (tsp_FF < tsp) {
                        cPick = 0; // from start
                        delta_t = tsp_FF;
                    } else {
                        cPick = 1; // from tsp
                        delta_t = tsp_FF-tsp;
                    }
                    delta_t *= dt;
                    if (threadIdx.x < nE) {
                        #pragma unroll max_nLearnTypeFF_E
                        for (PosInt j=0; j<learnE_pre.n; j++) {
                            Float A_LTD = learnE_post.A_ratio[j] * learnE_pre.tauLTP[j] * lAvg[cPick] * lAvg[cPick]/learnE_post.targetFR;
                            //Float A_LTD = learnFF_E.A_LTP[j]; TODO: alternative homeostatic design
                            /*debug
							if (tid == 0 && i == 0) {
                                printf("%u-%u, A_LTD: %e = %e*%e*%e^2/%e\n", tid, i, A_LTD, learnE_post.A_ratio[j], learnE_pre.tauLTP[j], lAvg[cPick], learnE_post.targetFR);
								printf("%u-%u, old_f: %e\n", tid, i, f);
                            }*/
                            f -= if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnE_post.tau[2*j+0], delta_t) * A_LTD;
                            /*debug
							if (tid == 0 && i == 0) {
								printf("%u-%u, new_f: %e\n", tid, i, f);
								Float df = if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnE_post.tau[2*j+0], delta_t) * A_LTD;
								printf("%u-%u, df %e = %e*%e\n", tid, i, df, if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnE_post.tau[2*j+0], delta_t), A_LTD);
							}*/
                        }
                    } else {
                        #pragma unroll max_nLearnTypeFF_I
                        for (PosInt j=0; j<learnI_pre.n; j++) {
                            Float A_LTD = learnI_post.A_ratio[j] * learnI_pre.tauLTP[j] * lAvg[cPick] * lAvg[cPick]/learnE_post.targetFR;
                            f -= if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnI_post.tau[2*j+0], delta_t) * A_LTD;
                        }
                    }
                } 
                if (nsp > 0) { // LTP, regardless of pre spike
                    PosInt fPick;
                    Float delta_t;
                    if (tsp_FF < tsp) {
                        fPick = 2;
                        delta_t = tsp-tsp_FF;
                    } else {
                        fPick = varSlot;
                        delta_t = tsp;
                    }
                    delta_t *= dt;
                    if (threadIdx.x < nE) {
                        #pragma unroll max_nLearnTypeFF_E
                        for (PosInt j=0; j<learnE_pre.n; j++) {
                            Float lFF_pre;
                            surf2DLayeredread(&lFF_pre, LGNspikeSurface, 4*x, y, 1+3*j+fPick);
                            /*debug
                            if (tid == 0 && i == 0) {
                                printf("%u-%u, LTP, old_f = %e, lFF_pre = %e\n", tid, i, f, lFF_pre);
                            }*/
                            f += if_decay(lFF_pre, learnE_pre.tauLTP[j], delta_t) * lFF[max_nLearnTypeFF*2 + 2*j+1] * learnE_post.A_LTP[j];
                            /*debug
                            if (tid == 0 && i == 0) {
                                printf("%u-%u, new_f:%e += %e*%e*%e\n", tid, i, f, if_decay(lFF_pre, learnE_pre.tauLTP[j], delta_t), lFF[max_nLearnTypeFF*2 + 2*j+1], learnE_post.A_LTP[j]);
                            }*/
                        }
                    } else {
                        #pragma unroll max_nLearnTypeFF_I
                        for (PosInt j=0; j<learnI_pre.n; j++) {
                            Float lFF_pre;
                            surf2DLayeredread(&lFF_pre, LGNspikeSurface, 4*x, y, 1+3*j+fPick);
                            f += if_decay(lFF_pre, learnI_pre.tauLTP[j], delta_t) * lFF[max_nLearnTypeFF*2 + 2*j+1] * learnI_post.A_LTP[j];
                        }
                    }
                }
                if (threadIdx.x < nE) {
                   if (f < learnE_post.gmin) {
                        f = learnE_post.gmin;
                   }
                   if (f > learnE_post.gmax) {
                        f = learnE_post.gmax;
                   }
                } else {
                   if (f < learnI_post.gmin) {
                        f = learnI_post.gmin;
                   }
                   if (f > learnI_post.gmax) {
                        f = learnI_post.gmax;
                   }
                }
                sLGN[lid] = f;
            }
            // update FF vars; lAvg(E) to be updated after cortical learning if nLearnTypeE > 0
            Float delta_t = 1;
            PosInt cPick = nsp > 0? 1: 0;
            if (nsp > 0) { 
                delta_t -= tsp;
            }
            delta_t *= dt;
            if (threadIdx.x < nE) {
                #pragma unroll max_nLearnTypeFF_E
                for (PosInt i=0; i<learnE_post.n; i++) {
                    lFF[cPick*2*max_nLearnTypeFF + 2*i+0] += nsp; // LTD_E
                    lFF[cPick*2*max_nLearnTypeFF + 2*i+1] += nsp; // TripE
                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+0], learnE_post.tau[2*i+0], delta_t);
                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+1], learnE_post.tau[2*i+1], delta_t);
                }
                if (learning == 3) { // no E, only FF_E, otherwise to be used again and update in recal_G
                    lAvg[cPick] += nsp;
                    decay(lAvg[cPick], learnE_post.tau[2*learnE_post.n], delta_t);
                }
            } else {
                #pragma unroll max_nLearnTypeFF_I
                for (PosInt i=0; i<learnI_post.n; i++) {
                    lFF[cPick*2*max_nLearnTypeFF + 2*i+0] += nsp; // LTD_I
                    lFF[cPick*2*max_nLearnTypeFF + 2*i+1] += nsp; // TripI
                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+0], learnI_post.tau[2*i+0], delta_t);
                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+1], learnI_post.tau[2*i+1], delta_t);
                }
                lAvg[cPick] += nsp;
                decay(lAvg[cPick], learnI_post.tau[2*learnI_post.n], delta_t);
            }
            // store LGN lVars 
            if (threadIdx.x < nE) {
                PosInt eid = nE*blockIdx.x+threadIdx.x;
                #pragma unroll max_nLearnTypeFF_E
                #pragma unroll max_nLearnTypeFF_E
                for (PosInt i=0; i<learnE_post.n; i++) {
                    vLTD_FF_E[nE*gridDim.x*i + eid]  = lFF[cPick*2*max_nLearnTypeFF + 2*i+0];
                    vTrip_FF_E[nE*gridDim.x*i + eid] = lFF[cPick*2*max_nLearnTypeFF + 2*i+1];
                }
                if (learning == 3) { // no E, only FF_E
                    vAvgE[eid*2] = lAvg[cPick]; 
                }
            } else {
                PosInt iid = nI*blockIdx.x+threadIdx.x-nE;
                #pragma unroll max_nLearnTypeFF_I
                for (PosInt i=0; i<learnI_post.n; i++) {
                    vLTD_FF_I[nI*gridDim.x*i + iid]  = lFF[cPick*2*max_nLearnTypeFF + 2*i+0];
                    vTrip_FF_I[nI*gridDim.x*i + iid] = lFF[cPick*2*max_nLearnTypeFF + 2*i+1];
                }
                vAvgI[iid] = lAvg[cPick];
            }
        }
    }
}

//template<int ntimesE, int ntimesI>
__launch_bounds__(1024, 2)
__global__  // <<< nblock[partial], blockSize >>>
void recal_G_mat(
        Float* __restrict__ spikeTrain, // [depth, nblock, blockSize]
        Float* __restrict__ conMat, // [nblock, nearNeighborBlock, blockSize, blockSize]
        Float* __restrict__ delayMat, // [nblock, nearNeighborBlock, blockSize, blockSize]
        Size* __restrict__ nNeighborBlock,
        PosInt* __restrict__ neighborBlockId,
        Float* __restrict__ gE, // [ngTypeE, nV1]
        Float* __restrict__ gI, // [ngTypeI, nV1] 
        Float* __restrict__ hE,
        Float* __restrict__ hI,
        Float* __restrict__ vAvgE, //        post, [                       nblock, nE,       2]
        Float* __restrict__ vLTP_E, //        pre, [nLearnTypeE,    depth, nblock, nE,       2]
        Float* __restrict__ vLTD_E, //       post, [nLearnTypeE,           nblock, nE,       2]
        Float* __restrict__ vTripE, //       post, [nLearnTypeE,           nblock, nE,       2]
        Float* __restrict__ vSTDP_QE,  //  E post, [nLearnTypeQ,           nblock, nE        2]
        Float* __restrict__ vSTDP_QI,  //   I pre, [nLearnTypeQ,    depth, nblock, nI,       2]
        Float* __restrict__ pE,
        Float* __restrict__ pI,
        Size* __restrict__ typeAcc,
        hiprandStateMRG32k3a* __restrict__ rGenCond,
        Float* __restrict__ noisyCondE,
        Float* __restrict__ noisyCondI,
        Float* __restrict__ synFailE,
        Float* __restrict__ synFailI,
        Float dt, ConductanceShape condE, ConductanceShape condI, Size ngTypeE, Size ngTypeI, PosInt currentTimeSlot, Size trainDepth, Size nearNeighborBlock, Size nE, Size nI, Size nV1, Float speedOfThought, int learning, PosInt block_offset, Size nType,
        LearnVarShapeE lE, LearnVarShapeQ lQ, PosInt iChunk, bool noisyH)
{
    // each thread is the post neuron that collects its presynaptic input conductances
    Float ipE[MAX_NGTYPE_E];
    Float ipI[MAX_NGTYPE_I];
    // initialize
    Float local_gE[MAX_NGTYPE_E];
    Float local_hE[MAX_NGTYPE_E];
    PosInt itype;
    #pragma unroll (max_nType)
    for (PosInt i=0; i<nType; i++) {
        if (threadIdx.x < typeAcc[i]) {
            itype = i;
            break;
        }
    }
    #pragma unroll (max_ngTypeE)
    for (PosInt ig=0; ig<ngTypeE; ig++) {
        local_gE[ig] = 0.0f;
        local_hE[ig] = 0.0f;
        ipE[ig] = pE[itype*ngTypeE + ig];
    }
    Float local_gI[MAX_NGTYPE_I];
    Float local_hI[MAX_NGTYPE_I];
    //#pragma unroll (ntimesI)
    #pragma unroll (max_ngTypeI)
    for (PosInt ig=0; ig<ngTypeI; ig++) {
        local_gI[ig] = 0.0f;
        local_hI[ig] = 0.0f;
        ipI[ig] = pI[itype*ngTypeI + ig];
    }
    // TODO: cortical learning
    //Float trip_post[2*max_nLearnTypeE];
    //Float LTD_post[2*max_nLearnTypeE];
    PosInt ipost = (block_offset+blockIdx.x)*blockSize + threadIdx.x;
    hiprandStateMRG32k3a localState = rGenCond[ipost];
    Float post_sInfo = spikeTrain[nV1*currentTimeSlot + ipost];
    Float postNsp = flooring(post_sInfo);
    Float postTsp = postNsp>0? post_sInfo - postNsp: 1;
    Float lAvgE;
    if (learning != 3) {
        if (threadIdx.x < nE) {
            PosInt cPick = postNsp>0? 1:0;
            PosInt eid = (block_offset+blockIdx.x)*nE + threadIdx.x;
            lAvgE = vAvgE[2*eid+cPick];
        }
    }

    __syncthreads();
    #pragma unroll (4)
    for (PosInt ib = 0; ib < nNeighborBlock[blockIdx.x]; ib++) {
		PosInt local_bid = blockIdx.x*nearNeighborBlock + ib;
		PosInt bid = neighborBlockId[local_bid];
        // check for old spikes
        #pragma unroll
        for (PosInt i=0; i<blockSize; i++) {
			PosInt ipre = bid*blockSize + i;
            // access each presynaptic neurons in stride
            // conMat: [nblock,nearNeighborBlock,blockDim.x,blockDim.x] last dim is the post-id: second-last pre-id
            PosIntL mid = static_cast<PosIntL>((local_bid*blockSize + i)*blockSize + threadIdx.x);
            Float strength = conMat[mid];
            if (strength != 0) {
                //Float LTP_pre[max_nLearnTypeE];
                //Float Q_pre[max_nLearnTypeQ];
                Float time2post = delayMat[mid]/speedOfThought;
                Float *local_g;
                Float *local_h;
                Float *ip;
                Float *noisyCond;
                Float synFail;
                Size ngType;
                ConductanceShape *cond;
                if (i < nE) {
                    local_g = local_gE;
                    local_h = local_hE;
                    ngType = ngTypeE;
                    cond = &condE;
                    ip = ipE;
                    noisyCond = noisyCondE;
                    synFail = synFailE[itype];
                } else {
                    local_g = local_gI;
                    local_h = local_hI;
                    ngType = ngTypeI;
                    cond = &condI;
                    ip = ipI;
                    noisyCond = noisyCondI;
                    synFail = synFailI[itype];
                }
                PosInt it2post = static_cast<PosInt>(ceiling(time2post/dt));
                time2post = it2post*dt - time2post;
                if (time2post < 0) {
                    printf("time2post = distance/speed = %1.3e/%1.3e = %1.3e, it2post*dt, %1.3e*%1.3e = %1.3e\n", delayMat[mid], speedOfThought, delayMat[mid]/speedOfThought, it2post, dt, it2post*dt);
                    assert(time2post>=0);
                }
                assert(time2post<dt);
                PosInt j0 = currentTimeSlot - it2post + trainDepth;
                //|<-   it2post               ->|
                //|j0                           |currentTimeSlot
                //|--*--o---|o-*------|---------|---------| thus 2
                //   | tsp  tsp|               
                // ->|         |<- distance adjusted dt
                // ->| distance/speedOfThought  |<-
                //|  |<- time2post
                #pragma unroll (max_ngType)
                for (PosInt ig=0; ig<ngType; ig++) {
                    Float g0 = 0.0;
					Float h0 = 0.0;
                    #pragma unroll 2
                    for (PosInt j=0; j<2; j++) { 
                        // from older to newer
                        PosInt isp = nV1*((j0 + j)%trainDepth) + ipre;
                        Float sInfo = spikeTrain[isp];
                        if (sInfo > 0) { // could fire at the instant t = t_i
                            Float nsp = flooring(sInfo);
                            Float tsp = (sInfo - nsp + j)*dt - time2post;
			        	    /* DEBUG
                                PosInt id = neighborBlockId[blockIdx.x*nearNeighborBlock]*blockDim.x + threadIdx.x;
			        	        if (ipre == 0 && id == 1) {
			        	        	printf("0: %f -> 1 %f, from %u + %u -> %u, %u, %1.7e\n", sInfo, tsp, j0, j, currentTimeSlot, it2post, time2post/dt);
                                    for (PosInt k = 0; k<trainDepth; k++) {
                                        printf("%f,", spikeTrain[nV1*k+ipre]);
                                    }
                                    printf("\n");
			        	        }
                            */
                            if (tsp < dt && tsp >= 0) {
								if (uniform(&localState) > synFail) {
                                	cond->compute_single_input_conductance(g0, h0, strength*nsp*ip[ig], dt-tsp, ig);
								}
                            }
                        }
                        //__syncwarp(); // may not be needed
                    }
                    if (noisyCond[ig] > 0) {
                        Float rand = normal(&localState);
                        Float noise = noisyCond[ig]*strength*ip[ig]*rand;
						/* debug
                        	if (abs(noise) > strength) {
                        	    printf("%u-%u: noise:%e = %e * %e * %f, %e\n", ipost, ipre, noise, noisyCond[ig], strength, ip[ig], rand);
                        	    assert(abs(noise) < strength);
                        	}
						*/
						if (noisyH) {
                        	h0 += noise;
                        	if (h0<0) h0 = 0;
						} else {
                        	g0 += noise;
                        	if (g0<0) g0 = 0;
						}
                    }
                    local_g[ig] += g0;
					local_h[ig] += h0;
                }
            }
            __syncwarp(); // may not be needed
        }
    }
    rGenCond[ipost] = localState;
    if (learning != 3) { // update learning variables
        if (threadIdx.x < nE) {
            PosInt eid = (block_offset+blockIdx.x)*nE + threadIdx.x;
            Float delta_t = dt;
            if (postNsp > 0) {
                delta_t = dt*(1 - postTsp);
            }
            lAvgE += postNsp;
            decay(lAvgE, lE.tau[3*lE.n], delta_t);
            vAvgE[eid*2] = lAvgE;
            /* DEBUG
            if (postNsp > 0) {
                printf("lAvgE:%e of %u, eid:%u is updated\n", lAvgE, ipost, eid);
            }*/
            /*
            #pragma unroll (max_nLearnTypeE)
            for (PosInt i=0; i<lE.n; i++) {
            }*/
        }
    }

    PosInt id = blockIdx.x*blockSize + threadIdx.x;
    //#pragma unroll (ntimesE)
    #pragma unroll (max_ngTypeE)
    for (PosInt ig=0; ig<ngTypeE; ig++) {
        PosInt gid = ig*gridDim.x*blockDim.x + id;
        gE[gid] += local_gE[ig];
        hE[gid] += local_hE[ig];
    }
    //#pragma unroll (ntimesI)
    #pragma unroll (max_ngTypeI)
    for (PosInt ig=0; ig<ngTypeI; ig++) {
        PosInt gid = ig*gridDim.x*blockDim.x + id;
        gI[gid] += local_gI[ig];
        hI[gid] += local_hI[ig];
    }
}

//template<int ntimesE, int ntimesI>
__launch_bounds__(1024, 2)
__global__
void sum_G(
        Size* __restrict__ nVec,
        Float* __restrict__ gEt,
        Float* __restrict__ gE,
        Float* __restrict__ gIt,
        Float* __restrict__ gI,
        Float* __restrict__ hEt,
        Float* __restrict__ hE,
        Float* __restrict__ hIt,
        Float* __restrict__ hI,
        Size ngTypeE, Size ngTypeI)
{
    PosInt id = blockIdx.x*blockDim.x + threadIdx.x;
    if (nVec[id] > 0) {
        //#pragma unroll (ntimesE)
        #pragma unroll (max_ngTypeE)
        for (PosInt ig=0; ig<ngTypeE; ig++) {
            PosInt gid = ig*gridDim.x*blockDim.x + id;
            gE[gid] += gEt[gid];
            hE[gid] += hEt[gid];
        }
        //#pragma unroll (ntimesI) 
        #pragma unroll (max_ngTypeI) 
        for (PosInt ig=0; ig<ngTypeI; ig++) {
            PosInt gid = ig*gridDim.x*blockDim.x + id;
            gI[gid] += gIt[gid];
            hI[gid] += hIt[gid];
        }
    }
    #ifdef DEBUG
        if (id == 1||id == 2) {
            printf("#%u gE[0] = %f, gE[1] = %f, gI = %f\n", id, gE[id], gE[gridDim.x*blockDim.x + id], gI[id]);
        }
    #endif
}

////template<int ntimesFF, int ntimesE, int ntimesI>
//__launch_bounds__(1024,2)
//__global__ 
//void compute_V_collect_spike_learnFF0(
//        Float* __restrict__ v,
//        Float* __restrict__ gFF, // not in chunks
//        Float* __restrict__ hFF,
//        Float** __restrict__ gE, // in chunks
//        Float** __restrict__ gI,
//        Float** __restrict__ hE,
//        Float** __restrict__ hI,
//        Size* __restrict__ nLGN,
//        Float* __restrict__ sLGN,
//        int* __restrict__ LGN_idx,
//        int* __restrict__ LGN_idy,
//        Float* __restrict__ tBack,
//        Float* __restrict__ spikeTrain, //         [                depth, nblock, blockSize  ]
//        Float* __restrict__ vLTD_FF_E, //    post, [nLearnTypeFF_E,        nblock, nE         ]
//        Float* __restrict__ vTrip_FF_E, //   post, [nLearnTypeFF_E,        nblock, nE         ]
//        Float* __restrict__ vLTD_FF_I, //    post, [nLearnTypeFF_I,        nblock, nI         ]
//        Float* __restrict__ vTrip_FF_I, //   post, [nLearnTypeFF_I,        nblock, nI         ]
//        Float* __restrict__ vAvgE, //        post, [                       nblock, nE,       2]
//        Float* __restrict__ vAvgI, //        post, [                       nblock, nI         ]
//        Float* __restrict__ vLTP_E, //        pre, [nLearnTypeE,    depth, nblock, nE,       2]
//        Float* __restrict__ vLTD_E, //       post, [nLearnTypeE,           nblock, nE,       2]
//        Float* __restrict__ vTripE, //       post, [nLearnTypeE,           nblock, nE,       2]
//        Float* __restrict__ vSTDP_QE,  //  E post, [nLearnTypeQ,           nblock, nE        2]
//        Float* __restrict__ vSTDP_QI,  //   I pre, [nLearnTypeQ,    depth, nblock, nI,       2]
//        Float* __restrict__ pFF,
//        Float* __restrict__ vR_type,
//        Float* __restrict__ vT_type,
//        Float* __restrict__ vThres_type,
//        Float* __restrict__ gL_type,
//        Float* __restrict__ tRef_type,
//        Size* __restrict__ typeAcc,
//        hiprandStateMRG32k3a* __restrict__ rGenCond,
//        Float* __restrict__ noisyCondFF,
//        Float* __restrict__ synFailFF,
//        PosInt currentTimeSlot, Size trainDepth, Size max_nLGN, Size ngTypeFF, Size ngTypeE, Size ngTypeI, ConductanceShape condFF, ConductanceShape condE, ConductanceShape condI, Float dt, Size maxChunkSize, Size remainChunkSize, PosInt iSizeSplit, Size nChunk, Size nE, Size nI, Size nV1, int learning, int varSlot, Size nType,
//        LearnVarShapeFF_E_pre  learnE_pre,  LearnVarShapeFF_I_pre  learnI_pre, 
//        LearnVarShapeFF_E_post learnE_post, LearnVarShapeFF_I_post learnI_post, 
//        LearnVarShapeE learnE, LearnVarShapeQ learnQ, int iModel)
//{
//	//assert(blockDim.x == blockSize);
//    PosInt tid = blockIdx.x * blockDim.x + threadIdx.x;
//    PosInt iChunk;
//    Size chunkSize;
//    PosInt cid;
//    if (blockIdx.x >= iSizeSplit*maxChunkSize) {
//        iChunk = iSizeSplit + (blockIdx.x-iSizeSplit*maxChunkSize)/remainChunkSize;
//        chunkSize = remainChunkSize*blockDim.x;
//        cid = tid - (iSizeSplit*maxChunkSize + (iChunk-iSizeSplit)*remainChunkSize)*blockDim.x;
//    } else {
//        iChunk = blockIdx.x/maxChunkSize;
//        chunkSize = maxChunkSize*blockDim.x;
//        cid = tid - iChunk*maxChunkSize*blockDim.x;
//    }
//
//    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
//    // TODO: load individual gl, tref
//    PosInt itype;
//    #pragma unroll max_nType
//    for (PosInt j=0; j<nType; j++) {
//        if (threadIdx.x < typeAcc[j]) {
//            itype = j;
//            break;
//        }
//    }
//    IF* singleNeuron;
//    if (iModel == 0) {
//        singleNeuron = new LIF(v[tid], tBack[tid], vR[itype], vT[itype], tRef[itype], gL[itype]);
//    }
//    /* set a0 b0 and a1 b1 */
//    // cond FF
//    //#pragma unroll (MAX_NGTYPE_FF)
//    Size m = nLGN[tid];
//    //Size nsp_FFt = 0;
//    hiprandStateMRG32k3a localState = rGenCond[tid];
//    Float gE_t0 = 0.0;
//	Float gE_t1 = 0.0;
//    #pragma unroll (max_ngTypeFF) //(ntimesFF)
//    for (PosInt ig=0; ig<ngTypeFF; ig++) {
//        PosInt gid = nV1*ig + tid; // not in chunks
//        Float g = gFF[gid];
//        Float h = hFF[gid];
//        //if (tid == 16737) {
//        //    printf("%u-%u: g:%e h:%e\n", tid, ig, g, h);
//        //}
//        gE_t0 += g;
//        // conductance of the end of the last time step
//        condFF.decay_conductance(g, h, dt, ig); //  decayed to the end of the current step
//        // Get LGN input
//
//        #pragma unroll (4)
//        for (PosInt i = 0; i<m; i++) {
//            PosInt lid = tid*max_nLGN + i;
//            Float f = sLGN[lid];
//            int x = LGN_idx[lid];
//            int y = LGN_idy[lid];
//            Float sInfo;
//            surf2DLayeredread(&sInfo, LGNspikeSurface, 4*x, y, 0);
//            Float nsp = flooring(sInfo); // integer part: #spikes
//            Float tsp = sInfo - nsp; // decimal part: normalized mean tsp
//            Float str = f * pFF[itype*ngTypeFF + ig];
//            Float g0 = 0.0;
//            if (nsp > 0 && uniform(&localState) > synFailFF[ig]) {
//                condFF.compute_single_input_conductance(g0, h, str*nsp, dt*(1-tsp), ig);
//            }
//            if (noisyCondFF[ig] > 0) {
//                Float rand = normal(&localState);
//                Float noise = noisyCondFF[ig]*str*pFF[itype*ngTypeFF + ig]*rand;
//                g0 += noise;
//                if (g0<0) g0 = 0;
//            //if (abs(noise) > str || tid == 16737) {
//            //    printf("%u-%u: noise:%e = %e * %e * %f, %e\n", tid, lid, noise, noisyCondFF[ig], str, pFF[itype*ngTypeFF+ig], rand);
//            //    assert(abs(noise) < str);
//            //}
//            //if (tid == 16737) {
//            //    printf("str:%e * noisyCondFF:%e * rand:%e = %e\n", str, noisyCondFF[ig], rand, str*noisyCondFF[ig] * rand);
//            //}
//            }
//            g += g0;
//        }
//        gE_t1 += g;
//        gFF[gid] = g;
//        hFF[gid] = h;
//    }
//    rGenCond[tid] = localState;
//    // cond E 
//    //#pragma unroll (MAX_NGTYPE_E)
//    #pragma unroll (max_ngTypeE) 
//    for (PosInt ig=0; ig<ngTypeE; ig++) {
//        PosInt gid = chunkSize*ig + cid;
//        Float g = gE[iChunk][gid];
//        Float h = hE[iChunk][gid];
//        gE_t0 += g;
//        condE.decay_conductance(g, h, dt, ig); 
//        gE_t1 += g;
//        gE[iChunk][gid] = g;
//        hE[iChunk][gid] = h;
//    }
//    // cond I 
//    Float gI_t0 = 0.0;
//	Float gI_t1 = 0.0;
//    //#pragma unroll (MAX_NGTYPE_I)
//    //#pragma unroll (ntimesI)
//    #pragma unroll (max_ngTypeI)
//    for (PosInt ig=0; ig<ngTypeI; ig++) {
//        PosInt gid = chunkSize*ig + cid;
//        Float g = gI[iChunk][gid];
//        Float h = hI[iChunk][gid];
//        gI_t0 += g;
//        condI.decay_conductance(g, h, dt, ig); 
//        gI_t1 += g;
//        gI[iChunk][gid] = g;
//        hI[iChunk][gid] = h;
//    }
//    singleNeuron->set_p0(gE_t0, gI_t0);
//    singleNeuron->set_p1(gE_t1, gI_t1);
//    /* evolve g to t+dt with ff input only */
//    // step
//    Float sInfo = step(singleNeuron, dt, /*the last 3 args are for deugging*/ tid, gE_t1, gI_t1);
//    spikeTrain[nV1*currentTimeSlot + tid] = sInfo;
//
//    //if (isnan(sInfo) || tid == 16737) {
//    //    Size nsp = flooring(sInfo);
//    //    printf("%u(%u): spiked at sInfo: %f, %u + %f, gFF[0] = %f, gFF[1] = %f, gE[0] = %f, gE[1] = %f, gE_t = %f, gI_t = %f\n", tid, cid, sInfo, nsp, sInfo - nsp, gFF[tid], gFF[tid+nV1], gE[iChunk][cid], gE[iChunk][cid + chunkSize], gE_t1, gI_t1);
//    //    assert(!isnan(sInfo));
//    //}
//    /*DEBUG
//    //if (sInfo > 0 && (threadIdx.x == 0 || threadIdx.x == 768)) {
//    if (sInfo > 0 && (gI_t0 > 0 || threadIdx.x >= nE)) {
//        Size nsp = flooring(sInfo);
//        printf("%u(%u): spiked at sInfo: %u + %f, gF = %e(%u), gE = %e, gI = %e\n", tid, cid, nsp, sInfo - nsp, gFF[tid], m, gE[iChunk][cid], gI_t0);
//    }*/
//	v[tid] = singleNeuron->v;
//    tBack[tid] = singleNeuron->tBack;
//    delete []singleNeuron;
//    if (learning) {
//        Float nsp = flooring(sInfo);
//        Float tsp = sInfo>0? sInfo - nsp: 1;
//        // will compute ff learning, first row at start of time step, second row at tsp
//        Float lFF[2*2*max_nLearnTypeFF]; // row 0: start, row 1: sp
//        Float lAvg[2];
//        // only temporary store
//        Float lE[3*max_nLearnTypeE];
//        Float lQ[max_nLearnTypeQ];
//        // read ff (post) lVar
//        PosInt eid = nE*blockIdx.x+threadIdx.x;
//        if (learning < 4) { // read regardless of cortical spike 
//            if (threadIdx.x < nE) {
//                #pragma unroll max_nLearnTypeFF_E
//                for (PosInt i=0; i<learnE_post.n; i++) {
//                    lFF[2*i+0] =  vLTD_FF_E[nE*gridDim.x*i + eid];
//                    lFF[2*i+1] = vTrip_FF_E[nE*gridDim.x*i + eid];
//                }
//                lAvg[0] = vAvgE[eid*2];
//            } else {
//                if (learnI_post.n) {
//                    PosInt iid = nI*blockIdx.x+threadIdx.x-nE;
//                    #pragma unroll max_nLearnTypeFF_I
//                    for (PosInt i=0; i<learnI_post.n; i++) {
//                        lFF[2*i+0] =  vLTD_FF_I[nI*gridDim.x*i + iid];
//                        lFF[2*i+1] = vTrip_FF_I[nI*gridDim.x*i + iid];
//                    }
//                    lAvg[0] = vAvgI[iid];
//                }
//            }
//        }
//        if (nsp > 0) {
//            if (learning !=3) { // E and Q are active, read cortical lVar and AvgE if previouly not read
//                if (threadIdx.x < nE) {
//                    // E
//                    #pragma unroll max_nLearnTypeE
//                    for (PosInt i=0; i<learnE.n; i++) {
//                        lE[3*i+0] = vLTP_E[(nE*gridDim.x*trainDepth*i + nE*gridDim.x*currentTimeSlot + eid)*2];
//                        lE[3*i+1] = vLTD_E[(nE*gridDim.x*i + eid)*2];
//                        lE[3*i+2] = vTripE[(nE*gridDim.x*i + eid)*2];
//                    }
//                    // Q_E
//                    #pragma unroll max_nLearnTypeQ
//                    for (PosInt i=0; i<learnQ.n; i++) {
//                        lQ[i] = vSTDP_QE[(nE*gridDim.x*i + eid)*2];
//                    }
//                    if (learning == 4) { // otherwise already read
//                        lAvg[0] = vAvgE[eid*2];
//                    }
//                } else {
//                    // Q_I
//                    PosInt iid = nI*(gridDim.x*currentTimeSlot + blockIdx.x) + threadIdx.x-nE;
//                    #pragma unroll max_nLearnTypeQ
//                    for (PosInt i=0; i<learnQ.n; i++) {
//                        lQ[i] = vSTDP_QI[(nI*gridDim.x*trainDepth*i + iid)*2];
//                    }
//                }
//            }
//            if (learning < 4) { // compute ff post vars' decay till tsp
//                if (threadIdx.x < nE) {
//                    #pragma unroll max_nLearnTypeFF_E
//                    for (PosInt i=0; i<learnE_post.n; i++) {
//                        lFF[2*max_nLearnTypeFF + 2*i+0] = lFF[2*i+0];
//                        lFF[2*max_nLearnTypeFF + 2*i+1] = lFF[2*i+1];
//                    }
//                    #pragma unroll max_nLearnTypeFF_E
//                    for (PosInt i=0; i<learnE_post.n; i++) {
//                        decay(lFF[2*max_nLearnTypeFF + 2*i+0], learnE_post.tau[2*i+0], tsp);
//                        decay(lFF[2*max_nLearnTypeFF + 2*i+1], learnE_post.tau[2*i+1], tsp);
//                    }
//                } else {
//                    if (learnI_post.n) {
//                        #pragma unroll max_nLearnTypeFF_I
//                        for (PosInt i=0; i<learnI_post.n; i++) {
//                            lFF[2*max_nLearnTypeFF + 2*i+0] = lFF[2*i+0];
//                            lFF[2*max_nLearnTypeFF + 2*i+1] = lFF[2*i+1];
//                        }
//                        #pragma unroll max_nLearnTypeFF_I
//                        for (PosInt i=0; i<learnI_post.n; i++) {
//                            decay(lFF[2*max_nLearnTypeFF + 2*i+0], learnI_post.tau[2*i+0], tsp);
//                            decay(lFF[2*max_nLearnTypeFF + 2*i+1], learnI_post.tau[2*i+1], tsp);
//                        }
//                        lAvg[1] = lAvg[0];
//                        decay(lAvg[1], learnI_post.tau[2*learnI_post.n], tsp);
//                    }
//                }
//            }
//            if (threadIdx.x < nE) { // compute AvgE
//                lAvg[1] = lAvg[0];
//                decay(lAvg[1], learnE_post.tau[2*learnE_post.n], tsp);
//            }
//            if (learning !=3) { // compute and store lVars of E, Q and AvgE
//                // compute
//                if (threadIdx.x < nE) {
//                    #pragma unroll max_nLearnTypeE
//                    for (PosInt i=0; i<learnE.n; i++) {
//                        decay(lE[3*i+0], learnE.tau[3*i+0], tsp);
//                        decay(lE[3*i+1], learnE.tau[3*i+1], tsp);
//                        decay(lE[3*i+2], learnE.tau[3*i+2], tsp);
//                    }
//                    #pragma unroll max_nLearnTypeQ
//                    for (PosInt i=0; i<learnQ.n; i++) {
//                        decay(lQ[i], learnQ.tau[2*i+0], tsp); // Q_E
//                    }
//                } else {
//                    #pragma unroll max_nLearnTypeQ
//                    for (PosInt i=0; i<learnQ.n; i++) {
//                        decay(lQ[i], learnQ.tau[2*i+1], tsp); // Q_I
//                    }
//                }
//                // store
//                if (threadIdx.x < nE) {
//                    #pragma unroll max_nLearnTypeE
//                    for (PosInt i=0; i<learnE.n; i++) {
//                         vLTP_E[(nE*gridDim.x*trainDepth*i + nE*gridDim.x*currentTimeSlot + eid)*2 + 1] = lE[3*i+0];
//                         vLTD_E[(nE*gridDim.x*i + eid)*2 + 1] = lE[3*i+1];
//                         vTripE[(nE*gridDim.x*i + eid)*2 + 1] = lE[3*i+2];
//                    }
//                    vAvgE[2*eid+1] = lAvg[1];
//                    #pragma unroll max_nLearnTypeQ
//                    for (PosInt i=0; i<learnQ.n; i++) { // store to the second slot of the array
//                        vSTDP_QE[(nE*gridDim.x*i + eid)*2 + 1] =  lQ[i];
//                    }
//                } else {
//                    PosInt iid = nI*(gridDim.x*currentTimeSlot + blockIdx.x) + threadIdx.x-nE;
//                    #pragma unroll max_nLearnTypeQ
//                    for (PosInt i=0; i<learnQ.n; i++) { // store to the second slot of the array
//                        vSTDP_QI[(nI*gridDim.x*trainDepth*i + iid)*2 + 1] =  lQ[i];
//                    }
//                }
//            }
//        }
//        // learn LGN connection and update LGN lVars
//        if (learning < 4 && (threadIdx.x < nE || learnI_pre.n)) { 
//            // learn
//            for (PosInt i = 0; i<m; i++) {
//                PosInt lid = tid*max_nLGN + i;
//                Float f = sLGN[lid];
//                int x = LGN_idx[lid];
//                int y = LGN_idy[lid];
//                Float sInfo_FF;
//                surf2DLayeredread(&sInfo_FF, LGNspikeSurface, 4*x, y, 0);
//                Float nsp_FF = flooring(sInfo_FF);
//                Float tsp_FF = sInfo_FF > 0? sInfo_FF - nsp_FF: 1;
//                if (nsp_FF > 0) { // LTD, regarless of post spike
//                    PosInt cPick;
//                    Float delta_t;
//                    if (tsp_FF < tsp) {
//                        cPick = 0; // from start
//                        delta_t = tsp_FF;
//                    } else {
//                        cPick = 1; // from tsp
//                        delta_t = tsp_FF-tsp;
//                    }
//                    delta_t *= dt;
//                    if (threadIdx.x < nE) {
//                        #pragma unroll max_nLearnTypeFF_E
//                        for (PosInt j=0; j<learnE_pre.n; j++) {
//                            Float A_LTD = learnE_post.A_ratio[j] * learnE_pre.tauLTP[j] * lAvg[cPick] * lAvg[cPick]/learnE_post.targetFR;
//                            //Float A_LTD = learnFF_E.A_LTP[j]; TODO: alternative homeostatic design
//                            /*debug
//							if (tid == 0 && i == 0) {
//                                printf("%u-%u, A_LTD: %e = %e*%e*%e^2/%e\n", tid, i, A_LTD, learnE_post.A_ratio[j], learnE_pre.tauLTP[j], lAvg[cPick], learnE_post.targetFR);
//								printf("%u-%u, old_f: %e\n", tid, i, f);
//                            }*/
//                            f -= if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnE_post.tau[2*j+0], delta_t) * A_LTD;
//                            /*debug
//							if (tid == 0 && i == 0) {
//								printf("%u-%u, new_f: %e\n", tid, i, f);
//								Float df = if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnE_post.tau[2*j+0], delta_t) * A_LTD;
//								printf("%u-%u, df %e = %e*%e\n", tid, i, df, if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnE_post.tau[2*j+0], delta_t), A_LTD);
//							}*/
//                        }
//                    } else {
//                        #pragma unroll max_nLearnTypeFF_I
//                        for (PosInt j=0; j<learnI_pre.n; j++) {
//                            Float A_LTD = learnI_post.A_ratio[j] * learnI_pre.tauLTP[j] * lAvg[cPick] * lAvg[cPick]/learnE_post.targetFR;
//                            f -= if_decay(lFF[cPick*max_nLearnTypeFF*2 + 2*j+0], learnI_post.tau[2*j+0], delta_t) * A_LTD;
//                        }
//                    }
//                } 
//                if (nsp > 0) { // LTP, regardless of pre spike
//                    PosInt fPick;
//                    Float delta_t;
//                    if (tsp_FF < tsp) {
//                        fPick = 2;
//                        delta_t = tsp-tsp_FF;
//                    } else {
//                        fPick = varSlot;
//                        delta_t = tsp;
//                    }
//                    delta_t *= dt;
//                    if (threadIdx.x < nE) {
//                        #pragma unroll max_nLearnTypeFF_E
//                        for (PosInt j=0; j<learnE_pre.n; j++) {
//                            Float lFF_pre;
//                            surf2DLayeredread(&lFF_pre, LGNspikeSurface, 4*x, y, 1+3*j+fPick);
//                            /*debug
//                            if (tid == 0 && i == 0) {
//                                printf("%u-%u, LTP, old_f = %e, lFF_pre = %e\n", tid, i, f, lFF_pre);
//                            }*/
//                            f += if_decay(lFF_pre, learnE_pre.tauLTP[j], delta_t) * lFF[max_nLearnTypeFF*2 + 2*j+1] * learnE_post.A_LTP[j];
//                            /*debug
//                            if (tid == 0 && i == 0) {
//                                printf("%u-%u, new_f:%e += %e*%e*%e\n", tid, i, f, if_decay(lFF_pre, learnE_pre.tauLTP[j], delta_t), lFF[max_nLearnTypeFF*2 + 2*j+1], learnE_post.A_LTP[j]);
//                            }*/
//                        }
//                    } else {
//                        #pragma unroll max_nLearnTypeFF_I
//                        for (PosInt j=0; j<learnI_pre.n; j++) {
//                            Float lFF_pre;
//                            surf2DLayeredread(&lFF_pre, LGNspikeSurface, 4*x, y, 1+3*j+fPick);
//                            f += if_decay(lFF_pre, learnI_pre.tauLTP[j], delta_t) * lFF[max_nLearnTypeFF*2 + 2*j+1] * learnI_post.A_LTP[j];
//                        }
//                    }
//                }
//                if (threadIdx.x < nE) {
//                   if (f < learnE_post.gmin) {
//                        f = learnE_post.gmin;
//                   }
//                   if (f > learnE_post.gmax) {
//                        f = learnE_post.gmax;
//                   }
//                } else {
//                   if (f < learnI_post.gmin) {
//                        f = learnI_post.gmin;
//                   }
//                   if (f > learnI_post.gmax) {
//                        f = learnI_post.gmax;
//                   }
//                }
//                sLGN[lid] = f;
//            }
//            // update FF vars; lAvg(E) to be updated after cortical learning if nLearnTypeE > 0
//            Float delta_t = 1;
//            PosInt cPick = nsp > 0? 1: 0;
//            if (nsp > 0) { 
//                delta_t -= tsp;
//            }
//            delta_t *= dt;
//            if (threadIdx.x < nE) {
//                #pragma unroll max_nLearnTypeFF_E
//                for (PosInt i=0; i<learnE_post.n; i++) {
//                    lFF[cPick*2*max_nLearnTypeFF + 2*i+0] += nsp; // LTD_E
//                    lFF[cPick*2*max_nLearnTypeFF + 2*i+1] += nsp; // TripE
//                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+0], learnE_post.tau[2*i+0], delta_t);
//                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+1], learnE_post.tau[2*i+1], delta_t);
//                }
//                if (learning == 3) { // no E, only FF_E, otherwise to be used again and update in recal_G
//                    lAvg[cPick] += nsp;
//                    decay(lAvg[cPick], learnE_post.tau[2*learnE_post.n], delta_t);
//                }
//            } else {
//                #pragma unroll max_nLearnTypeFF_I
//                for (PosInt i=0; i<learnI_post.n; i++) {
//                    lFF[cPick*2*max_nLearnTypeFF + 2*i+0] += nsp; // LTD_I
//                    lFF[cPick*2*max_nLearnTypeFF + 2*i+1] += nsp; // TripI
//                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+0], learnI_post.tau[2*i+0], delta_t);
//                    decay(lFF[cPick*2*max_nLearnTypeFF + 2*i+1], learnI_post.tau[2*i+1], delta_t);
//                }
//                lAvg[cPick] += nsp;
//                decay(lAvg[cPick], learnI_post.tau[2*learnI_post.n], delta_t);
//            }
//            // store LGN lVars 
//            if (threadIdx.x < nE) {
//                PosInt eid = nE*blockIdx.x+threadIdx.x;
//                #pragma unroll max_nLearnTypeFF_E
//                #pragma unroll max_nLearnTypeFF_E
//                for (PosInt i=0; i<learnE_post.n; i++) {
//                    vLTD_FF_E[nE*gridDim.x*i + eid]  = lFF[cPick*2*max_nLearnTypeFF + 2*i+0];
//                    vTrip_FF_E[nE*gridDim.x*i + eid] = lFF[cPick*2*max_nLearnTypeFF + 2*i+1];
//                }
//                if (learning == 3) { // no E, only FF_E
//                    vAvgE[eid*2] = lAvg[cPick]; 
//                }
//            } else {
//                PosInt iid = nI*blockIdx.x+threadIdx.x-nE;
//                #pragma unroll max_nLearnTypeFF_I
//                for (PosInt i=0; i<learnI_post.n; i++) {
//                    vLTD_FF_I[nI*gridDim.x*i + iid]  = lFF[cPick*2*max_nLearnTypeFF + 2*i+0];
//                    vTrip_FF_I[nI*gridDim.x*i + iid] = lFF[cPick*2*max_nLearnTypeFF + 2*i+1];
//                }
//                vAvgI[iid] = lAvg[cPick];
//            }
//        }
//    }
//}
