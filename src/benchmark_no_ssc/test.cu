#include "hip/hip_runtime.h"
#include "test.h"

int main(int argc, char *argv[])
{
    std::ofstream v_file, spike_file, gE_file, gI_file;
    float time;
    //hipEventCreateWithFlags(&gReady, hipEventDisableTiming);
    hiprandStateMRG32k3a *state;
    unsigned long long seed;
    //seed = 183765712;
    seed = std::time(0);
    int device;
    int b1,b2;
	int ms = 1;
    b1 = 160;
    b2 = 128;
    bool printStep = false;
    bool moreSharedMemThanBlocks = true;
    double flatRate = 16000.0f; // Hz
    double t = 2.5f;
    unsigned int nstep = 200;
    double ffsE = 3e-3;
    double s0 = 1e-2*ffsE;
    double ffsI = 5e-2;
    /* Overwrite parameters */
    for (int i = 0; i<argc; i++) {
        printf(argv[i]);
        printf(" ");
    }
    printf("\n");
    if (argc == 2) {
        sscanf(argv[argc-1],"%u",&seed); 
    }
    if (argc == 3) {
        sscanf(argv[argc-1],"%d",&b2); 
        sscanf(argv[argc-2],"%d",&b1); 
    }
    if (argc == 4) {
        sscanf(argv[argc-1],"%d",&b2); 
        sscanf(argv[argc-2],"%d",&b1); 
        sscanf(argv[argc-3],"%d",&nstep);
    }
    if (argc == 5) {
        sscanf(argv[argc-1],"%u",&seed);
        sscanf(argv[argc-2],"%d",&b2);
        sscanf(argv[argc-3],"%d",&b1);
        sscanf(argv[argc-4],"%d",&nstep);
    }
	if (argc == 6) {
		sscanf(argv[argc - 1], "%d", &ms);
		sscanf(argv[argc - 2], "%u", &seed);
		sscanf(argv[argc - 3], "%d", &b2);
		sscanf(argv[argc - 4], "%d", &b1);
		sscanf(argv[argc - 5], "%d", &nstep);
	}
    printf("%i x %i, %i steps, seed = %u\n", b1, b2, nstep, seed);
	unsigned int networkSize = b1*b2;
	double s = s0/ (networkSize);
    if (networkSize/10.0 != float(networkSize/10)) {
        printf("To have higher computation occupancy make a factor of 10 in networkSize\n");
    }
	int warpSize = 32;
	int init_b2 = warpSize;
	int init_b1 = networkSize / init_b2;
    unsigned int nE = networkSize*3/4;
    unsigned int nI = networkSize-nE;
    double dt = t/float(nstep); // ms
    //double flatRate = 0.0f; // Hz
    printf("designated input rate = %3.1fHz\n", flatRate);
	printf("dt = %f ms\n", dt);
    printf("nE = %i, nI = %i\n", nE, networkSize-nE);
    printf("t = %f x %i = %f\n", dt, nstep, t);
	double fInput = flatRate / 1000.0f * dt;
	int _nInput = ceil(flatRate / 1000.0f * dt);
    int nskip = 1;
    if (fInput < 1.0) {
        assert(_nInput == 1);
        nskip = ceil(_nInput/fInput);
    }
	#ifdef TEST_WITH_MANUAL_FFINPUT
		printf("for testing purpose, feedforward input is set to %i per %fms\n", _nInput, nskip*dt);
        printf("realized input rate = %fkHz\n", float(_nInput)/(dt*nskip));
		printf("for manual testing, please change the inputTime manually in source and recompile\n");
		cpu_version(networkSize, _nInput, nskip, nstep, dt, nE, s, ffsE, ffsI);
	#endif
	if (networkSize / float(warpSize) != float(networkSize / warpSize)) {
		printf("please make networkSize multiples of %i to run the gpu test.\n", warpSize);
		return EXIT_FAILURE;
	}
    struct hipDeviceProp_t properties;  
    double *v, *gE, *gI, *preMat; 
    int *eventRate, *d_eventRate;
    double *d_v, *d_gE, *d_gI, *d_hE, *d_hI, *d_fE, *d_fI, *d_preMat, *d_inputRate;
    double *d_a, *d_b;
    double *gactVec, *hactVec;
    double *leftTimeRate, *lastNegLogRand;
    double *spikeTrain, *d_spikeTrain, *tBack;
    /* to be extended */
    bool presetInit = false;
    unsigned int ngTypeE = 2;
    unsigned int ngTypeI = 1;
    double riseTimeE[2] = {1.0f, 5.0f}; // ms
    double riseTimeI[1] = {1.0f};
    double decayTimeE[2] = {3.0f, 80.0f};
    double decayTimeI[1] = {5.0f};

    ConductanceShape condE(riseTimeE, decayTimeE, ngTypeE);
    ConductanceShape condI(riseTimeI, decayTimeI, ngTypeI);

    while (init_b2 < 256 && init_b1 > 1) {
        init_b2 = init_b2*2;
        init_b1 = init_b1/2;
    }
    printf("init size %i, %i\n", init_b1, init_b2);

	/* check for double precision support */
	CUDA_CALL(hipGetDevice(&device));
	CUDA_CALL(hipGetDeviceProperties(&properties, device));
	if (!(properties.major >= 2 || (properties.major == 1 && properties.minor >= 3))) {
		printf(" double precision not supported\n");
		return EXIT_FAILURE;
	}

    unsigned int nbatch, batchEnd, batchStep;
    unsigned int ngType;
    if (ngTypeE > ngTypeI) {
        ngType = ngTypeE;
    } else {
        ngType = ngTypeI;
    }
    // v, gE, gI, spikeTrain
    unsigned int hostMemToDiskPerStep = ceil(networkSize * (sizeof(double) + ngTypeE*sizeof(double) + ngTypeI*sizeof(double) + sizeof(int) )/(1024*1024));
    //batchStep = floor(HALF_MEMORY_OCCUPANCY/hostMemToDiskPerStep);
    batchStep = 1;
    if (batchStep < 10) {
        printf("consider increase HALF_MEMORY_OCCUPANCY, batch step = %i\n", batchStep);
    } else {
        if (batchStep == 0) {
            printf("increase HALF_MEMORY_OCCUPANCY, memory to write on disk per step: %i Mb", hostMemToDiskPerStep);
            return EXIT_FAILURE;
        }
    }
    nbatch = nstep/batchStep; 
    batchEnd = nstep - batchStep*nbatch;
    int alt = 1;
    hipEvent_t iStart, iStop;
    hipEventCreate(&iStart);
    hipEventCreate(&iStop);
    CUDA_CALL(hipEventRecord(iStart, 0));
    /* Allocate space for results on host */
    //pinned memory
    CUDA_CALL(hipHostMalloc((void**)&v,          networkSize * sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&gE,         networkSize * ngTypeE * sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&gI,         networkSize * ngTypeI *sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&spikeTrain, networkSize * sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&eventRate,  networkSize * sizeof(int) * batchStep * alt));
    preMat = (double *)calloc(networkSize, sizeof(double));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&d_v,            networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_gE,           networkSize * ngTypeE *sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_gI,           networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_hE,           networkSize * ngTypeE *sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_hI,           networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_fE,           networkSize * ngTypeE *sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_fI,           networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_a,            networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_b,            networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_inputRate,    networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_eventRate,    networkSize * sizeof(int)));
    CUDA_CALL(hipMalloc((void **)&d_spikeTrain,   networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&tBack,          networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&gactVec,        networkSize * ngType * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&hactVec,        networkSize * ngType * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_preMat,       networkSize * networkSize * sizeof(double)));
    /* Allocate space for rng on device */
    CUDA_CALL(hipMalloc((void **)&state,          networkSize * sizeof(hiprandStateMRG32k3a)));
    CUDA_CALL(hipMalloc((void **)&leftTimeRate,   networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&lastNegLogRand, networkSize * sizeof(double)));
    /* Allocate space for partial reduce results on device */
    
    int maxTPB = properties.maxThreadsPerBlock/ms;
    int EmaxTPB, ImaxTPB;
    int mE, mI; 
    if (maxTPB < nE) {
        EmaxTPB = maxTPB;
        mE = (nE+EmaxTPB-1)/EmaxTPB;
        EmaxTPB = nE/mE;
    } else {
        mE = 1;
        EmaxTPB = nE;
    }
    while (EmaxTPB*mE != nE && EmaxTPB > EmaxTPB/2) {
        mE = mE + 1;
        EmaxTPB = nE/mE;
    }

    if (maxTPB < nI) {
        ImaxTPB = maxTPB;
        mI = (nI+ImaxTPB-1)/ImaxTPB;
        ImaxTPB = nI/mI;
    } else {
        mI = 1;
        ImaxTPB = nI;
    }
    while (ImaxTPB*mI != nI && ImaxTPB > ImaxTPB/2) {
        mI = mI + 1;
        ImaxTPB = nI/mI;
    }

    dim3 rgE_b1, rgI_b1;
    int EnTPB = networkSize/(networkSize/EmaxTPB);
    int InTPB = networkSize/(networkSize/ImaxTPB);
    if (EnTPB > maxTPB) {
        EnTPB = maxTPB;
    }
    if (InTPB > maxTPB) {
        InTPB = maxTPB;
    }
    dim3 rgE_b2(EnTPB,1);
    dim3 rgI_b2(InTPB,1);
    printf("mE = %i, mI = %i\n", mE, mI);
    //dim3 rgE_b2(EmaxTPB,1);
    //dim3 rgI_b2(ImaxTPB,1);
    int msE = 1; // multiple shared actVec load per thread
    int msI = 1;
    int s_actVec_lE; // length of shared actVec
    int s_actVec_lI;
    unsigned int rgE_shared;
    unsigned int rgI_shared;

    s_actVec_lE = EmaxTPB;
    rgE_shared = 2*ngTypeE*s_actVec_lE*sizeof(double);
    if (rgE_shared > properties.sharedMemPerBlock) {
        printf("E: The size of the requested shared memory %iKb by recal_G is not available\n", rgE_shared/1024);
        return EXIT_FAILURE;
    } else {
        if (moreSharedMemThanBlocks) {
            while (rgE_shared*2  < properties.sharedMemPerBlock && mE/float(msE*2) == float(mE/(msE*2))) {
                msE = msE * 2;
                rgE_shared = rgE_shared * 2;
            }
        }
    }
    s_actVec_lE = msE*s_actVec_lE; // number of actVec each chunk dump into shared mem, msE multiples of maxTPB
    rgE_b1.x = nE/s_actVec_lE; // chunks of maxTPB neurons
    rgE_b1.y = networkSize/EnTPB; // total number of presynaptic neurons divided by the the shared actVec
    printf("E: recal_G<<<(%i,%i,%i)x(%i,%i,%i), %iKb>>>, msE = %i\n", rgE_b1.x, rgE_b1.y, rgE_b1.z, rgE_b2.x, rgE_b2.y, rgE_b2.z, rgE_shared/1024, msE);

    s_actVec_lI = ImaxTPB;
    rgI_shared = 2*ngTypeI*s_actVec_lI*sizeof(double);
    if (rgI_shared > properties.sharedMemPerBlock) {
        printf("I: The size of the requested shared memory %iKb by recal_G is not available\n", rgI_shared/1024);
        return EXIT_FAILURE;
    } else {
        if (moreSharedMemThanBlocks) {
            while (rgI_shared*2  < properties.sharedMemPerBlock && mI/float(msI*2) == float(mI/(msI*2))) {
                msI = msI * 2;
                rgI_shared = rgI_shared * 2;
            }
        }
    }
    s_actVec_lI = msI*s_actVec_lI;
    rgI_b1.x = nI/s_actVec_lI;
    rgI_b1.y = networkSize/InTPB;
    printf("I: recal_G<<<(%i,%i,%i)x(%i,%i,%i), %iKb>>>, msI = %i\n", rgI_b1.x, rgI_b1.y, rgI_b1.z, rgI_b2.x, rgI_b2.y, rgI_b2.z, rgI_shared/1024, msI);

    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    int rE_b2, rI_b2;
    double *gE_b1x, *gI_b1x, *hE_b1x, *hI_b1x;
    if (rgE_b1.x >= 32) {
        int e = 5;
        while (rgE_b1.x > 1<<e) e++;
        rE_b2 = 1<<e;
        printf("blockdims for reduction of %i per thread : %i x %i \n", rgE_b1.x, networkSize, rE_b2);
        CUDA_CALL(hipMalloc((void **)&gE_b1x,  networkSize * rE_b2 * ngTypeE * sizeof(double)));
        CUDA_CALL(hipMalloc((void **)&hE_b1x,  networkSize * rE_b2 * ngTypeE * sizeof(double)));
    }
    if (rgI_b1.x >= 32) {
        int e = 5;
        while (rgI_b1.x > 1<<e) e++;
        rI_b2 = 1<<e;
        printf("blockdims for reduction of %i per thread : %i x %i \n", rgI_b1.x, networkSize, rI_b2);
        CUDA_CALL(hipMalloc((void **)&gI_b1x,  networkSize * rI_b2 * ngTypeI * sizeof(double)));
        CUDA_CALL(hipMalloc((void **)&hI_b1x,  networkSize * rI_b2 * ngTypeI * sizeof(double)));
    }




    /* Create CUDA events */
    hipEvent_t start, stop, gReady, spikeCorrected, initialSpikesObtained;
    hipEvent_t kStart, kStop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&gReady);
    hipEventCreate(&spikeCorrected);
    hipEventCreate(&initialSpikesObtained);
    hipEventCreate(&kStart);
    hipEventCreate(&kStop);
    /* Initialize device arrays */
    // CUDA streams for init
    hipStream_t i1, i2, i3;
    CUDA_CALL(hipStreamCreate(&i1));
    CUDA_CALL(hipStreamCreate(&i2));
    CUDA_CALL(hipStreamCreate(&i3));
    if (presetInit) {
    } else {
        for (unsigned int i=0; i<networkSize; i++) {
            v[i] = 0.0f;
            gE[i] = 0.0f;
            gI[i] = 0.0f;
            spikeTrain[i] = -1.0f;
        }
        // init rand generation for poisson
        logRand_init<<<init_b1,init_b2,0,i1>>>(lastNegLogRand, state, seed);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i2>>>(d_inputRate, flatRate/1000.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i3>>>(d_v, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i2>>>(leftTimeRate, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i3>>>(tBack, -1.0f); 
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i2>>>(d_fE, ffsE);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i3>>>(d_fI, ffsI);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i2>>>(d_gE, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i3>>>(d_gI, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i2>>>(d_hE, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i3>>>(d_hI, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngType,init_b2,0,i2>>>(gactVec, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngType,init_b2,0,i3>>>(hactVec, 0.0f);
        CUDA_CHECK();
        //CUDA_CALL(hipEventRecord(kStart, 0));
        printf("storage size of preMat %.1fMb\n", float(networkSize*networkSize*sizeof(double))/1024.0/1024.0);
        init<<<init_b1*init_b1*init_b2,init_b2,0,i2>>>(d_preMat, s);
        CUDA_CHECK();
        //CUDA_CALL(hipEventRecord(kStop, 0));
        //CUDA_CALL(hipEventSynchronize(kStop));
        //CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
        //printf("logRand_init<<<%ix%i>>> cost %.1fms\n", init_b1*init_b1*init_b2, init_b2, time);
    }
    CUDA_CALL(hipStreamDestroy(i1));
    CUDA_CALL(hipStreamDestroy(i2));
    CUDA_CALL(hipStreamDestroy(i3));
    CUDA_CALL(hipEventRecord(iStop, 0));
    CUDA_CALL(hipEventSynchronize(iStop));
    CUDA_CALL(hipEventElapsedTime(&time, iStart, iStop));
    printf("initialization cost %fms\n", time);

    /* Create CUDA streams */
    hipStream_t s1, s2, s3;
    CUDA_CALL(hipStreamCreate(&s1));
    CUDA_CALL(hipStreamCreate(&s2));
    CUDA_CALL(hipStreamCreate(&s3));
    unsigned int shared_mem = 0;
    v_file.open("v_ictorious.bin", std::ios::out|std::ios::binary);
    spike_file.open("s_uspicious.bin", std::ios::out|std::ios::binary);
    gE_file.open("gE_nerous.bin", std::ios::out|std::ios::binary);
    gI_file.open("gI_berish.bin", std::ios::out|std::ios::binary);
    CUDA_CALL(hipEventRecord(start, 0));
    double events = 0.0f;
    int spikes = 0;
    unsigned int ibatch = 0;
    unsigned int bStep = 0;
    unsigned int batchOffset = 0;
    unsigned int copySize = batchStep;
    unsigned int n = networkSize*copySize;
    
    //for (int ibatch=0; i<nbatch; ibatch++) {
    //    if(ibatch == nbatch-1) {
    //        copySize = batchEnd;
    //    }
        bool it = true;
        double time1 = 0.0f;
        double time2 = 0.0f;
        int iskip = 1;
        int nInput;
        for (int i=0; i<nstep; i++) {
            if (iskip % nskip == 0) {
                nInput = _nInput; 
            } else {
                nInput = 0;
            }
            unsigned int offset; 
            //offset = networkSize*(batchOffset + i);
            offset = 0;
            CUDA_CALL(hipStreamWaitEvent(s1, gReady, 0));
            /* Compute voltage (acquire initial spikes) */
            CUDA_CALL(hipEventRecord(kStart, 0));
            compute_V<<<b1, b2, shared_mem, s1>>>(d_v, d_gE, d_gI, d_hE, d_hI, d_a, d_b, d_preMat, d_inputRate, d_eventRate, d_spikeTrain, tBack, gactVec, hactVec, d_fE, d_fI, leftTimeRate, lastNegLogRand, state, ngTypeE, ngTypeI, ngType, condE, condI, dt, networkSize, nE, seed, nInput, it);
            CUDA_CHECK();
            CUDA_CALL(hipEventRecord(kStop, 0));
            CUDA_CALL(hipEventSynchronize(kStop));
            CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
            if (printStep) {
                printf("A single step of compute_V cost %fms\n", time);
            }
            time1 += time;
            it = false;
            //CUDA_CALL(hipEventRecord(initialSpikesObtained, s1));
            /* Spike correction */
            CUDA_CALL(hipEventRecord(spikeCorrected, s1));
            /* Write voltage of last step to disk */
            v_file.write((char*)&(v[n*batchOffset]),               n*sizeof(double));
            /* Write spikeTrain of last step to disk */
            spike_file.write((char*)&(spikeTrain[n*batchOffset]),  n*sizeof(double));
            /* Copy voltage to host */
            CUDA_CALL(hipMemcpyAsync(&(v[offset]), d_v, networkSize * sizeof(double), hipMemcpyDeviceToHost, s1));
            CUDA_CALL(hipMemcpyAsync(eventRate, d_eventRate, networkSize * sizeof(int), hipMemcpyDeviceToHost, s1));
            /* Copy spikeTrain to host */
            CUDA_CALL(hipMemcpyAsync(&(spikeTrain[offset]), d_spikeTrain, networkSize * sizeof(double), hipMemcpyDeviceToHost, s1));

            CUDA_CALL(hipStreamWaitEvent(s2, spikeCorrected, 0));
            /* Recalibrate conductance */
            // recal E
            CUDA_CALL(hipEventRecord(kStart, 0));
            recal_G<<<rgE_b1,rgE_b2,rgE_shared,s2>>>(d_gE, d_hE, d_preMat,
                                                     gactVec, hactVec,
                                                     gE_b1x, hE_b1x,
                                                     networkSize, 0, ngTypeE, s_actVec_lE, msE);
            CUDA_CHECK();
            // recal I
            CUDA_CALL(hipStreamWaitEvent(s3, spikeCorrected, 0));
            recal_G<<<rgI_b1,rgI_b2,rgI_shared,s3>>>(d_gI, d_hI, d_preMat,
                                                     gactVec, hactVec,
                                                     gI_b1x, hI_b1x,
                                                     networkSize, nE, ngTypeI, s_actVec_lI, msI);
            CUDA_CHECK();
            if (rgE_b1.x >= 32) {
                //  reduce sum
                reduce_G<<<networkSize, rE_b2, sizeof(double)*2*rE_b2, s2>>>(d_gE, d_hE, gE_b1x, hE_b1x, ngTypeE, rgE_b1.x);
                CUDA_CHECK();
            }
            if (rgI_b1.x >= 32) {
                reduce_G<<<networkSize, rI_b2, sizeof(double)*2*rI_b2, s3>>>(d_gI, d_hI, gI_b1x, hI_b1x, ngTypeI, rgI_b1.x);
                CUDA_CHECK();
            }
            CUDA_CALL(hipEventRecord(kStop, 0));
            CUDA_CALL(hipEventSynchronize(kStop));
            CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
            if (printStep) {
                printf("A single step of recal_G cost %fms\n", time);
            }
            time2 += time;
            /* Write conductance of last step to disk */
            gE_file.write((char*)&(gE[n*ngTypeE*batchOffset]),     n*ngTypeE*sizeof(double));
            gI_file.write((char*)&(gI[n*ngTypeI*batchOffset]),     n*ngTypeI*sizeof(double));
            /* Copy conductance to host */
            CUDA_CALL(hipMemcpyAsync(&(gE[offset*ngTypeE]), d_gE, networkSize * ngTypeE * sizeof(double), hipMemcpyDeviceToHost, s2));
            CUDA_CALL(hipMemcpyAsync(&(gI[offset*ngTypeI]), d_gI, networkSize * ngTypeI * sizeof(double), hipMemcpyDeviceToHost, s2));
            CUDA_CALL(hipEventRecord(gReady, s2));
            //printf("\r total: %3.1f, batch: %3.1f", 100.0f*float(ibatch+1)/nbatch, float(i)/copySize);
            printf("\r stepping: %3.1f%%", 100.0f*float(i+1)/nstep);
            fflush(stdout);
            double _events = 0.0f;
            int _spikes = 0;
            for (int j=0; j<networkSize; j++) {
                _events += eventRate[j];
                if (spikeTrain[j] > 0.0f) {
                    _spikes++;
                }
            }
            events += _events;
            spikes += _spikes;
            if (printStep) {
                printf("instant input rate = %fkHz, dt = %f, networkSize = %i\n", _events/(dt*networkSize), dt, networkSize);
                printf("instant firing rate = %fHz\n", _spikes/(dt*networkSize)*1000.0);
            }
            if (batchOffset == 0) {
                batchOffset = 0;
                //batchOffset = batchStep;
            } else {
                batchOffset = 0;
            }
            iskip++;
        }
        /* WHen hit HALF_MEMORY_OCCUPANCY, write half of the array to disk, the other half left to receive from device */
        // Alternating
        // switch batchOffset
    //}

    v_file.write((char*)v, networkSize * sizeof(double));
    //spike_file.write((char*)spikeTrain, networkSize * sizeof(int));
    gE_file.write((char*)gE, networkSize * ngTypeE * sizeof(double));
    gI_file.write((char*)gI, networkSize * ngTypeI * sizeof(double));
    printf("\n");

    printf("flatRate = %fkHz, realized mean input rate = %fkHz\n", flatRate/1000.0, float(events)/(dt*nstep*networkSize));
    printf("mean firing rate = %fHz\n", float(spikes)/(dt*nstep*networkSize)*1000.0);

    CUDA_CALL(hipEventRecord(stop, 0));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipEventElapsedTime(&time, start, stop));
    printf("CUDA takes %fms, runtime/realtime ratio ms %fms\n", time, time/(dt*nstep));
    printf("compute_V takes %fms, ratio ms %fms\n", time1, time1/(dt*nstep));
    printf("recal_G takes %fms, ratio ms %fms\n", time2, time2/(dt*nstep));

    /* Cleanup */
    printf("Cleaning up:\n");
    CUDA_CALL(hipStreamDestroy(s1));
    CUDA_CALL(hipStreamDestroy(s2));
    CUDA_CALL(hipStreamDestroy(s3));
    printf("    CUDA streams destroyed\n");
    if (v_file.is_open()) v_file.close();
    if (spike_file.is_open()) spike_file.close();
    if (gE_file.is_open()) gE_file.close();
    if (gI_file.is_open()) gI_file.close();
    printf("    Output files closed\n");
    
    CUDA_CALL(hipFree(state));
    CUDA_CALL(hipFree(d_v));
    CUDA_CALL(hipFree(d_gE));
    CUDA_CALL(hipFree(d_gI));
    CUDA_CALL(hipFree(d_hE));
    CUDA_CALL(hipFree(d_hI));
    CUDA_CALL(hipFree(d_fE));
    CUDA_CALL(hipFree(d_fI));
    CUDA_CALL(hipFree(gactVec));
    CUDA_CALL(hipFree(hactVec));
    CUDA_CALL(hipFree(d_preMat));
    CUDA_CALL(hipFree(d_a));
    CUDA_CALL(hipFree(d_b));
    if (rgE_b1.x >= 32) {
        CUDA_CALL(hipFree(gE_b1x));
        CUDA_CALL(hipFree(hE_b1x));
    }
    if (rgI_b1.x >= 32) {
        CUDA_CALL(hipFree(gI_b1x));
        CUDA_CALL(hipFree(hI_b1x));
    }
    CUDA_CALL(hipFree(leftTimeRate));
    CUDA_CALL(hipFree(lastNegLogRand));
    CUDA_CALL(hipFree(d_inputRate));
    CUDA_CALL(hipFree(d_eventRate));
    CUDA_CALL(hipFree(d_spikeTrain));
    CUDA_CALL(hipFree(tBack));
    printf("    Device memory freed\n");
    CUDA_CALL(hipHostFree(v));
    CUDA_CALL(hipHostFree(gE));
    CUDA_CALL(hipHostFree(gI));
    CUDA_CALL(hipHostFree(eventRate));
    CUDA_CALL(hipHostFree(spikeTrain));
    free(preMat);
    printf("    Host memory freed\n");
    return EXIT_SUCCESS;
}
