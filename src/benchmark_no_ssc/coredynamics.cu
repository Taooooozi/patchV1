#include "hip/hip_runtime.h"
#include "coredynamics.h"

__global__ void recal_G(double* __restrict__ g,
                        double* __restrict__ h,
                        double* __restrict__ preMat,
                        double* __restrict__ gactVec,
                        double* __restrict__ hactVec,
                        double* __restrict__ g_b1y,
                        double* __restrict__ h_b1y,
                        unsigned int n, unsigned int offset, unsigned int ngType, unsigned int ns, int m) 
{
    // 2D blockGrid
    // -> D-1 pieces of actVec 
    // -> D-2 pieces of post-synaptic neurons 
    // 1D threadBlock
    extern __shared__ double actVec[];
    double *gaV = actVec;
    double *haV = &(actVec[ngType*ns]);
    unsigned int id = blockDim.x*blockIdx.y + threadIdx.x;
    #pragma unroll
    for (int ig=0; ig<ngType; ig++) {
        #pragma unroll
        for (int i=0; i<m; i++) {
            // av = double[ngType,#(ns),ns]
            // actVec = double[ngType,n]
            if (threadIdx.x < ns) {
                unsigned int sid = ig*ns + (i*blockDim.x + threadIdx.x);
                unsigned int gid = (ig*n + offset + ns*blockIdx.x) + (i*blockDim.x + threadIdx.x);
                gaV[sid] = gactVec[gid];
                haV[sid] = hactVec[gid];
            }
        }
    }
    __syncthreads();
    for (int ig=0; ig<ngType; ig++) {
        double g_t = 0.0f;
        double h_t = 0.0f;
        for (int i = 0; i<ns; i++) {
            unsigned sid = ig*ns + i;
            unsigned pid = (offset + blockIdx.x*ns + i)*n + id;
            g_t += gaV[sid] * preMat[pid];
            h_t += haV[sid] * preMat[pid];
        }
        if (gridDim.x < 32) {
            unsigned int gid = ig*n + id;
            atomicAdd(&(g[gid]), g_t);
            atomicAdd(&(h[gid]), h_t);
        } else {
            // b1y = double[ngType, m, n]
            unsigned int b1yid = ig*n*gridDim.x + n*blockIdx.x + id;
            g_b1y[b1yid] = g_t;
            h_b1y[b1yid] = h_t;
        }
    }
}

__global__ void reduce_G(double* __restrict__ g,
                         double* __restrict__ h,
                         double* __restrict__ g_b1y, 
                         double* __restrict__ h_b1y,
                         unsigned int ngType, int n) 
{ 
    // n x #(ns)
    extern __shared__ double blk[];
    double* g_blk = blk;
    double* h_blk = &(blk[blockDim.x]);
    for (int ig=0; ig<ngType; ig++) {
        unsigned int gid = ig*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
        if (gid < n) {
            g_blk[threadIdx.x] = g_b1y[gid];
            h_blk[threadIdx.x] = g_b1y[gid];
        } else {
            g_blk[threadIdx.x] = 0.0f;
            h_blk[threadIdx.x] = 0.0f;
        }
        for (int i=blockDim.x/2; i>=32; i>>=1) {
             if (threadIdx.x < i) {
                 g_blk[threadIdx.x] += g_blk[threadIdx.x + i];
                 h_blk[threadIdx.x] += h_blk[threadIdx.x + i];
             }
             __syncthreads();
         }

        double g_warp = g_blk[threadIdx.x];
        double h_warp = h_blk[threadIdx.x];
        for (int offset = 16; offset > 0; offset /= 2) {
            g_warp += __shfl_down_sync(FULL_MASK, g_warp, offset);  
            h_warp += __shfl_down_sync(FULL_MASK, h_warp, offset);  
        }
        if (threadIdx.x == 0) {
            unsigned int id = ig*gridDim.x + blockIdx.x;
            g[id] = g_warp;
            h[id] = g_warp;
        }
    }
}

__global__ void logRand_init(double *logRand, hiprandStateMRG32k3a *state, unsigned long long seed) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    logRand[id] = -log(hiprand_uniform_double(&localState));
    //printf("logRand0 = %f\n", logRand[id]);
    //logRand[id] = 1.0f;
    state[id] = localState;
}

__device__ int set_input_time(double inputTime[],
                              double dt,
                              double rate,
                              double *leftTimeRate,
                              double *lastNegLogRand,
                              hiprandStateMRG32k3a* __restrict__ state)
{
    int i = 0;
    double tau, dTau, negLogRand;
    tau = (*lastNegLogRand - (*leftTimeRate))/rate;
    if (tau > dt) {
        *leftTimeRate += (dt * rate);
        return i;
    } else do {
        inputTime[i] = tau;
        negLogRand = -log(hiprand_uniform_double(state));
        dTau = negLogRand/rate;
        tau += dTau;
        i++;
        if (i == MAX_FFINPUT_PER_DT) {
            printf("exceeding max input per dt %i\n", MAX_FFINPUT_PER_DT);
            //printf("rate = %f, lastNegLogRand = %f, leftTimeRate = %f \n", rate, *lastNegLogRand, *leftTimeRate);
            //printf("inputTime[0]: %f, inputTime[1]: %f\n", inputTime[0], inputTime[1]);
            break;
        }
    } while (tau <= dt);
    *lastNegLogRand = negLogRand;
    *leftTimeRate = (dt - tau + dTau) * rate;
    return i;
}

__host__ __device__ void evolve_g(ConductanceShape &cond,
                                  double* __restrict__ g, 
                                  double* __restrict__ h, 
                                  double* __restrict__ f,
                                  double inputTime[],
                                  unsigned int nInput, double dt, unsigned int ig)
{
    cond.decay_conductance(g, h, dt, ig); 
    for (int i=0; i<nInput; i++) {
        cond.compute_single_input_conductance(g, h, *f, dt-inputTime[i], ig);
    }
}

__device__  double step(Func_RK2* lif, double dt, double tRef, unsigned int id, double gE_t) {
    lif->tsp = -1.0f;
    // not in refractory period
    if (lif->tBack < dt) {
        // return from refractory period
        if (lif->tBack > 0.0f) {
            lif->compute_pseudo_v0(dt);
            lif->tBack = -1.0f;
        }
        __syncthreads();
        lif->runge_kutta_2(dt);
        while (lif->v > vT && lif->tBack < 0.0f) {
            // crossed threshold

            if (lif->v > vE) {
                printf("#%i exc conductance is too high %f\n", id, gE_t);
            }
            
            lif->tsp = lif->compute_spike_time(dt); 
            lif->tBack = lif->tsp + tRef;
            //printf("neuron #%i fired initially\n", id);
            //assert(lif->tBack > 0);
            if (lif->tBack < dt) {
                // refractory period ended during dt
                lif->compute_v(dt);
                lif->tBack = -1.0f;
                if (lif->v > vT) {
                    printf("multiple spike in one time step, only the last spike is counted, refractory period = %f ms, dt = %f\n", tRef, dt);
                    //assert(lif->v <= vT);
                }
            }
        }
    } 
    if (lif->tBack >= dt) {
        // during refractory period
        lif->reset_v(); 
        lif->tBack -= dt;
    }
    return lif->tsp;
}

__global__ void compute_V(double* __restrict__ v,
                          double* __restrict__ gE,
                          double* __restrict__ gI,
                          double* __restrict__ hE,
                          double* __restrict__ hI,
                          double* __restrict__ a,
                          double* __restrict__ b,
                          double* __restrict__ preMat,
                          double* __restrict__ inputRate,
                          int* __restrict__ eventRate,
                          double* __restrict__ spikeTrain,
                          double* __restrict__ tBack,
                          double* __restrict__ gactVec,
                          double* __restrict__ hactVec,
                          double* __restrict__ fE,
                          double* __restrict__ fI,
                          double* __restrict__ leftTimeRate,
                          double* __restrict__ lastNegLogRand,
                          hiprandStateMRG32k3a* __restrict__ state,
                          unsigned int ngTypeE, unsigned int ngTypeI, unsigned int ngType, ConductanceShape condE, ConductanceShape condI, double dt, unsigned int networkSize, unsigned int nE, unsigned long long seed, int nInput, bool it)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    double gL, tRef;
    if (id < nE) {
        tRef = tRef_E;
        gL = gL_E;
    } else {
        tRef = tRef_I;
        gL = gL_I;
    }
    LIF lif(v[id], tBack[id]);
    /* set a0 b0 for the first step */
    double gI_t;
    double gE_t;
    // init cond E 
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gE_t += gE[networkSize*ig + id];
    }
    //  cond I 
    gI_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gI_t += gI[networkSize*ig + id];
    }
    lif.set_p0(gE_t, gI_t, gL);

    /* Get feedforward input */
    // consider use shared memory for dynamic allocation
    double inputTime[MAX_FFINPUT_PER_DT];
    hiprandStateMRG32k3a localState = state[id];
    #ifdef TEST_WITH_MANUAL_FFINPUT
        #pragma unroll
        for (int iInput = 0; iInput < nInput; iInput++) {
            inputTime[iInput] = (iInput + double(id)/networkSize)*dt/nInput;
        }
        // not used if not RAND
        lastNegLogRand[id] = 1.0f;
        leftTimeRate[id] = 0.0f;
    #else
        nInput = set_input_time(inputTime, dt, inputRate[id], &(leftTimeRate[id]), &(lastNegLogRand[id]), &localState);
    #endif
    //__syncwarp();
    //if (it) {
    //    printf("nInput = %i\n", nInput);
    //}
    //}
    // return a realization of Poisson input rate
    eventRate[id] = nInput;
    // update rng state 
    state[id] = localState;
    /* evolve g to t+dt with ff input only */
    unsigned int gid;
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gid = networkSize*ig + id;
        double g_i = gE[gid];
        double h_i = hE[gid];
        double f_i = fE[gid];
        evolve_g(condE, &g_i, &h_i, &f_i, inputTime, nInput, dt, ig);
        //__syncwarp();
        gE_t += g_i;
        gE[gid] = g_i;
        hE[gid] = h_i;
        // for learning
        //fE[gid] = f_i;
    }
    //printf("id %i, exc cond ready.\n",id);
    gI_t = 0.0f;
    /* no feed-forward inhibitory input (setting nInput = 0) */
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gid = networkSize*ig + id;
        double g_i = gI[gid];
        double h_i = hI[gid];
        double f_i = fI[gid];
        evolve_g(condI, &g_i, &h_i, &f_i, inputTime, 0, dt, ig);
        //__syncwarp();
        gI_t += g_i;
        gI[gid] = g_i;
        hI[gid] = h_i;
        // for learning
        //fI[gid] = f_i;
    }
    lif.set_p1(gE_t, gI_t, gL);
    // rk2 step

    spikeTrain[id] = step(&lif, dt, tRef, /*the last 2 args are for deugging*/ id, gE_t);
	v[id] = lif.v;
    tBack[id] = lif.tBack;

    //setup acting vectors
    double g_end, h_end;
    if (spikeTrain[id]>0.0f) {
        if (id < nE) {
            #pragma unroll
            for (int ig=0; ig<ngTypeE; ig++) {
                g_end = 0.0f;
                h_end = 0.0f;
                condE.compute_single_input_conductance(&g_end, &h_end, 1.0f, dt-lif.tsp, ig);
                gid = networkSize*ig+id;
                gactVec[gid] = g_end;
                hactVec[gid] = h_end;
            }
        } else {
            #pragma unroll
            for (int ig=0; ig<ngTypeI; ig++) {
                g_end = 0.0f;
                h_end = 0.0f;
                condI.compute_single_input_conductance(&g_end, &h_end, 1.0f, dt-lif.tsp, ig);
                gid = networkSize*ig+id;
                gactVec[gid] = g_end;
                hactVec[gid] = h_end;
            }
        }
    } else {
        for (int ig=0; ig<ngType; ig++) {
            gid = networkSize*ig+id;
            gactVec[gid] = 0.0f;
            hactVec[gid] = 0.0f;
        }
    }
}

__device__ void Func_RK2::runge_kutta_2(double dt) {
    double fk0 = eval0(v0);
    double fk1 = eval1(v0 + dt*fk0);
    v = v0 + dt*(fk0+fk1)/2.0f;
}

__device__ double LIF::compute_spike_time(double dt) {
    return (vT-v0)/(v-v0)*dt;
}

__device__ void LIF::compute_v(double dt) {
    v = compute_v1(dt, a0, b0, a1, b1, vL, tBack);
}

__device__ void LIF:: compute_pseudo_v0(double dt) {
    v0 = (vL-tBack*(b0 + b1 - a1*b0*dt)/2.0f)/(1.0f+tBack*(-a0 - a1 + a1*a0*dt)/2.0f);
    runge_kutta_2(dt);
}

__device__ void LIF::set_p0(double gE, double gI, double gL) {
    a0 = get_a(gE, gI, gL);
    b0 = get_b(gE, gI, gL); 
}

__device__ void LIF::set_p1(double gE, double gI, double gL) {
    a1 = get_a(gE, gI, gL);
    b1 = get_b(gE, gI, gL); 
}

__device__ double LIF::eval0(double _v) {
    return eval_LIF(a0,b0,_v);
}

__device__ double LIF::eval1(double _v) {
    return eval_LIF(a1,b1,_v);
}

__device__ void LIF::reset_v() {
    v = vL;
}
