#include "hip/hip_runtime.h"
#include "coredynamics.h"

__global__ void recal_G(double* __restrict__ g,
                        double* __restrict__ h,
                        double* __restrict__ preMat,
                        double* __restrict__ gactVec,
                        double* __restrict__ hactVec,
                        double* __restrict__ g_b1x,
                        double* __restrict__ h_b1x,
                        unsigned int n, unsigned int offset, unsigned int ngType, unsigned int ns, int m) 
{
    // 2D blockGrid
    // -> D-1 pieces of actVec 
    // -> D-2 pieces of post-synaptic neurons 
    // 1D threadBlock
    extern __shared__ double actVec[];
    double *gaV = actVec;
    double *haV = &(actVec[ngType*ns]);
    unsigned int id = blockDim.x*blockIdx.y + threadIdx.x;
    unsigned int ss = ns/m;
    #pragma unroll
    for (int ig=0; ig<ngType; ig++) {
        #pragma unroll
        for (int i=0; i<m; i++) {
            // av = double[ngType,#(ns),ns]
            // actVec = double[ngType,n]
            if (threadIdx.x < ss) {
                unsigned int sid = ig*ns + (i*ss + threadIdx.x);
                unsigned int gid = (ig*n + offset + ns*blockIdx.x) + (i*ss + threadIdx.x);
                gaV[sid] = gactVec[gid];
                haV[sid] = hactVec[gid];
            }
        }
    }
    __syncthreads();
    for (int ig=0; ig<ngType; ig++) {
        double g_t = 0.0f;
        double h_t = 0.0f;
        for (int i = 0; i<ns; i++) {
            unsigned sid = ig*ns + i;
            if (gaV[sid] > 0) {
                unsigned pid = (offset + blockIdx.x*ns + i)*n + id;
                double s = preMat[pid];
                g_t += gaV[sid] * s;
                h_t += haV[sid] * s;
            }
        }
        if (gridDim.x < 32) {
            if (g_t > 0) {
                unsigned int gid = ig*n + id;
                atomicAdd(&(g[gid]), g_t);
                atomicAdd(&(h[gid]), h_t);
            }
        } else {
            // b1x = double[ngType, n/ns(gridDim.x), n]
            unsigned int b1xid = ig*n*gridDim.x + blockIdx.x*n + id;
            g_b1x[b1xid] = g_t;
            h_b1x[b1xid] = h_t;
        }
    }
}

__global__ void reduce_G(double* __restrict__ g,
                         double* __restrict__ h,
                         double* __restrict__ g_b1x, 
                         double* __restrict__ h_b1x,
                         unsigned int ngType, int n) 
{ 
    // b1x = double[ngType, n/ns(gridDim.x), n]
    // n x #(ns)
    extern __shared__ double blk[];
    double* g_blk = blk;
    double* h_blk = &(blk[blockDim.x]);
    for (int ig=0; ig<ngType; ig++) {
        unsigned int gid = ig*blockDim.x*gridDim.x + threadIdx.x*gridDim.x + blockIdx.x;
        if (gid < n) {
            // can do coalesce read optimization here (transpose in shared mem)
            g_blk[threadIdx.x] = g_b1x[gid];
            h_blk[threadIdx.x] = g_b1x[gid];
        } else {
            g_blk[threadIdx.x] = 0.0f;
            h_blk[threadIdx.x] = 0.0f;
        }
        __syncthreads();
        for (int i=blockDim.x/2; i>=32; i>>=1) {
            if (threadIdx.x < i) {
                g_blk[threadIdx.x] += g_blk[threadIdx.x + i];
                h_blk[threadIdx.x] += h_blk[threadIdx.x + i];
            }
            __syncthreads();
        }
        if (threadIdx.x < 32) {
            double g_warp = g_blk[threadIdx.x];
            double h_warp = h_blk[threadIdx.x];
            for (int offset = 16; offset > 0; offset /= 2) {
                g_warp += __shfl_down_sync(FULL_MASK, g_warp, offset);  
                h_warp += __shfl_down_sync(FULL_MASK, h_warp, offset);  
            }
            if (threadIdx.x == 0) {
                unsigned int id = ig*gridDim.x + blockIdx.x;
                g[id] += g_warp;
                h[id] += g_warp;
            }
        }
    }
}

__global__ void logRand_init(double *logRand, hiprandStateMRG32k3a *state, unsigned long long seed) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    logRand[id] = -log(hiprand_uniform_double(&localState));
    //printf("logRand0 = %f\n", logRand[id]);
    //logRand[id] = 1.0f;
    state[id] = localState;
}

__global__ void randInit(double* __restrict__ preMat, 
						 double* __restrict__ v, 
						 double* __restrict__ lTR, 
						 hiprandStateMRG32k3a* __restrict__ state,
double sE, double sI, unsigned int networkSize, unsigned int nE, unsigned long long seed, double dInput) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    v[id] = vL + hiprand_uniform_double(&localState) * (vT-vL);
    double mean = log(sI/sqrt(1.0f+1.0f/sI));
    double std = sqrt(log(1.0f+1.0f/sI));
    for (unsigned int i=0; i<nE; i++) {
        preMat[i*networkSize + id] = hiprand_log_normal_double(&localState, mean, std);
        // lTR works as firstInputTime
        #ifdef TEST_WITH_MANUAL_FFINPUT
            lTR[id] = hiprand_uniform_double(&localState)*dInput;
        #endif
    }
    mean = log(sI/sqrt(1.0f+1.0f/sI));
    std = sqrt(log(1.0f+1.0f/sI));
    for (unsigned int i=nE; i<networkSize; i++) {
        preMat[i*networkSize + id] = hiprand_log_normal_double(&localState, mean, std);

        #ifdef TEST_WITH_MANUAL_FFINPUT
            lTR[id] = hiprand_uniform_double(&localState)*dInput;
        #endif
    }
}

__device__ int set_input_time(double inputTime[],
                              double dt,
                              double rate,
                              double *leftTimeRate,
                              double *lastNegLogRand,
                              hiprandStateMRG32k3a* __restrict__ state)
{
    int i = 0;
    double tau, dTau, negLogRand;
    tau = (*lastNegLogRand - (*leftTimeRate))/rate;
    if (tau > dt) {
        *leftTimeRate += (dt * rate);
        return i;
    } else do {
        inputTime[i] = tau;
        negLogRand = -log(hiprand_uniform_double(state));
        dTau = negLogRand/rate;
        tau += dTau;
        i++;
        if (i == MAX_FFINPUT_PER_DT) {
            printf("exceeding max input per dt %i\n", MAX_FFINPUT_PER_DT);
            //printf("rate = %f, lastNegLogRand = %f, leftTimeRate = %f \n", rate, *lastNegLogRand, *leftTimeRate);
            //printf("inputTime[0]: %f, inputTime[1]: %f\n", inputTime[0], inputTime[1]);
            break;
        }
    } while (tau <= dt);
    *lastNegLogRand = negLogRand;
    *leftTimeRate = (dt - tau + dTau) * rate;
    return i;
}

__host__ __device__ void evolve_g(ConductanceShape &cond,
                                  double* __restrict__ g, 
                                  double* __restrict__ h, 
                                  double* __restrict__ f,
                                  double inputTime[],
                                  int nInput, double dt, unsigned int ig)
{
    cond.decay_conductance(g, h, dt, ig); 
    for (int i=0; i<nInput; i++) {
        cond.compute_single_input_conductance(g, h, *f, dt-inputTime[i], ig);
    }
}

__device__  double step(Func_RK2* lif, double dt, double tRef, unsigned int id, double gE, double gI, double tsp[]) {
    lif->tsp = dt;
    lif->spikeCount = 0;
    // not in refractory period
    if (lif->tBack < dt) {
        // return from refractory period
        if (lif->tBack > 0.0f) {
            lif->compute_pseudo_v0(dt);
            lif->tBack = -1.0f;
        }
        __syncthreads();
        lif->runge_kutta_2(dt);
        while (lif->v > vT && lif->tBack < 0.0f) {
            // crossed threshold
            lif->tsp = lif->compute_spike_time(dt); 
            tsp[lif->spikeCount] = lif->tsp;
            lif->spikeCount++;
            lif->tBack = lif->tsp + tRef;
            if (lif->tBack < dt) {
                // refractory period ended during dt
                lif->compute_pseudo_v0(dt);
                lif->runge_kutta_2(dt);
                lif->tBack = -1.0f;
            }
        }
    } 
    if (lif->tBack >= dt) {
        // during refractory period
        lif->reset_v(); 
        lif->tBack -= dt;
    }
    if (lif->spikeCount > 1) {
#ifdef DEBUG
        printf("#%i spiked %i in one time step %f, refractory period = %f ms, only the last tsp is recorded\n", id, lif->spikeCount, dt, tRef);
#endif
    }
    return lif->tsp;
}

__device__ void Func_RK2::runge_kutta_2(double dt) {
    double fk0 = eval0(v0);
    double fk1 = eval1(v0 + dt*fk0);
    v = v0 + dt*(fk0+fk1)/2.0f;
}

__device__ double LIF::compute_spike_time(double dt) {
    return (vT-v0)/(v-v0)*dt;
}

__device__ void LIF::compute_v(double dt) {
    v = compute_v1(dt, a0, b0, a1, b1, vL, tBack);
}

__device__ void LIF::compute_pseudo_v0(double dt) {
    v0 = (vL-tBack*(b0 + b1 - a1*b0*dt)/2.0f)/(1.0f+tBack*(-a0 - a1 + a1*a0*dt)/2.0f);
}

__device__ void LIF::set_p0(double gE, double gI, double gL) {
    a0 = get_a(gE, gI, gL);
    b0 = get_b(gE, gI, gL); 
}

__device__ void LIF::set_p1(double gE, double gI, double gL) {
    a1 = get_a(gE, gI, gL);
    b1 = get_b(gE, gI, gL); 
}

__device__ double LIF::eval0(double _v) {
    return eval_LIF(a0,b0,_v);
}

__device__ double LIF::eval1(double _v) {
    return eval_LIF(a1,b1,_v);
}

__device__ void LIF::reset_v() {
    v = vL;
}

__global__ void compute_V(double* __restrict__ v,
                          double* __restrict__ gE,
                          double* __restrict__ gI,
                          double* __restrict__ hE,
                          double* __restrict__ hI,
                          double* __restrict__ a,
                          double* __restrict__ b,
                          double* __restrict__ preMat,
                          double* __restrict__ inputRate,
                          int* __restrict__ eventRate,
                          double* __restrict__ spikeTrain,
                          unsigned int* __restrict__ nSpike,
                          double* __restrict__ tBack,
                          double* __restrict__ gactVec,
                          double* __restrict__ hactVec,
                          double* __restrict__ fE,
                          double* __restrict__ fI,
                          double* __restrict__ leftTimeRate,
                          double* __restrict__ lastNegLogRand,
                          hiprandStateMRG32k3a* __restrict__ state,
                          unsigned int ngTypeE, unsigned int ngTypeI, unsigned int ngType, ConductanceShape condE, ConductanceShape condI, double dt, unsigned int networkSize, unsigned int nE, unsigned long long seed, double dInput)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    LIF lif(v[id], tBack[id]);
    double gL, tRef;
    if (id < nE) {
        tRef = tRef_E;
        gL = gL_E;
    } else {
        tRef = tRef_I;
        gL = gL_I;
    }
    /* set a0 b0 for the first step */
    double gI_t;
    double gE_t;
    // init cond E 
    gE_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeE; ig++) {
        gE_t += gE[networkSize*ig + id];
    }
    //  cond I 
    gI_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeI; ig++) {
        gI_t += gI[networkSize*ig + id];
    }
    lif.set_p0(gE_t, gI_t, gL);
    /* Get feedforward input */
    // consider use shared memory for dynamic allocation
    double inputTime[MAX_FFINPUT_PER_DT];
    hiprandStateMRG32k3a localState = state[id];
    int nInput;
    #ifdef TEST_WITH_MANUAL_FFINPUT
        nInput = 0;
        if (leftTimeRate[id] < dt) {
            inputTime[nInput] = leftTimeRate[id];
            nInput++;
            double tmp = leftTimeRate[id] + dInput;
            while (tmp < dt){
                inputTime[nInput] = tmp;
                nInput++;
                tmp += dInput;
            }
            leftTimeRate[id] = tmp - dt;
        } else {
            leftTimeRate[id] -= dt;
        }
    #else
        nInput = set_input_time(inputTime, dt, inputRate[id], &(leftTimeRate[id]), &(lastNegLogRand[id]), &localState);
    #endif
    //__syncwarp();
    // return a realization of Poisson input rate
    eventRate[id] = nInput;
    // update rng state 
    state[id] = localState;
    /* evolve g to t+dt with ff input only */
    unsigned int gid;
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gid = networkSize*ig + id;
        double g_i = gE[gid];
        double h_i = hE[gid];
        double f_i = fE[gid];
        evolve_g(condE, &g_i, &h_i, &f_i, inputTime, nInput, dt, ig);
        //__syncwarp();
        gE_t += g_i;
        gE[gid] = g_i;
        hE[gid] = h_i;
        // for learning
        //fE[gid] = f_i;
    }
    //printf("id %i, exc cond ready.\n",id);
    gI_t = 0.0f;
    /* no feed-forward inhibitory input (setting nInput = 0) */
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gid = networkSize*ig + id;
        double g_i = gI[gid];
        double h_i = hI[gid];
        double f_i = fI[gid];
        evolve_g(condI, &g_i, &h_i, &f_i, inputTime, 0, dt, ig);
        //__syncwarp();
        gI_t += g_i;
        gI[gid] = g_i;
        hI[gid] = h_i;
        // for learning
        //fI[gid] = f_i;
    }
    lif.set_p1(gE_t, gI_t, gL);
    // rk2 step
    double tsp[MAX_SPIKE_PER_DT];
    spikeTrain[id] = step(&lif, dt, tRef, /*the last 2 args are for deugging*/ id, gE_t, gI_t, tsp);
    nSpike[id] = lif.spikeCount;
    if (lif.v < vI) {
#ifdef DEBUG
		printf("#%i something is off gE = %f, gI = %f, v = %f\n", id, gE_t, gI_t, lif.v);
#endif
        lif.v = vI;
    }   
	v[id] = lif.v;
    tBack[id] = lif.tBack;

    //setup acting vectors
    double g_end, h_end;
    if (lif.spikeCount > 0) {
        int ngType;
        ConductanceShape *cond; 
        if (id < nE) {
            ngType = ngTypeE;
            cond = &condE;
        } else {
            ngType = ngTypeI;
            cond = &condI;
        }
        #pragma unroll
        for (int ig=0; ig<ngType; ig++) {
            gid = networkSize*ig+id;
            gactVec[gid] = 0.0f;
            hactVec[gid] = 0.0f;
            for (int i=0; i<lif.spikeCount; i++) {
                g_end = 0.0f;
                h_end = 0.0f;
                cond->compute_single_input_conductance(&g_end, &h_end, 1.0f, dt-tsp[i], ig);
                gactVec[gid] += g_end;
                hactVec[gid] += h_end;
            }
        }
    } else {
        for (int ig=0; ig<ngType; ig++) {
            gid = networkSize*ig + id;
            gactVec[gid] = 0.0f;
            hactVec[gid] = 0.0f;
        }
    }
    delete []tsp;
}
