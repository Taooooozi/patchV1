#include "hip/hip_runtime.h"
#include "coredynamics.h"

__global__ void recal_G(double* __restrict__ g,
                        double* __restrict__ h,
                        double* __restrict__ preMat,
                        double* __restrict__ gactVec,
                        double* __restrict__ hactVec,
                        double* __restrict__ g_b1x,
                        double* __restrict__ h_b1x,
                        unsigned int n, unsigned int offset, unsigned int ngType, unsigned int ns, int m) 
{
    // 2D blockGrid
    // -> D-1 pieces of actVec 
    // -> D-2 pieces of post-synaptic neurons 
    // 1D threadBlock
    extern __shared__ double actVec[];
    double *gaV = actVec;
    double *haV = &(actVec[ngType*ns]);
    unsigned int id = blockDim.x*blockIdx.y + threadIdx.x;
    unsigned int ss = ns/m;
    #pragma unroll
    for (int ig=0; ig<ngType; ig++) {
        #pragma unroll
        for (int i=0; i<m; i++) {
            // av = double[ngType,#(ns),ns]
            // actVec = double[ngType,n]
            if (threadIdx.x < ss) {
                unsigned int sid = ig*ns + (i*ss + threadIdx.x);
                unsigned int gid = (ig*n + offset + ns*blockIdx.x) + (i*ss + threadIdx.x);
                gaV[sid] = gactVec[gid];
                haV[sid] = hactVec[gid];
            }
        }
    }
    __syncthreads();
    for (int ig=0; ig<ngType; ig++) {
        double g_t = 0.0f;
        double h_t = 0.0f;
        for (int i = 0; i<ns; i++) {
            unsigned sid = ig*ns + i;
            if (gaV[sid] > 0) {
                unsigned pid = (offset + blockIdx.x*ns + i)*n + id;
                double s = preMat[pid];
                g_t += gaV[sid] * s;
                h_t += haV[sid] * s;
            }
        }
        if (gridDim.x < 32) {
            if (g_t > 0) {
                unsigned int gid = ig*n + id;
                atomicAdd(&(g[gid]), g_t);
                atomicAdd(&(h[gid]), h_t);
            }
        } else {
            // b1x = double[ngType, n/ns(gridDim.x), n]
            unsigned int b1xid = ig*n*gridDim.x + blockIdx.x*n + id;
            g_b1x[b1xid] = g_t;
            h_b1x[b1xid] = h_t;
        }
    }
}

__global__ void reduce_G(double* __restrict__ g,
                         double* __restrict__ h,
                         double* __restrict__ g_b1x, 
                         double* __restrict__ h_b1x,
                         unsigned int ngType, int n) 
{ 
    // b1x = double[ngType, n/ns(gridDim.x), n]
    // n x #(ns)
    extern __shared__ double blk[];
    double* g_blk = blk;
    double* h_blk = &(blk[blockDim.x]);
    for (int ig=0; ig<ngType; ig++) {
        unsigned int gid = ig*blockDim.x*gridDim.x + threadIdx.x*gridDim.x + blockIdx.x;
        if (gid < n) {
            // can do coalesce read optimization here (transpose in shared mem)
            g_blk[threadIdx.x] = g_b1x[gid];
            h_blk[threadIdx.x] = g_b1x[gid];
        } else {
            g_blk[threadIdx.x] = 0.0f;
            h_blk[threadIdx.x] = 0.0f;
        }
        __syncthreads();
        for (int i=blockDim.x/2; i>=32; i>>=1) {
            if (threadIdx.x < i) {
                g_blk[threadIdx.x] += g_blk[threadIdx.x + i];
                h_blk[threadIdx.x] += h_blk[threadIdx.x + i];
            }
            __syncthreads();
        }
        if (threadIdx.x < 32) {
            double g_warp = g_blk[threadIdx.x];
            double h_warp = h_blk[threadIdx.x];
            for (int offset = 16; offset > 0; offset /= 2) {
                g_warp += __shfl_down_sync(FULL_MASK, g_warp, offset);  
                h_warp += __shfl_down_sync(FULL_MASK, h_warp, offset);  
            }
            if (threadIdx.x == 0) {
                unsigned int id = ig*gridDim.x + blockIdx.x;
                g[id] += g_warp;
                h[id] += g_warp;
            }
        }
    }
}

__global__ void logRand_init(double *logRand, hiprandStateMRG32k3a *state, unsigned long long seed) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    logRand[id] = -log(hiprand_uniform_double(&localState));
    //printf("logRand0 = %f\n", logRand[id]);
    //logRand[id] = 1.0f;
    state[id] = localState;
}

__global__ void randInit(double* __restrict__ preMat, 
						 double* __restrict__ v, 
						 double* __restrict__ lTR, 
						 hiprandStateMRG32k3a* __restrict__ state,
double s, unsigned int networkSize, unsigned long long seed, double dInput) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    v[id] = vL + hiprand_uniform_double(&localState) * (vT-vL);
    for (unsigned int i=0; i<networkSize; i++) {
        preMat[i*networkSize + id] = hiprand_uniform_double(&localState) * s;
        #ifdef TEST_WITH_MANUAL_FFINPUT
            // lTR works as firstInputTime
            lTR[id] = hiprand_uniform_double(&localState)*dInput;
        #endif
    }
}

__device__ int set_input_time(double inputTime[],
                              double dt,
                              double rate,
                              double *leftTimeRate,
                              double *lastNegLogRand,
                              hiprandStateMRG32k3a* __restrict__ state)
{
    int i = 0;
    double tau, dTau, negLogRand;
    tau = (*lastNegLogRand - (*leftTimeRate))/rate;
    if (tau > dt) {
        *leftTimeRate += (dt * rate);
        return i;
    } else do {
        inputTime[i] = tau;
        negLogRand = -log(hiprand_uniform_double(state));
        dTau = negLogRand/rate;
        tau += dTau;
        i++;
        if (i == MAX_FFINPUT_PER_DT) {
            printf("exceeding max input per dt %i\n", MAX_FFINPUT_PER_DT);
            //printf("rate = %f, lastNegLogRand = %f, leftTimeRate = %f \n", rate, *lastNegLogRand, *leftTimeRate);
            //printf("inputTime[0]: %f, inputTime[1]: %f\n", inputTime[0], inputTime[1]);
            break;
        }
    } while (tau <= dt);
    *lastNegLogRand = negLogRand;
    *leftTimeRate = (dt - tau + dTau) * rate;
    return i;
}

__host__ __device__ void evolve_g(ConductanceShape &cond,
                                  double* __restrict__ g, 
                                  double* __restrict__ h, 
                                  double* __restrict__ f,
                                  double inputTime[],
                                  unsigned int nInput, double dt, unsigned int ig)
{
    cond.decay_conductance(g, h, dt, ig); 
    for (int i=0; i<nInput; i++) {
        cond.compute_single_input_conductance(g, h, *f, dt-inputTime[i], ig);
    }
}

__device__  double step(Func_RK2* lif, double dt, double tRef, unsigned int id, double gE, double gI, double tsp[]) {
    lif->tsp = dt;
    lif->spikeCount = 0;
    // not in refractory period
    if (lif->tBack < dt) {
        // return from refractory period
        if (lif->tBack > 0.0f) {
            lif->compute_pseudo_v0(dt);
            lif->tBack = -1.0f;
        }
        __syncthreads();
        lif->runge_kutta_2(dt);
        while (lif->v > vT && lif->tBack < 0.0f) {
            // crossed threshold
            lif->tsp = lif->compute_spike_time(dt); 
            tsp[lif->spikeCount] = lif->tsp;
            lif->spikeCount++;
            lif->tBack = lif->tsp + tRef;
            if (lif->tBack < dt) {
                // refractory period ended during dt
                lif->compute_pseudo_v0(dt);
                lif->runge_kutta_2(dt);
                lif->tBack = -1.0f;
            }
        }
    } 
    if (lif->tBack >= dt) {
        // during refractory period
        lif->reset_v(); 
        lif->tBack -= dt;
    }
    if (lif->spikeCount > 1) {
        printf("#%i spiked %i in one time step %f, refractory period = %f ms, only the last tsp is recorded\n", id, lif->spikeCount, dt, tRef);
    }
    return lif->tsp;
}

__device__ void Func_RK2::runge_kutta_2(double dt) {
    double fk0 = eval0(v0);
    v_hlf = v0 + dt*fk0;
    double fk1 = eval1(v_hlf);
    v = v0 + dt*(fk0+fk1)/2.0f;
}

__device__ double LIF::compute_spike_time(double dt) {
    return (vT-v0)/(v-v0)*dt;
}

__device__ void LIF::compute_v(double dt) {
    v = compute_v1(dt, a0, b0, a1, b1, vL, tBack);
}

__device__ void LIF::compute_pseudo_v0(double dt) {
    v0 = (vL-tBack*(b0 + b1 - a1*b0*dt)/2.0f)/(1.0f+tBack*(-a0 - a1 + a1*a0*dt)/2.0f);
}

__device__ void LIF::set_p0(double gE, double gI, double gL) {
    a0 = get_a(gE, gI, gL);
    b0 = get_b(gE, gI, gL); 
}

__device__ void LIF::set_p1(double gE, double gI, double gL) {
    a1 = get_a(gE, gI, gL);
    b1 = get_b(gE, gI, gL); 
}

__device__ double LIF::eval0(double _v) {
    return eval_LIF(a0,b0,_v);
}

__device__ double LIF::eval1(double _v) {
    return eval_LIF(a1,b1,_v);
}

__device__ void LIF::reset_v() {
    v = vL;
}

__device__ double compute_pseudo_v0(double a0, double b0, double a1, double b1, double dt, double tBack) {
    return (vL-tBack*(b0 + b1 - a1*b0*dt)/2.0f)/(1.0f+tBack*(-a0 - a1 + a1*a0*dt)/2.0f);
}

__device__ double runge_kutta_2(double a0, double b0, double a1, double b1, double v0, double dt, double &v_hlf) {
    double fk0 = eval_LIF(a0, b0, v0);
    v_hlf = v0 + dt*fk0;
    double fk1 = eval_LIF(a1, b1, v_hlf);
    return v0 + dt*(fk0+fk1)/2.0f;
}

__device__  double dab(Func_RK2* lif, double dt, double tRef, unsigned int id, double gE, double gI) {
    lif->tsp = dt;
    lif->spikeCount = 0;
    // not in refractory period
    if (lif->tBack < dt) {
        // return from refractory period
        if (lif->tBack > 0.0f) {
            lif->compute_pseudo_v0(dt);
            lif->tBack = -1.0f;
        }
        lif->runge_kutta_2(dt);
		if (lif->v > vT) {
			// crossed threshold
            lif->spikeCount++;
			lif->tsp = lif->compute_spike_time(dt);
			// dabbing not commiting, doest not reset v or recored tBack, TBD by spike correction.
		}
    } else {
        // during refractory period
        lif->reset_v(); 
    }
    return lif->tsp;
}

__global__ void compute_dV(double* __restrict__ v0,
                           double* __restrict__ dv,
                           double* __restrict__ gE,
                           double* __restrict__ gI,
                           double* __restrict__ hE,
                           double* __restrict__ hI,
                           double* __restrict__ a0,
                           double* __restrict__ b0,
                           double* __restrict__ a1,
                           double* __restrict__ b1,
                           double* __restrict__ preMat,
                           double* __restrict__ inputRate,
                           int* __restrict__ eventRate,
                           double* __restrict__ spikeTrain,
                           unsigned int* __restrict__ nSpike,
						   double* __restrict__ tBack,
                           double* __restrict__ gactVec,
                           double* __restrict__ hactVec,
                           double* __restrict__ fE,
                           double* __restrict__ fI,
                           double* __restrict__ leftTimeRate,
                           double* __restrict__ lastNegLogRand,
                           double* __restrict__ v_hlf,
                           hiprandStateMRG32k3a* __restrict__ state,
                           unsigned int ngTypeE, unsigned int ngTypeI, unsigned int ngType, ConductanceShape condE, ConductanceShape condI, double dt, unsigned int networkSize, unsigned int nE, unsigned long long seed, double dInput)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    LIF lif(v0[id], tBack[id]);
    double gL, tRef;
    if (id < nE) {
        tRef = tRef_E;
        gL = gL_E;
    } else {
        tRef = tRef_I;
        gL = gL_I;
    }
    /* set a0 b0 for the first step */
    double gI_t;
    double gE_t;
    // init cond E 
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gE_t += gE[networkSize*ig + id];
    }
    //  cond I 
    gI_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gI_t += gI[networkSize*ig + id];
    }
    lif.set_p0(gE_t, gI_t, gL);
    // storing for spike correction
    a0[id] = lif.a0;
    b0[id] = lif.b0;
    /* Get feedforward input */
    // consider use shared memory for dynamic allocation
    double inputTime[MAX_FFINPUT_PER_DT];
    hiprandStateMRG32k3a localState = state[id];
    int nInput;
    #ifdef TEST_WITH_MANUAL_FFINPUT
        nInput = 0;
        if (leftTimeRate[id] < dt) {
            inputTime[nInput] = leftTimeRate[id];
            nInput++;
            double tmp = leftTimeRate[id] + dInput;
            while (tmp < dt){
                inputTime[nInput] = tmp;
                nInput++;
                tmp += dInput;
            }
            leftTimeRate[id] = tmp - dt;
        } else {
            leftTimeRate[id] -= dt;
        }
    #else
        nInput = set_input_time(inputTime, dt, inputRate[id], &(leftTimeRate[id]), &(lastNegLogRand[id]), &localState);
    #endif
    //__syncwarp();
    // return a realization of Poisson input rate
    eventRate[id] = nInput;
    // update rng state 
    state[id] = localState;
    /* evolve g to t+dt with ff input only */
    unsigned int gid;
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gid = networkSize*ig + id;
        double g_i = gE[gid];
        double h_i = hE[gid];
        double f_i = fE[gid];
        evolve_g(condE, &g_i, &h_i, &f_i, inputTime, nInput, dt, ig);
        //__syncwarp();
        gE_t += g_i;
        gE[gid] = g_i;
        hE[gid] = h_i;
        // for learning
        //fE[gid] = f_i;
    }
    //printf("id %i, exc cond ready.\n",id);
    gI_t = 0.0f;
    /* no feed-forward inhibitory input (setting nInput = 0) */
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gid = networkSize*ig + id;
        double g_i = gI[gid];
        double h_i = hI[gid];
        double f_i = fI[gid];
        evolve_g(condI, &g_i, &h_i, &f_i, inputTime, 0, dt, ig);
        //__syncwarp();
        gI_t += g_i;
        gI[gid] = g_i;
        hI[gid] = h_i;
        // for learning
        //fI[gid] = f_i;
    }
    lif.set_p1(gE_t, gI_t, gL);
    // storing for spike correction
    a1[id] = lif.a1;
    b1[id] = lif.b1;
    // rk2 step
    spikeTrain[id] = dab(&lif, dt, tRef, /*the last 2 args are for deugging*/ id, gE_t, gI_t);
    nSpike[id] = lif.spikeCount;
    v_hlf[id] = lif.v_hlf;
    v0[id] = lif.v0;
	//tBack[id] = lif.tBack; // TBD after spike correction, comment this line if SSC is naive.
    if (lif.v < vI) {
		printf("#%i something is off gE = %f, gI = %f, v0 = %f, v1/2 = %f, v = %f, a0 = %f, b0 = %f, a1 = %f, b1 = %f\n", id, gE_t, gI_t, lif.v0, lif.v_hlf, lif.v, lif.a0, lif.b0, lif.a1, lif.b1);
        lif.v = vI;
    }   
    if (lif.tsp < 0.0f) {
		printf("#%i backfired v0 = %f, v1/2 = %f, v = %f, tsp = %f\n", id, lif.v0, lif.v_hlf, lif.v, lif.tsp);
        assert(lif.tsp >= 0.0f);
    }
	dv[id] = lif.v - lif.v0; // TBD after spike correction to reset etc.
}

__global__ void correct_spike(bool*   __restrict__ not_matched,
                              double* __restrict__ spikeTrain,
                              double* __restrict__ v_hlf,
                              double* __restrict__ v0,
                              double* __restrict__ dv,
                              double* __restrict__ a0,
                              double* __restrict__ b0,
                              double* __restrict__ a1,
                              double* __restrict__ b1,
                              double* __restrict__ vnew,
                              double* __restrict__ preMat,
                              double* __restrict__ tBack,
                              unsigned int* __restrict__ nSpike,
                              unsigned int ngTypeE, unsigned int ngTypeI, ConductanceShape condE, ConductanceShape condI, double dt, unsigned int poolSizeE, unsigned int poolSize) 
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    bool local_not_matched = false;
    double v_new = vL;
    double tsp = spikeTrain[id];
    double tRef;
    if (id < poolSizeE) {
        tRef = tRef_E;
    } else {
        tRef = tRef_I;
    }
    double vhlf =  v_hlf[id];
    double minTsp_i = tsp;
    double dvE = vhlf - vE;
    double dvI = vhlf - vI;
    double deltaV = 0.0; // init with old dv to be new dv
    double dg = 0.0;
    double dgV = 0.0;
    unsigned int ns = 0;
    if (tBack[id] < dt) {
        for (unsigned int i = 0; i < poolSizeE; i++) {
            double tsp_i = spikeTrain[i];  // possible share_mem optimization
            //if (tsp > tsp_i) {
            double dtij = dt - tsp_i;
            if (dtij > 0) {
                if (tsp_i < minTsp_i) {
                    minTsp_i = tsp_i;
                }
                #pragma unroll
                for (unsigned int ig = 0; ig < ngTypeE; ig++) {
                    double g = preMat[i*poolSize + id] * condE.dg(dtij, ig);
                    dg += g;
                    dgV += g*vE;
                    //if (tsp > tsp_i) {
                        deltaV -= g*dvE;
                    //}
                    if ((id == 14 || id == 436) && (i == 11 || i == 436)) {
                        printf("g-%i from %i = %e, dV = %e\n", id, i, g, -g*dvE/2.0f*dt);
                    }
                }
            }
        }
        for (unsigned int i = poolSizeE; i < poolSize; i++) {
            double tsp_i = spikeTrain[i]; // possible share_mem optimization
            //if (tsp > tsp_i) {
            double dtij = dt - tsp_i;
            if (dtij > 0) {
                #pragma unroll
                for (unsigned int ig = 0; ig < ngTypeI; ig++) {
                    double g = preMat[i*poolSize + id] * condI.dg(dtij, ig);
                    dg += g;
                    dgV += g*vI;
                    //if (tsp > tsp_i) {
                        deltaV -= g*dvI;
                    //}
                }
            }
        }
        double v0i = v0[id];
        double v_hlf0;
        double wtf_v = runge_kutta_2(a0[id], b0[id], a1[id] + dg, b1[id] + dgV, v0i, dt, v_hlf0)
        if (tsp < dt && (id == 14 || id == 436)) {
            printf("#%i, old_tsp = %e, dvold = %.15e, deltaV = %e, v0 = %e, dvhlf = %e, dvnew = %.15e, wtfv = %.15e\n", id, tsp, dv[id], deltaV/2.0f*dt, v0i, vhlf-v0i, dv[id] + deltaV/2.0f*dt, wtf_v);
        }
        tsp = dt;
        deltaV = dv[id] + deltaV/2.0f*dt;
        v_new = v0i + deltaV;
        if (v_new > vT) {
            ns++;
            tsp = dt * (vT - v0i) / deltaV;
            if (tsp < minTsp_i) {
                tsp = minTsp_i;
            }
            if (tsp + tRef < dt) {
                double wasted_tsp = tsp;
                while (v_new > vT) {
                    double v_old = v_new;
                    //v_new = compute_v1(dt, a0[id], b0[id], a1[id] + dg, b1[id] + dgV, vL, tsp + tRef);
                    double pseudo_v0 = compute_pseudo_v0(a0[id], b0[id], a1[id] + dg, b1[id] + dgV, dt, wasted_tsp + tRef);
                    v_new = runge_kutta_2(a0[id], b0[id], a1[id] + dg, b1[id] + dgV, pseudo_v0, dt, v_hlf0);
                    if (v_new > vT) {
                        wasted_tsp = dt*(vT-pseudo_v0)/(v_new-pseudo_v0);
                        //tsp += wasted_tsp;
                        ns++;
                    }
                    if (ns > 1 && (id == 436 || id == 14)) {
                        printf("i'm not here %i-%u, tsp = %e, v_old = %e, v_new = %e, v0i = %e, pseudo_v0 = %e, v_hlf = %e\n ", id, ns, tsp, v_old, v_new, v0i, pseudo_v0, v_hlf0);
                    }
                }
                //tsp = tsp/ns;
            } else {
                v_new = vL;
            }
        }
    }
    if (ns != nSpike[id]) {
        local_not_matched = true;
    }
    __syncthreads();
    not_matched[id] = local_not_matched; // OPTIMIZE per block basis
    spikeTrain[id] = tsp;
    vnew[id] = v_new;
    nSpike[id] = ns;
}

__global__ void prepare_cond(double* __restrict__ tBack,
                             double* __restrict__ spikeTrain,
                             double* __restrict__ gactVec,
                             double* __restrict__ hactVec,
                             unsigned int* __restrict__ nSpike,
                             ConductanceShape cond, double dt, unsigned int ngType, unsigned int offset, unsigned int networkSize) 
{
    unsigned int id = offset + blockIdx.x * blockDim.x + threadIdx.x;
    //setup acting vectors
    double g_end, h_end;
    double tsp = spikeTrain[id];
    double tB = tBack[id];
    unsigned int ns = nSpike[id];
    if (tsp < dt) {
        if (offset == 0) {
            tB = tsp + tRef_E;
        } else {
            tB = tsp + tRef_I;
        }
    }
    tB -= dt;
    __syncthreads();
    tBack[id] = tB;
    #pragma unroll
    for (int ig=0; ig<ngType; ig++) {
        g_end = 0.0f;
        h_end = 0.0f;
        if (tsp < dt) {
            cond.compute_single_input_conductance(&g_end, &h_end, 1.0f, dt-tsp, ig);
            g_end = g_end* ns;
            h_end = h_end* ns;
        }
        unsigned int gid = networkSize*ig + id;
        gactVec[gid] = g_end;
        hactVec[gid] = h_end;
    }
}
