#include "hip/hip_runtime.h"
#include "coredynamics.h"

__device__ void warp0_min(double* array, unsigned int* id) {
    double value = array[threadIdx.x];
    double index = id[threadIdx.x];
    __syncwarp();
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        double compare = __shfl_down_sync(FULL_MASK, value, offset);
        unsigned int comp_id = __shfl_down_sync(FULL_MASK, index, offset);
        if (value > compare) {
            value = compare;
            index = comp_id;
        }
        __syncwarp();
    }
    if (threadIdx.x == 0) {
        array[0] = value;
        id[0] = index;
    }
}
__device__ void warps_min(double* array, double data, unsigned int* id) {
	double index = threadIdx.x;
    __syncwarp();
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        double comp_data = __shfl_down_sync(FULL_MASK, data, offset);
        unsigned int comp_id = __shfl_down_sync(FULL_MASK, index, offset);
        if (data > comp_data) {
            data = comp_data;
            index = comp_id;
        }
        __syncwarp();
    }
    __syncthreads();
    if (threadIdx.x % warpSize == 0) {
        unsigned int head = threadIdx.x/warpSize;
        array[head] = data;
        id[head] = index;
    }
}

__device__ void find_min(double* array, double data, unsigned int* id) { 
	warps_min(array, data, id);
    __syncthreads();
    if (threadIdx.x < warpSize) {
        warp0_min(array, id);
    }
    __syncthreads();
}

__device__ void warps_reduce(unsigned int* array) {
    unsigned int data = array[threadIdx.x];
    __syncwarp();
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        data += __shfl_down_sync(FULL_MASK, data, offset);
    }
    __syncthreads();
    if (threadIdx.x % warpSize == 0) {
        array[threadIdx.x/warpSize] = data;
    }
}

__device__ void warp0_reduce(unsigned int* array) {
    unsigned int data = array[threadIdx.x];
    __syncwarp();
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        data += __shfl_down_sync(FULL_MASK, data, offset);
    }
    if (threadIdx.x == 0) {
        array[0] = data;
    }
}

__device__ void block_reduce(unsigned int* array) {
	warps_reduce(array);
    __syncthreads();
    if (threadIdx.x < warpSize) {
        warp0_reduce(array);
    }
    __syncthreads();
}

__global__ void logRand_init(double *logRand, hiprandStateMRG32k3a *state, unsigned long long seed, double *lTR, double dInput, unsigned int offset) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id+offset, 0, 0, &localState);
    logRand[id] = -log(hiprand_uniform_double(&localState));
    state[id] = localState;

    #ifdef TEST_WITH_MANUAL_FFINPUT
        lTR[id] = hiprand_uniform_double(&localState)*dInput;
    #else
        lTR[id] = hiprand_uniform_double(&localState);
    #endif
}

__global__ void randInit(double* __restrict__ preMat, 
						 double* __restrict__ v, 
						 hiprandStateMRG32k3a* __restrict__ state,
double sEE, double sIE, double sEI, double sII, unsigned int networkSize, unsigned int nE, unsigned long long seed) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    v[id] = vL + hiprand_uniform_double(&localState) * (vT-vL) * 0.5;
    double mean, std, ratio;
    if (id < nE) {
        mean = log(sEE/sqrt(1.0f+1.0f/sEE));
        std = sqrt(log(1.0f+1.0f/sEE));
        ratio = 0.0;
        for (unsigned int i=0; i<nE; i++) {
            double x = hiprand_log_normal_double(&localState, mean, std);
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sEE > 0) {
            ratio = sEE * nE / ratio;
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
        //mean = log(sEI/sqrt(1.0f+1.0f/sEI));
        //std = sqrt(log(1.0f+1.0f/sEI));
        mean = sEI;
        std = sEI*0.125;
        ratio = 0.0;
        for (unsigned int i=nE; i<networkSize; i++) {
            //double x = hiprand_log_normal_double(&localState, mean, std);
            double x = hiprand_normal_double(&localState)*std+mean;
            if (x<0) x = 0;
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sEI > 0){
            ratio = sEI * (networkSize-nE) / ratio;
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
    } else {
        //mean = log(sIE/sqrt(1.0f+1.0f/sIE));
        //std = sqrt(log(1.0f+1.0f/sIE));
        mean = sIE;
        std = sIE*0.125;
        ratio = 0.0;
        for (unsigned int i=0; i<nE; i++) {
            //double x = hiprand_log_normal_double(&localState, mean, std);
            double x = hiprand_normal_double(&localState)*std+mean;
            if (x<0) x = 0;
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sIE > 0) {
            ratio = sIE * nE / ratio;
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
        //mean = log(sII/sqrt(1.0f+1.0f/sII));
        //std = sqrt(log(1.0f+1.0f/sII));
        mean = sII;
        std = sII*0.125;
        ratio = 0.0;
        for (unsigned int i=nE; i<networkSize; i++) {
            //double x = hiprand_log_normal_double(&localState, mean, std);
            double x = hiprand_normal_double(&localState)*std+mean;
            if (x<0) x = 0;
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sII > 0){
            ratio = sII * (networkSize-nE) / ratio;
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
    }
}

__global__ void f_init(double* __restrict__ f, unsigned networkSize, unsigned int nE, unsigned int ngType, double Ef, double If) {
    unsigned long id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < nE) {
        for (unsigned int ig=0; ig<ngType; ig++) {
            f[ig*networkSize + id] = Ef;
        }
    } else {
        for (unsigned int ig=0; ig<ngType; ig++) {
            f[ig*networkSize + id] = If;
        }
    }
}

__device__ int set_input_time(double inputTime[],
                              double dt,
                              double rate,
                              double *leftTimeRate,
                              double *lastNegLogRand,
                              hiprandStateMRG32k3a* __restrict__ state)
{
    int i = 0;
    double tau, dTau, negLogRand;
    tau = (*lastNegLogRand - (*leftTimeRate))/rate;
    if (tau > dt) {
        *leftTimeRate += (dt * rate);
        return i;
    } else do {
        inputTime[i] = tau;
        negLogRand = -log(hiprand_uniform_double(state));
        dTau = negLogRand/rate;
        tau += dTau;
        i++;
        if (i == MAX_FFINPUT_PER_DT) {
            printf("exceeding max input per dt %i\n", MAX_FFINPUT_PER_DT);
            break;
        }
    } while (tau <= dt);
    *lastNegLogRand = negLogRand;
    *leftTimeRate = (dt - tau + dTau) * rate;
    return i;
}

__host__ __device__ void evolve_g(ConductanceShape &cond,
                                  double* __restrict__ g, 
                                  double* __restrict__ h, 
                                  double* __restrict__ f,
                                  double inputTime[],
                                  int nInput, double dt, unsigned int ig)
{
    cond.decay_conductance(g, h, dt, ig);
    for (int i=0; i<nInput; i++) {
        cond.compute_single_input_conductance(g, h, *f, dt-inputTime[i], ig);
    }
}

__device__ void prep_cond(LIF* lif, ConductanceShape &condE, double gE[], double hE[], double fE[], double inputTimeE[], int nInputE,
	ConductanceShape &condI, double gI[], double hI[], double fI[], double inputTimeI[], int nInputI, double gL, double dt) {
    // p0 should already be ready.
	double gE_t = 0.0f;
    #pragma unroll
	for (int ig=0; ig<ngTypeE; ig++) {
		evolve_g(condE, &gE[ig], &hE[ig], &fE[ig], inputTimeE, nInputE, dt, ig);
		gE_t += gE[ig];
	}
	double gI_t = 0.0f;
    #pragma unroll
	for (int ig=0; ig<ngTypeI; ig++) {
		evolve_g(condI, &gI[ig], &hI[ig], &fI[ig], inputTimeI, nInputI, dt, ig);
		gI_t += gI[ig];
	}
	lif->set_p1(gE_t, gI_t, gL);
}

__device__ void set_p(LIF* lif, double gE0[], double gI0[], double gE1[], double gI1[], double gL) {
	double gE_t = 0.0f;
    #pragma unroll
	for (unsigned int ig=0; ig<ngTypeE; ig++) {
		gE_t += gE0[ig];
	}
	double gI_t = 0.0f;
    #pragma unroll
	for (unsigned int ig=0; ig<ngTypeI; ig++) {
		gI_t += gI0[ig];
	}
	lif->set_p0(gE_t, gI_t, gL);

	gE_t = 0.0f;
    #pragma unroll
	for (unsigned int ig=0; ig<ngTypeE; ig++) {
		gE_t += gE1[ig];
	}
	gI_t = 0.0f;
    #pragma unroll
	for (unsigned int ig=0; ig<ngTypeI; ig++) {
		gI_t += gI1[ig];
	}
	lif->set_p1(gE_t, gI_t, gL);
}

__device__  void one(LIF* lif, double dt, double tRef, unsigned int id, double gE, double gI) {
    lif->tsp = dt;
    lif->spikeCount = 0;
    // not in refractory period
    if (lif->tBack < dt) {
        // return from refractory period
        if (lif->tBack > 0.0f) {
            lif->recompute_v0(dt);
            lif->tBack = -1.0;
        }
        lif->compute_v(dt);
        while (lif->v > vT) {
            // crossed threshold
            lif->compute_spike_time(dt);
            lif->spikeCount++;
            lif->tBack = lif->tsp + tRef;
            if (lif->tBack < dt) {
                lif->recompute(dt);
            } else {
                break;
            }
        }
    }
    __syncwarp();
    if (lif->tBack >= dt) {
        lif->reset_v();
        lif->tBack -= dt;
    }
}

__device__  void initial(LIF* lif, double dt) {
    lif->tsp = dt;
    lif->correctMe = true;
    lif->spikeCount = 0;
    // not in refractory period
    if (lif->tBack < dt) {
        // return from refractory period
        if (lif->tBack > 0) {
            lif->recompute_v0(dt);
        }
        lif->compute_v(dt);
        if (lif->v > vT) {
            // crossed threshold
            lif->compute_spike_time(dt); 
        }
    } else {
        lif->reset_v();
        lif->correctMe = false;
    }
}

__device__  void step(LIF* lif, double t0, double t1, double tRef) {
    // not in refractory period
    if (lif->tBack < t1) {
        double dt = t1 - t0;
        // return from refractory period
        if (lif->tBack > t0) {
            lif->recompute_v0(dt, t0);
        }
        lif->compute_v(dt);
        while (lif->v > vT) {
            // crossed threshold
            lif->compute_spike_time(dt, t0); 
            lif->spikeCount++;
            lif->tBack = lif->tsp + tRef;
            if (lif->tBack < t1) {
                lif->recompute(dt, t0);
            } else {
                break;
            }
        }
    }
    if (lif->v < vI) {
        lif->v = vI;
    }
}

__device__  void dab(LIF* lif, double t0, double _dt) {
    double dt = _dt - t0;
    // return from refractory period
    //#ifdef DEBUG
    assert(lif->tBack < _dt);
	//#endif
    if (lif->tBack > t0) {
        lif->recompute_v0(dt, t0);
    }
    lif->compute_v(dt);
    if (lif->v > vT) {
        // crossed threshold
        lif->compute_spike_time(dt, t0); 
    }
}

__global__ void 
__launch_bounds__(blockSize, 1)
compute_V(double* __restrict__ v,
          double* __restrict__ gE,
          double* __restrict__ gI,
          double* __restrict__ hE,
          double* __restrict__ hI,
          double* __restrict__ preMat,
          double* __restrict__ inputRateE,
          double* __restrict__ inputRateI,
          int* __restrict__ eventRateE,
          int* __restrict__ eventRateI,
          double* __restrict__ spikeTrain,
          unsigned int* __restrict__ nSpike,
          double* __restrict__ tBack,
          double* __restrict__ fE,
          double* __restrict__ fI,
          double* __restrict__ leftTimeRateE,
          double* __restrict__ leftTimeRateI,
          double* __restrict__ lastNegLogRandE,
          double* __restrict__ lastNegLogRandI,
          hiprandStateMRG32k3a* __restrict__ stateE,
          hiprandStateMRG32k3a* __restrict__ stateI,
          ConductanceShape condE, ConductanceShape condI, double dt, unsigned int networkSize, unsigned int nE, unsigned long long seed, double dInputE, double dInputI)
{
    __shared__ double tempSpike[blockSize];
    __shared__ unsigned int spid[warpSize];
    __shared__ unsigned int spikeCount[blockSize];
    unsigned int id = threadIdx.x;
    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    #ifdef IMPLICIT
        impl_rk2 lif(v[id], tBack[id]);
    #else
        rk2 lif(v[id], tBack[id]);
    #endif
    double gL, tRef;
    if (id < nE) {
        tRef = tRef_E;
        gL = gL_E;
    } else {
        tRef = tRef_I;
        gL = gL_I;
    }
    // for committing conductance
    double gE_local[ngTypeE];
    double hE_local[ngTypeE];
    double gI_local[ngTypeI];
    double hI_local[ngTypeI];
    // for not yet committed conductance
    double gE_retrace[ngTypeE];
    double hE_retrace[ngTypeE];
    double gI_retrace[ngTypeI];
    double hI_retrace[ngTypeI];

    // init cond E 
	double fE_local[ngTypeE];
	double gE_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeE; ig++) {
		unsigned int gid = networkSize * ig + id;
        gE_local[ig] = gE[gid];
        hE_local[ig] = hE[gid];
		fE_local[ig] = fE[gid];
        gE_retrace[ig] = gE_local[ig];
        hE_retrace[ig] = hE_local[ig];
		gE_t += gE_local[ig];
    }
    //  cond I 
	double fI_local[ngTypeI];
	double gI_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeI; ig++) {
		unsigned int gid = networkSize * ig + id;
		gI_local[ig] = gI[gid];
        hI_local[ig] = hI[gid];
		fI_local[ig] = fI[gid];
        gI_retrace[ig] = gI_local[ig];
        hI_retrace[ig] = hI_local[ig];
		gI_t += gI_local[ig];
    }
	lif.set_p0(gE_t, gI_t, gL);
    /* Get feedforward input */
    double inputTimeE[MAX_FFINPUT_PER_DT];
    double inputTimeI[MAX_FFINPUT_PER_DT];
    int nInputE=0, nInputI=0;
    #ifdef TEST_WITH_MANUAL_FFINPUT
        if (leftTimeRateE[id] < dt) {
            inputTimeE[nInputE] = leftTimeRateE[id];
            nInputE++;
            double tmp = leftTimeRateE[id] + dInputE;
            while (tmp < dt){
                inputTimeE[nInputE] = tmp;
                nInputE++;
                tmp += dInputE;
            }
            leftTimeRateE[id] = tmp - dt;
        } else {
            leftTimeRateE[id] -= dt;
        }

        if (leftTimeRateI[id] < dt) {
            inputTimeI[nInputI] = leftTimeRateI[id];
            nInputI++;
            double tmp = leftTimeRateI[id] + dInputI;
            while (tmp < dt){
                inputTimeI[nInputI] = tmp;
                nInputI++;
                tmp += dInputI;
            }
            leftTimeRateI[id] = tmp - dt;
        } else {
            leftTimeRateI[id] -= dt;
        }
    #else
        hiprandStateMRG32k3a localStateE;
        hiprandStateMRG32k3a localStateI;
        double irE = inputRateE[id];
        double irI = inputRateI[id];
        if (irE > 0) {
            localStateE = stateE[id];
            nInputE = set_input_time(inputTimeE, dt, irE, &(leftTimeRateE[id]), &(lastNegLogRandE[id]), &localStateE);
		    stateE[id] = localStateE;
        }
        if (irI > 0) {
            localStateI = stateI[id];
		    nInputI = set_input_time(inputTimeI, dt, irI, &(leftTimeRateI[id]), &(lastNegLogRandI[id]), &localStateI);
		    stateI[id] = localStateI;
        }
    #endif
    // return a realization of Poisson input rate
	#ifndef FULL_SPEED
		eventRateE[id] = nInputE;
		eventRateI[id] = nInputI;
	#endif
    // set conductances
    prep_cond(&lif, condE, gE_local, hE_local, fE_local, inputTimeE, nInputE, condI, gI_local, hI_local, fI_local, inputTimeI, nInputI, gL, dt); 
    spikeTrain[id] = dt;
    // initial spike guess
    #ifdef DEBUG
        double old_v0 = lif.v0;
        double old_tBack = lif.tBack;
    #endif
    initial(&lif, dt);
    #ifdef DEBUG
    	if (lif.tsp < dt) {
    		printf("first %u: v0 = %e, v = %e->%e, tBack %e->%e tsp %e\n", id, old_v0, lif.v0, lif.v, old_tBack, lif.tBack, lif.tsp);	
    	}
    #endif
	assert(lif.tsp > 0);
    // spike-spike correction
	//__syncthreads();
	find_min(tempSpike, lif.tsp, spid);
	double t0 = 0.0;
    double t_hlf = tempSpike[0];
    unsigned int imin = spid[0];
	__syncthreads();
    #ifdef DEBUG
    	if (id == 0) {
    		if (t_hlf == dt) {
    			printf("first_ no spike\n");
    		} else {
    			printf("first_ %u: %e < %e ?\n", imin, t_hlf, dt);
    		}
    	}
    #endif
    int iInputE = 0, iInputI = 0;
    int jInputE, jInputI;
    while (t_hlf < dt) {
        // t0 ------- min ---t_hlf
        lif.tsp = dt;
        /************ This may be optimized to be per warp decision **************/
        //unsigned int MASK = __ballot_sync(FULL_MASK, lif.correctMe);
        if (lif.correctMe) {
            double new_dt = t_hlf - t0;
            // prep inputTime
            jInputE = iInputE;
            if (jInputE < nInputE) {
                while (inputTimeE[jInputE] < t_hlf) {
                    jInputE++;
                    if (jInputE == nInputE) break;
                }
            }
            jInputI = iInputI;
            if (jInputI < nInputI) {
                while (inputTimeI[jInputI] < t_hlf) {
                    jInputI++;
                    if (jInputI == nInputI) break;
                }
            }
            // prep retracable conductance
            prep_cond(&lif, condE, gE_retrace, hE_retrace, fE_local, &inputTimeE[iInputE], jInputE-iInputE, condI, gI_retrace, hI_retrace, hI_local, &inputTimeI[iInputI], jInputI-iInputI, gL, new_dt); 
            // commit for next ext. inputs.
            iInputE = jInputE;
            iInputI = jInputI;
            // get tsp decided
            #ifdef DEBUG
			    double old_v0 = lif.v0;
			    double old_tBack = lif.tBack;
            #endif
            unsigned int old_count = lif.spikeCount;
            step(&lif, t0, t_hlf, tRef);
            if (id == imin && lif.tsp == dt) {
                lif.spikeCount++;
                lif.tsp = t_hlf;
                lif.tBack = t_hlf + tRef;
            }
            if (lif.tsp < dt) {
                spikeTrain[id] = lif.tsp;
                spikeCount[id] = lif.spikeCount - old_count;
                //lif.tsp = t_hlf;
                if (lif.tBack >= dt) {
                    lif.reset_v();
                    lif.correctMe = false;
                }
                #ifdef DEBUG
                    printf("t0: %e, t_hlf: %e\n", t0, t_hlf);
		            printf("hlf %u: v0 = %e, v = %e->%e, tBack %e->%e tsp %ex%i\n", id, old_v0, lif.v0, lif.v, old_tBack, lif.tBack, lif.tsp, spikeCount[id]);
                #endif
				assert(lif.tsp <= t_hlf+EPS);
				assert(lif.tsp > t0-EPS);
            }
            if (lif.v > vT) {
                assert(lif.tBack < dt && lif.tBack >= t_hlf-EPS);
            }
        }
        __syncwarp();
        tempSpike[id] = lif.tsp;
        __syncthreads();
        // commit the spikes
        #ifdef DEBUG
            int counter = 0;
        #endif
        #pragma unroll
        for (unsigned int i=0; i<blockSize; i++) {
            double tsp = tempSpike[i];
            if (tsp < dt) {
                //tsp = t_hlf;
                double strength = preMat[i*networkSize + id] * spikeCount[i];
                double dtsp = t_hlf-tsp;
                #ifdef DEBUG
                    counter++;
                    if (id==0) {
			        	printf("%u: %e\n", i, tsp);
                    }
                #endif
                if (i < nE) {
                    #pragma unroll
			    	for (unsigned int ig=0; ig<ngTypeE; ig++) {
                        if (dtsp == 0) {
                            hE_retrace[ig] += strength;
                        } else {
                            condE.compute_single_input_conductance(&gE_retrace[ig], &hE_retrace[ig], strength, dtsp, ig);
                        }
                        condE.compute_single_input_conductance(&gE_local[ig], &hE_local[ig], strength, dt-tsp, ig);
                    }
			    } else {
			    	#pragma unroll
			    	for (unsigned int ig=0; ig<ngTypeI; ig++) {
                        if (dtsp == 0) {
                            hI_retrace[ig] += strength;
                        } else {
			    		    condI.compute_single_input_conductance(&gI_retrace[ig], &hI_retrace[ig], strength, dtsp, ig);
                        }
			    		condI.compute_single_input_conductance(&gI_local[ig], &hI_local[ig], strength, dt-tsp, ig);
			    	}
                }
            }
        }
		__syncthreads();
        #ifdef DEBUG
            if (id == 0) {
                printf("t0-t_hlf: %i spikes\n", counter);
            }
        #endif
        // t_hlf ------------- dt

        lif.tsp = dt;
        if (lif.correctMe) {
            set_p(&lif, gE_retrace, gI_retrace, gE_local, gI_local, gL);                                        
            lif.v0 = lif.v;
            //get putative tsp
            #ifdef DEBUG
			    double old_v0 = lif.v0;
			    double old_tBack = lif.tBack;
            #endif
            dab(&lif, t_hlf, dt);
            #ifdef DEBUG
			    if (lif.tsp < dt) {
		            printf("end %u: v0 = %e, v = %e->%e, tBack %e->%e tsp %e > %e\n", id, old_v0, lif.v0, lif.v, old_tBack, lif.tBack, lif.tsp, t_hlf); 
                }
            #endif
            assert(lif.tsp <= dt);
			assert(lif.tsp > t_hlf-EPS);
        }
		// next spike
        find_min(tempSpike, lif.tsp, spid);
		t0 = t_hlf;
        t_hlf = tempSpike[0];
        imin = spid[0];
		__syncthreads();
        #ifdef DEBUG
        	if (id == 0) {
        		if (t_hlf == dt) {
        			printf("end_ no spike\n");
        		} else {
        			printf("end_  %u: %e < %e ?\n", imin, t_hlf, dt);
        		}
        	}
        #endif
    }
    #ifdef DEBUG
        if (lif.v > vT) {
	        printf( "after_ %u-%i: v = %e->%e, tBack = %e, tsp = %e==%e, t_hlf = %e\n", id, lif.correctMe, lif.v0, lif.v, lif.tBack, lif.tsp, tempSpike[id], t_hlf);
	    }
    #endif
	assert(lif.v < vT);
    //__syncwarp();
    // commit conductance to global mem
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeE; ig++) {
	    unsigned int gid = networkSize * ig + id;
        gE[gid] = gE_local[ig];
        hE[gid] = hE_local[ig];
    }
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeI; ig++) {
	    unsigned int gid = networkSize * ig + id;
        gI[gid] = gI_local[ig];
        hI[gid] = hI_local[ig];
    }
    nSpike[id] = lif.spikeCount;
	v[id] = lif.v;
    if (lif.tBack > 0) {
        tBack[id] = lif.tBack - dt;
    }
}

__global__ void 
__launch_bounds__(blockSize, 1)
compute_V_without_ssc(double* __restrict__ v,
                      double* __restrict__ gE,
                      double* __restrict__ gI,
                      double* __restrict__ hE,
                      double* __restrict__ hI,
                      double* __restrict__ preMat,
                      double* __restrict__ inputRateE,
                      double* __restrict__ inputRateI,
                      int* __restrict__ eventRateE,
                      int* __restrict__ eventRateI,
                      double* __restrict__ spikeTrain,
                      unsigned int* __restrict__ nSpike,
                      double* __restrict__ tBack,
                      double* __restrict__ fE,
                      double* __restrict__ fI,
                      double* __restrict__ leftTimeRateE,
                      double* __restrict__ leftTimeRateI,
                      double* __restrict__ lastNegLogRandE,
                      double* __restrict__ lastNegLogRandI,
                      hiprandStateMRG32k3a* __restrict__ stateE,
                      hiprandStateMRG32k3a* __restrict__ stateI,
                      ConductanceShape condE, ConductanceShape condI, double dt, unsigned int networkSize, unsigned int nE, unsigned long long seed, double dInputE, double dInputI)
{
    __shared__ double spike[blockSize];
    __shared__ unsigned int nsp[blockSize];
    unsigned int id = threadIdx.x;
    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    #ifdef IMPL
        impl_rk2 lif(v[id], tBack[id]);
    #else
        rk2 lif(v[id], tBack[id]);
    #endif
    double gL, tRef;
    if (id < nE) {
        tRef = tRef_E;
        gL = gL_E;
    } else {
        tRef = tRef_I;
        gL = gL_I;
    }
    double gE_local[ngTypeE];
    double hE_local[ngTypeE];
    double gI_local[ngTypeI];
    double hI_local[ngTypeI];
    /* set a0 b0 for the first step */
    double gI_t;
    double gE_t;
    // init cond E 
    unsigned int gid;
    gE_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeE; ig++) {
        gid = networkSize*ig + id;
        gE_local[ig] = gE[gid];
        hE_local[ig] = hE[gid];
        gE_t += gE_local[ig];
    }
    //  cond I 
    gI_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeI; ig++) {
        gid = networkSize*ig + id;
        gI_local[ig] = gI[gid];
        hI_local[ig] = hI[gid];
        gI_t += gI_local[ig];
    }
    lif.set_p0(gE_t, gI_t, gL);
    /* Get feedforward input */
    // consider use shared memory for dynamic allocation
    double inputTimeE[MAX_FFINPUT_PER_DT];
    double inputTimeI[MAX_FFINPUT_PER_DT];
    int nInputE=0, nInputI=0;
    #ifdef TEST_WITH_MANUAL_FFINPUT
        if (leftTimeRateE[id] < dt) {
            inputTimeE[nInputE] = leftTimeRateE[id];
            nInputE++;
            double tmp = leftTimeRateE[id] + dInputE;
            while (tmp < dt) {
                inputTimeE[nInputE] = tmp;
                nInputE++;
                tmp += dInputE;
            }
            leftTimeRateE[id] = tmp - dt;
        } else {
            leftTimeRateE[id] -= dt;
        }

        if (leftTimeRateI[id] < dt) {
            inputTimeI[nInputI] = leftTimeRateI[id];
            nInputI++;
            double tmp = leftTimeRateI[id] + dInputI;
            while (tmp < dt){
                inputTimeI[nInputI] = tmp;
                nInputI++;
                tmp += dInputI;
            }
            leftTimeRateI[id] = tmp - dt;
        } else {
            leftTimeRateI[id] -= dt;
        }
    #else
        hiprandStateMRG32k3a localStateE;
        hiprandStateMRG32k3a localStateI;
        double irE = inputRateE[id];
        double irI = inputRateI[id];
        if (irE > 0) {
            localStateE = stateE[id];
            nInputE = set_input_time(inputTimeE, dt, irE, &(leftTimeRateE[id]), &(lastNegLogRandE[id]), &localStateE);
		    stateE[id] = localStateE;
        }
        if (irI > 0) {
            localStateI = stateI[id];
		    nInputI = set_input_time(inputTimeI, dt, irI, &(leftTimeRateI[id]), &(lastNegLogRandI[id]), &localStateI);
		    stateI[id] = localStateI;
        }
    #endif
    //__syncwarp();
    // return a realization of Poisson input rate
    #ifndef FULL_SPEED
        eventRateE[id] = nInputE;
        eventRateI[id] = nInputI;
    #endif
    /* evolve g to t+dt with ff input only */
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gid = networkSize*ig + id;
        evolve_g(condE, &gE_local[ig], &hE_local[ig], &fE[gid], inputTimeE, nInputE, dt, ig);
        gE_t += gE_local[ig];
    }
    gI_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gid = networkSize*ig + id;
        evolve_g(condI, &gI_local[ig], &hI_local[ig], &fI[gid], inputTimeI, nInputI, dt, ig);
        gI_t += gI_local[ig];
    }
    lif.set_p1(gE_t, gI_t, gL);

    // implicit rk2 step
    one(&lif, dt, tRef, id, gE_t, gI_t);
    __syncthreads();
	assert(lif.v <= vT);
    assert(lif.tsp > 0);
    __syncwarp();

    // write data to global
    spikeTrain[id] = lif.tsp;
    nSpike[id] = lif.spikeCount;
    tBack[id] = lif.tBack;
	v[id] = lif.v;
    spike[id] = lif.tsp;
    nsp[id] = lif.spikeCount;
    __syncthreads();

    // recalibrate conductance from cortical spikes
    #pragma unroll
    for (unsigned int i=0; i<blockSize; i++) {
        double spikeCount = nsp[i];
        if (spikeCount > 0) {
            double strength = preMat[i*networkSize + id] * spikeCount;
            if (i < nE) {
                #pragma unroll
                for (unsigned int ig=0; ig<ngTypeE; ig++) {
                    condE.compute_single_input_conductance(&gE_local[ig], &hE_local[ig], strength, dt-spike[i], ig);
                }
            } else {
                #pragma unroll
                for (unsigned int ig=0; ig<ngTypeI; ig++) {
                    condI.compute_single_input_conductance(&gI_local[ig], &hI_local[ig], strength, dt-spike[i], ig);
                }
            }
        }
    }

    // update conductance to global memory
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeE; ig++) {
        gE[id] = gE_local[ig];
        hE[id] = hE_local[ig];
    }
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeI; ig++) {
        gI[id] = gI_local[ig];
        hI[id] = hI_local[ig];
    }
}
