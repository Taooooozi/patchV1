#include "hip/hip_runtime.h"
#include "coredynamics.h"

__global__ void recal_G(double* __restrict__ g,
                        double* __restrict__ h,
                        double* __restrict__ preMat,
                        double* __restrict__ gactVec,
                        double* __restrict__ hactVec,
                        double* __restrict__ g_b1x,
                        double* __restrict__ h_b1x,
                        unsigned int n, unsigned int offset, unsigned int ngType, unsigned int ns, int m) 
{
    // 2D blockGrid
    // -> D-1 pieces of actVec 
    // -> D-2 pieces of post-synaptic neurons 
    // 1D threadBlock
    extern __shared__ double actVec[];
    double *gaV = actVec;
    double *haV = &(actVec[ngType*ns]);
    unsigned int id = blockDim.x*blockIdx.y + threadIdx.x;
    unsigned int ss = ns/m;
    #pragma unroll
    for (int ig=0; ig<ngType; ig++) {
        #pragma unroll
        for (int i=0; i<m; i++) {
            // av = double[ngType,#(ns),ns]
            // actVec = double[ngType,n]
            if (threadIdx.x < ss) {
                unsigned int sid = ig*ns + (i*ss + threadIdx.x);
                unsigned int gid = (ig*n + offset + ns*blockIdx.x) + (i*ss + threadIdx.x);
                gaV[sid] = gactVec[gid];
                haV[sid] = hactVec[gid];
            }
        }
    }
    __syncthreads();
    for (int ig=0; ig<ngType; ig++) {
        double g_t = 0.0f;
        double h_t = 0.0f;
        for (int i = 0; i<ns; i++) {
            unsigned sid = ig*ns + i;
            if (gaV[sid] > 0) {
                unsigned pid = (offset + blockIdx.x*ns + i)*n + id;
                double s = preMat[pid];
                g_t += gaV[sid] * s;
                h_t += haV[sid] * s;
            }
        }
        if (gridDim.x < 32) {
            if (g_t > 0) {
                unsigned int gid = ig*n + id;
                atomicAdd(&(g[gid]), g_t);
                atomicAdd(&(h[gid]), h_t);
            }
        } else {
            // b1x = double[ngType, n/ns(gridDim.x), n]
            unsigned int b1xid = ig*n*gridDim.x + blockIdx.x*n + id;
            g_b1x[b1xid] = g_t;
            h_b1x[b1xid] = h_t;
        }
    }
}

__global__ void reduce_G(double* __restrict__ g,
                         double* __restrict__ h,
                         double* __restrict__ g_b1x, 
                         double* __restrict__ h_b1x,
                         unsigned int ngType, int n) 
{ 
    // b1x = double[ngType, n/ns(gridDim.x), n]
    // n x #(ns)
    extern __shared__ double blk[];
    double* g_blk = blk;
    double* h_blk = &(blk[blockDim.x]);
    for (int ig=0; ig<ngType; ig++) {
        unsigned int gid = ig*blockDim.x*gridDim.x + threadIdx.x*gridDim.x + blockIdx.x;
        if (gid < n) {
            // can do coalesce read optimization here (transpose in shared mem)
            g_blk[threadIdx.x] = g_b1x[gid];
            h_blk[threadIdx.x] = g_b1x[gid];
        } else {
            g_blk[threadIdx.x] = 0.0f;
            h_blk[threadIdx.x] = 0.0f;
        }
        __syncthreads();
        for (int i=blockDim.x/2; i>=32; i>>=1) {
            if (threadIdx.x < i) {
                g_blk[threadIdx.x] += g_blk[threadIdx.x + i];
                h_blk[threadIdx.x] += h_blk[threadIdx.x + i];
            }
            __syncthreads();
        }
        if (threadIdx.x < 32) {
            double g_warp = g_blk[threadIdx.x];
            double h_warp = h_blk[threadIdx.x];
            for (int offset = 16; offset > 0; offset /= 2) {
                g_warp += __shfl_down_sync(FULL_MASK, g_warp, offset);  
                h_warp += __shfl_down_sync(FULL_MASK, h_warp, offset);  
            }
            if (threadIdx.x == 0) {
                unsigned int id = ig*gridDim.x + blockIdx.x;
                g[id] += g_warp;
                h[id] += g_warp;
            }
        }
    }
}

__global__ void logRand_init(double *logRand, hiprandStateMRG32k3a *state, unsigned long long seed, double *lTR, double dInput) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    logRand[id] = -log(hiprand_uniform_double(&localState));
    state[id] = localState;

    // lTR works as firstInputTime
    #ifdef TEST_WITH_MANUAL_FFINPUT
        lTR[id] = hiprand_uniform_double(&localState)*dInput;
    #endif
}

__global__ void randInit(double* __restrict__ preMat, 
						 double* __restrict__ v, 
						 hiprandStateMRG32k3a* __restrict__ state,
double sEE, double sIE, double sEI, double sII, unsigned int networkSize, unsigned int nE, unsigned long long seed) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed+id, 0, 0, &localState);
    v[id] = vL + hiprand_uniform_double(&localState) * (vT-vL);
    double mean, std, ratio;
    if (id < nE) {
        mean = log(sEE/sqrt(1.0f+1.0f/sEE));
        std = sqrt(log(1.0f+1.0f/sEE));
        ratio = 0.0;
        for (unsigned int i=0; i<nE; i++) {
            double x = hiprand_log_normal_double(&localState, mean, std);
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sEE > 0) {
            ratio = sEE * nE / ratio;
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
        //mean = log(sEI/sqrt(1.0f+1.0f/sEI));
        //std = sqrt(log(1.0f+1.0f/sEI));
        mean = sEI;
        std = sEI*0.125;
        ratio = 0.0;
        for (unsigned int i=nE; i<networkSize; i++) {
            //double x = hiprand_log_normal_double(&localState, mean, std);
            double x = hiprand_normal_double(&localState)*std+mean;
            if (x<0) x = 0;
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sEI > 0){
            ratio = sEI * (networkSize-nE) / ratio;
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
    } else {
        //mean = log(sIE/sqrt(1.0f+1.0f/sIE));
        //std = sqrt(log(1.0f+1.0f/sIE));
        mean = sIE;
        std = sIE*0.125;
        ratio = 0.0;
        for (unsigned int i=0; i<nE; i++) {
            //double x = hiprand_log_normal_double(&localState, mean, std);
            double x = hiprand_normal_double(&localState)*std+mean;
            if (x<0) x = 0;
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sIE > 0) {
            ratio = sIE * nE / ratio;
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=0; i<nE; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
        //mean = log(sII/sqrt(1.0f+1.0f/sII));
        //std = sqrt(log(1.0f+1.0f/sII));
        mean = sII;
        std = sII*0.125;
        ratio = 0.0;
        for (unsigned int i=nE; i<networkSize; i++) {
            //double x = hiprand_log_normal_double(&localState, mean, std);
            double x = hiprand_normal_double(&localState)*std+mean;
            if (x<0) x = 0;
            preMat[i*networkSize + id] = x;
            ratio += x;
        }
        if (sII > 0){
            ratio = sII * (networkSize-nE) / ratio;
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = preMat[i*networkSize + id]*ratio;
            }
        } else {
            for (unsigned int i=nE; i<networkSize; i++) {
                preMat[i*networkSize + id] = 0.0f;
            }
        }
    }
}

__global__ void f_init(double* __restrict__ f, unsigned networkSize, unsigned int nE, unsigned int ngType, double Ef, double If) {
    unsigned long id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < nE) {
        for (unsigned int ig=0; ig<ngType; ig++) {
            f[ig*networkSize + id] = Ef;
        }
    } else {
        for (unsigned int ig=0; ig<ngType; ig++) {
            f[ig*networkSize + id] = If;
        }
    }
}

__device__ int set_input_time(double inputTime[],
                              double dt,
                              double rate,
                              double *leftTimeRate,
                              double *lastNegLogRand,
                              hiprandStateMRG32k3a* __restrict__ state)
{
    int i = 0;
    double tau, dTau, negLogRand;
    tau = (*lastNegLogRand - (*leftTimeRate))/rate;
    if (tau > dt) {
        *leftTimeRate += (dt * rate);
        return i;
    } else do {
        inputTime[i] = tau;
        negLogRand = -log(hiprand_uniform_double(state));
        dTau = negLogRand/rate;
        tau += dTau;
        i++;
        if (i == MAX_FFINPUT_PER_DT) {
            printf("exceeding max input per dt %i\n", MAX_FFINPUT_PER_DT);
            //printf("rate = %f, lastNegLogRand = %f, leftTimeRate = %f \n", rate, *lastNegLogRand, *leftTimeRate);
            //printf("inputTime[0]: %f, inputTime[1]: %f\n", inputTime[0], inputTime[1]);
            break;
        }
    } while (tau <= dt);
    *lastNegLogRand = negLogRand;
    *leftTimeRate = (dt - tau + dTau) * rate;
    return i;
}

__host__ __device__ void evolve_g(ConductanceShape &cond,
                                  double* __restrict__ g, 
                                  double* __restrict__ h, 
                                  double* __restrict__ f,
                                  double inputTime[],
                                  int nInput, double dt, unsigned int ig)
{
    cond.decay_conductance(g, h, dt, ig); 
    for (int i=0; i<nInput; i++) {
        cond.compute_single_input_conductance(g, h, *f, dt-inputTime[i], ig);
    }
}

__device__  double step(LIF* lif, double dt, double tRef, unsigned int id, double gE, double gI, double tsp[]) {
    lif->tsp = dt;
    lif->spikeCount = 0;
    // not in refractory period
    if (lif->tBack < dt) {
        // return from refractory period
        if (lif->tBack > 0.0f) {
            lif->recompute_v0(dt);
        }
        lif->implicit_rk2(dt);
        while (lif->v > vT && lif->tBack < dt) {
            // crossed threshold
            lif->compute_spike_time(dt); 
            tsp[lif->spikeCount] = lif->tsp;
            lif->spikeCount++;
            if (lif->spikeCount == MAX_SPIKE_PER_DT) {
                printf("increase MAX_SPIKE_PER_DT or decrease dt\n");
                assert(lif->spikeCount < MAX_SPIKE_PER_DT);
            }
            lif->tBack = lif->tsp + tRef;
            if (lif->tBack < dt) {
                // refractory period ended during dt
                lif->recompute(dt);
            }
        }
    } 
    if (lif->tBack >= dt) {
        // during refractory period
        lif->reset_v();
    }
    lif->tBack -= dt;
#ifdef DEBUG
    if (lif->spikeCount > 1) {
        printf("#%i spiked %i in one time step %f, refractory period = %f ms, only the last tsp is recorded\n", id, lif->spikeCount, dt, tRef);
    }
#endif
    if (lif->v < vI) {
#ifdef DEBUG
		printf("#%i implicit rk2 is A-Stable! something is off gE1 = %f, gI1 = %f, v = %f, v0 = %f, a0 = %f, b0 = %f, a1 = %f, b1 = %f\n", id, gE, gI, lif->v, lif->v0, lif->a0, lif->b0, lif->a1, lif->b1);
#endif
        lif->v = vI;
    }   
    return lif->tsp;
}

__device__ void LIF::implicit_rk2(double dt) {
    v = impl_rk2(dt, a0, b0, a1, b1, v0);
}

__device__ void LIF::compute_spike_time(double dt, double t0) {
    tsp = comp_spike_time(v, v0, dt, t0);
}

__device__ void LIF::recompute(double dt, double t0) {
    double rB = dt/(tBack-t0) - 1; 
    double denorm = 2 + a1*dt;
    double A = (2 - a0*dt)/denorm;
    double B = (b0 + b1)*dt/denorm;
    v0 = recomp_v0(A, B, rB);
    v = A*v0 + B;
}

__device__ void LIF::recompute_v(double dt, double t0) {
    double rB = dt/(tBack-t0) - 1; 
    double denorm = 2 + a1*dt;
    double A = (2 - a0*dt)/denorm;
    double B = (b0 + b1)*dt/denorm;
    v = recomp_v(A, B, rB);
}

__device__ void LIF::recompute_v0(double dt, double t0) {
    double rB = dt/(tBack-t0) - 1; 
    double denorm = 2 + a1*dt;
    double A = (2 - a0*dt)/denorm;
    double B = (b0 + b1)*dt/denorm;
    v0 = recomp_v0(A, B, rB);
}

__device__ void LIF::set_p0(double gE, double gI, double gL) {
    a0 = get_a(gE, gI, gL);
    b0 = get_b(gE, gI, gL); 
}

__device__ void LIF::set_p1(double gE, double gI, double gL) {
    a1 = get_a(gE, gI, gL);
    b1 = get_b(gE, gI, gL); 
}

__device__ void LIF::reset_v() {
    v = vL;
}

__global__ void compute_V(double* __restrict__ v,
                          double* __restrict__ gE,
                          double* __restrict__ gI,
                          double* __restrict__ hE,
                          double* __restrict__ hI,
                          double* __restrict__ a,
                          double* __restrict__ b,
                          double* __restrict__ preMat,
                          double* __restrict__ inputRateE,
                          double* __restrict__ inputRateI,
                          int* __restrict__ eventRateE,
                          int* __restrict__ eventRateI,
                          double* __restrict__ spikeTrain,
                          unsigned int* __restrict__ nSpike,
                          double* __restrict__ tBack,
                          double* __restrict__ gactVec,
                          double* __restrict__ hactVec,
                          double* __restrict__ fE,
                          double* __restrict__ fI,
                          double* __restrict__ leftTimeRateE,
                          double* __restrict__ leftTimeRateI,
                          double* __restrict__ lastNegLogRandE,
                          double* __restrict__ lastNegLogRandI,
                          hiprandStateMRG32k3a* __restrict__ stateE,
                          hiprandStateMRG32k3a* __restrict__ stateI,
                          unsigned int ngTypeE, unsigned int ngTypeI, unsigned int ngType, ConductanceShape condE, ConductanceShape condI, double dt, unsigned int networkSize, unsigned int nE, unsigned long long seed, double dInputE, double dInputI)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    LIF lif(v[id], tBack[id]);
    double gL, tRef;
    if (id < nE) {
        tRef = tRef_E;
        gL = gL_E;
    } else {
        tRef = tRef_I;
        gL = gL_I;
    }
    /* set a0 b0 for the first step */
    double gI_t;
    double gE_t;
    // init cond E 
    gE_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeE; ig++) {
        gE_t += gE[networkSize*ig + id];
    }
    //  cond I 
    gI_t = 0.0f;
    #pragma unroll
    for (unsigned int ig=0; ig<ngTypeI; ig++) {
        gI_t += gI[networkSize*ig + id];
    }
    lif.set_p0(gE_t, gI_t, gL);
    /* Get feedforward input */
    // consider use shared memory for dynamic allocation
    double inputTimeE[MAX_FFINPUT_PER_DT];
    double inputTimeI[MAX_FFINPUT_PER_DT];
    hiprandStateMRG32k3a localStateE = stateE[id];
    hiprandStateMRG32k3a localStateI = stateI[id];
    int nInputE, nInputI;
    #ifdef TEST_WITH_MANUAL_FFINPUT
        nInputE = 0;
        if (leftTimeRateE[id] < dt) {
            inputTimeE[nInputE] = leftTimeRateE[id];
            nInputE++;
            double tmp = leftTimeRateE[id] + dInputE;
            while (tmp < dt){
                inputTimeE[nInputE] = tmp;
                nInputE++;
                tmp += dInputE;
            }
            leftTimeRateE[id] = tmp - dt;
        } else {
            leftTimeRateE[id] -= dt;
        }

        nInputI = 0;
        if (leftTimeRateI[id] < dt) {
            inputTimeI[nInputI] = leftTimeRateI[id];
            nInputI++;
            double tmp = leftTimeRateI[id] + dInputI;
            while (tmp < dt){
                inputTimeI[nInputI] = tmp;
                nInputI++;
                tmp += dInputI;
            }
            leftTimeRateI[id] = tmp - dt;
        } else {
            leftTimeRateI[id] -= dt;
        }
    #else
        nInputE = set_input_time(inputTimeE, dt, inputRateE[id], &(leftTimeRateE[id]), &(lastNegLogRandE[id]), &localStateE);
        nInputI = set_input_time(inputTimeI, dt, inputRateI[id], &(leftTimeRateI[id]), &(lastNegLogRandI[id]), &localStateI);
    #endif
    //__syncwarp();
    // return a realization of Poisson input rate
    eventRateE[id] = nInputE;
    eventRateI[id] = nInputI;
    // update rng state 
    stateE[id] = localStateE;
    stateI[id] = localStateI;
    /* evolve g to t+dt with ff input only */
    unsigned int gid;
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gid = networkSize*ig + id;
        double g_i = gE[gid];
        double h_i = hE[gid];
        double f_i = fE[gid];
        evolve_g(condE, &g_i, &h_i, &f_i, inputTimeE, nInputE, dt, ig);
        //__syncwarp();
        gE_t += g_i;
        gE[gid] = g_i;
        hE[gid] = h_i;
        // for learning
        //fE[gid] = f_i;
    }
    gI_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gid = networkSize*ig + id;
        double g_i = gI[gid];
        double h_i = hI[gid];
        double f_i = fI[gid];
        evolve_g(condI, &g_i, &h_i, &f_i, inputTimeI, nInputI, dt, ig);
        //__syncwarp();
        gI_t += g_i;
        gI[gid] = g_i;
        hI[gid] = h_i;
        // for learning
        //fI[gid] = f_i;
    }
    lif.set_p1(gE_t, gI_t, gL);
    // rk2 step
    double tsp[MAX_SPIKE_PER_DT];
    spikeTrain[id] = step(&lif, dt, tRef, /*the last 2 args are for deugging*/ id, gE_t, gI_t, tsp);
    nSpike[id] = lif.spikeCount;
    if (lif.v < vI) {
#ifdef DEBUG
		printf("#%i something is off gE = %f, gI = %f, v = %f\n", id, gE_t, gI_t, lif.v);
#endif
        lif.v = vI;
    }   
	v[id] = lif.v;
    tBack[id] = lif.tBack;

    //setup acting vectors
    double g_end, h_end;
    if (lif.spikeCount > 0) {
        int ngType;
        ConductanceShape *cond; 
        if (id < nE) {
            ngType = ngTypeE;
            cond = &condE;
        } else {
            ngType = ngTypeI;
            cond = &condI;
        }
        #pragma unroll
        for (int ig=0; ig<ngType; ig++) {
            gid = networkSize*ig+id;
            gactVec[gid] = 0.0f;
            hactVec[gid] = 0.0f;
            for (int i=0; i<lif.spikeCount; i++) {
                g_end = 0.0f;
                h_end = 0.0f;
                cond->compute_single_input_conductance(&g_end, &h_end, 1.0f, dt-tsp[i], ig);
                gactVec[gid] += g_end;
                hactVec[gid] += h_end;
            }
        }
    } else {
        for (int ig=0; ig<ngType; ig++) {
            gid = networkSize*ig + id;
            gactVec[gid] = 0.0f;
            hactVec[gid] = 0.0f;
        }
    }
}
