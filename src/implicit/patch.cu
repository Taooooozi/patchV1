#include "hip/hip_runtime.h"
#include "patch.h"

int main(int argc, char *argv[])
{
	#pragma float_control( except, on )
    #ifndef SKIP_IO
        std::ofstream p_file, v_file, spike_file, nSpike_file, gE_file, gI_file;
    #endif
    float time;
    hiprandStateMRG32k3a *stateE, *randState;
    hiprandStateMRG32k3a *stateI;
    unsigned long long seed;
    //seed = 183765712;
    seed = std::time(0);
    int device;
    int b1,b2;
    b1 = 160;
    b2 = 128;
    bool printStep = false;
    bool moreSharedMemThanBlocks = true;
    double flatRateE = 100.0f; // Hz
    double flatRateI0 = 4;
    double t = 1.0f;
    unsigned int nstep = 200;
    double EffsE = 1e-1;
    double IffsE0 = 0.0;
    double sEE0 = 0.0;
    double sIE0 = 0.0;
    double sEI0 = 0.0;
    double sII0 = 0.0;
    double EffsI0 = 0.7;
    double IffsI0 = 0.7;
	char tmp[101];
    /* Overwrite parameters */
    for (int i = 0; i<argc; i++) {
        printf(argv[i]);
        printf(" ");
    }
    printf("\n");
    if (argc == 2) {
        sscanf(argv[argc-1],"%u",&seed);
    }
    if (argc == 3) {
        sscanf(argv[argc-1],"%d",&b2); 
        sscanf(argv[argc-2],"%d",&b1); 
    }
    if (argc == 4) {
        sscanf(argv[argc-1],"%d",&b2); 
        sscanf(argv[argc-2],"%d",&b1); 
        sscanf(argv[argc-3],"%d",&nstep);
    }
    if (argc == 5) {
        sscanf(argv[argc-1],"%u",&seed);
        sscanf(argv[argc-2],"%d",&b2);
        sscanf(argv[argc-3],"%d",&b1);
        sscanf(argv[argc-4],"%d",&nstep);
    }
	if (argc == 6) {
		sscanf(argv[argc-1], "%lf", &flatRateE);
		sscanf(argv[argc-2], "%u", &seed);
		sscanf(argv[argc-3], "%d", &b2);
		sscanf(argv[argc-4], "%d", &b1);
		sscanf(argv[argc-5], "%d", &nstep);
	}
	if (argc == 15) {
		sscanf(argv[argc-1], "%100s", tmp);
		sscanf(argv[argc-2], "%lf", &t);
		sscanf(argv[argc-3], "%lf", &flatRateI0);
		sscanf(argv[argc-4], "%lf", &flatRateE);
		sscanf(argv[argc-5], "%u", &seed);
		sscanf(argv[argc-6], "%d", &b2);
		sscanf(argv[argc-7], "%d", &b1);
		sscanf(argv[argc-8], "%lf", &sII0);
		sscanf(argv[argc-9], "%lf", &sEI0);
		sscanf(argv[argc-10], "%lf", &sIE0);
		sscanf(argv[argc-11], "%lf", &sEE0);
		sscanf(argv[argc-12], "%lf", &IffsE0);
		sscanf(argv[argc-13], "%lf", &EffsE);
		sscanf(argv[argc-14], "%d", &nstep);
	}

	std::string theme = tmp;
	std::cout << "theme = " << theme << "\n";
    if (!theme.empty()) {
        theme = '-'+theme;
    }
    printf("%i x %i, %i steps, seed = %u\n", b1, b2, nstep, seed);
	unsigned int networkSize = b1*b2;
    if (networkSize/10.0 != float(networkSize/10)) {
        printf("To have higher computation occupancy make a factor of 10 in networkSize\n");
    }
	int init_b2 = warpSize;
	int init_b1 = networkSize / init_b2;
    double eiRatio = 3.0f/4.0f;
    unsigned int nE = networkSize*eiRatio;
    unsigned int nI = networkSize-nE;
    double sEE = sEE0*EffsE/nE;
    double sIE = sIE0*EffsE/nE;
    double sEI = sEI0*EffsE/nI;
    double sII = sII0*EffsE/nI;
    double IffsE = IffsE0*EffsE;
    printf("EffsE = %e, IffsE = %e\n", EffsE, IffsE);
    double EffsI = EffsI0*EffsE;
    double IffsI = IffsI0*EffsE;
    printf("EffsI = %e, IffsI = %e\n", EffsI, IffsI);
    printf("sEE = %e\n", sEE);
    printf("sIE = %e\n", sIE);
    printf("sEI = %e\n", sEI);
    printf("sII = %e\n", sII);
    double dt0 = t/static_cast<double>(nstep); // ms
    printf("dt0 = %.16e\n",dt0);
    double dt=1;
    if (dt0 > 1) {
        double next_dt = static_cast<int>(dt) << 1;
        while (next_dt < dt0) {
            next_dt = static_cast<int>(next_dt) << 1;
            dt = static_cast<int>(dt) << 1;
        }
        if (next_dt-dt0 < dt0-dt) {
            dt = next_dt;
        }
    } else {
        double next_dt = dt/2;
        while (next_dt > dt0) {
            next_dt /= 2;
            dt /= 2;
        }
        if (dt0-next_dt < dt-dt0) {
            dt = next_dt;
        }
    }
    t = dt * nstep;
    /* to be extended */
    bool presetInit = false;
    double riseTimeE[2] = {1.0f, 5.0f}; // ms
    double riseTimeI[1] = {1.0f};
    double decayTimeE[2] = {3.0f, 80.0f};
    double decayTimeI[1] = {5.0f};

    ConductanceShape condE(riseTimeE, decayTimeE, ngTypeE);
    ConductanceShape condI(riseTimeI, decayTimeI, ngTypeI);
    double flatRateI = flatRateI0 * flatRateE;
    printf("designated input rateE = %3.1fHz\n", flatRateE);
    printf("designated input rateI = %3.1fHz\n", flatRateI);
	printf("dt = %f ms\n", dt);
    printf("nE = %i, nI = %i\n", nE, networkSize-nE);
    printf("t = %.16e x %i = %.16e\n", dt, nstep, t);
	double dInputE = 1000.0f/flatRateE;
	double dInputI = 1000.0f/flatRateI;
    if (dt/dInputE > MAX_FFINPUT_PER_DT) {
        printf("increase MAX_FFINPUT_PER_DT, or decrease input rate E.\n");
        return EXIT_FAILURE;
    }
    if (dt/dInputI > MAX_FFINPUT_PER_DT) {
        printf("increase MAX_FFINPUT_PER_DT, or decrease input rate I.\n");
        return EXIT_FAILURE;
    }
	if (networkSize / float(warpSize) != float(networkSize / warpSize)) {
		printf("please make networkSize multiples of %i to run on GPU\n", warpSize);
		return EXIT_FAILURE;
	}
    struct hipDeviceProp_t properties;  
    double *v, *gE, *gI, *preMat; 
    int *eventRateE, *d_eventRateE;
    int *eventRateI, *d_eventRateI;
    double *d_v, *d_gE, *d_gI, *d_hE, *d_hI, *d_fE, *d_fI, *d_preMat, *d_inputRateE, *d_inputRateI;
	#if SCHEME == 2
		double *d_dVs;
	#endif
    double *leftTimeRateE, *lastNegLogRandE;
    double *leftTimeRateI, *lastNegLogRandI;
    double *spikeTrain, *d_spikeTrain, *tBack;
    unsigned int *nSpike, *d_nSpike;

    while (init_b2 < 256 && init_b1 > 1) {
        init_b2 = init_b2*2;
        init_b1 = init_b1/2;
    }
    printf("init size %i, %i\n", init_b1, init_b2);

	/* check for double precision support */
	CUDA_CALL(hipGetDevice(&device));
	CUDA_CALL(hipGetDeviceProperties(&properties, device));
	if (!(properties.major >= 2 || (properties.major == 1 && properties.minor >= 3))) {
		printf(" double precision not supported\n");
		return EXIT_FAILURE;
	}
    /* inits that used by both cpu and gpu */
    CUDA_CALL(hipMalloc((void **)&leftTimeRateE,   networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&leftTimeRateI,   networkSize * sizeof(double)));
    /* Allocate space for rng on device */
    CUDA_CALL(hipMalloc((void **)&stateE,          networkSize * sizeof(hiprandStateMRG32k3a)));
    CUDA_CALL(hipMalloc((void **)&stateI,          networkSize * sizeof(hiprandStateMRG32k3a)));
    CUDA_CALL(hipMalloc((void **)&lastNegLogRandE, networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&lastNegLogRandI, networkSize * sizeof(double)));

    CUDA_CALL(hipMalloc((void **)&randState, networkSize * sizeof(hiprandStateMRG32k3a)));

    preMat = new double[networkSize * networkSize];
    CUDA_CALL(hipMalloc((void **)&d_preMat, networkSize * networkSize * sizeof(double)));

    CUDA_CALL(hipHostMalloc((void**)&v, networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_v,           networkSize * sizeof(double)));

    logRand_init<<<init_b1,init_b2>>>(lastNegLogRandE, stateE, seed, leftTimeRateE, dInputE, 0);
    CUDA_CHECK();
    logRand_init<<<init_b1,init_b2>>>(lastNegLogRandI, stateI, seed-networkSize, leftTimeRateI, dInputI, networkSize);
    CUDA_CHECK();

    randInit<<<init_b1,init_b2>>>(d_preMat, d_v, randState, sEE, sIE, sEI, sII, networkSize, nE, seed);
    CUDA_CHECK();
    CUDA_CALL(hipMemcpy(preMat, d_preMat, networkSize*networkSize*sizeof(double),hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(v, d_v, networkSize*sizeof(double),hipMemcpyDeviceToHost));
    printf("storage size of preMat %.1fMb\n", float(networkSize*networkSize*sizeof(double))/1024.0/1024.0);

    unsigned int nbatch, batchEnd, batchStep;
    // v, gE, gI, spikeTrain
    unsigned int hostMemToDiskPerStep = ceil(networkSize * (sizeof(double) + ngTypeE*sizeof(double) + ngTypeI*sizeof(double) + sizeof(int) )/(1024*1024));
    //batchStep = floor(HALF_MEMORY_OCCUPANCY/hostMemToDiskPerStep);
    batchStep = 1;
    if (batchStep < 10) {
        printf("consider increase HALF_MEMORY_OCCUPANCY, batch step = %i\n", batchStep);
    } else {
        if (batchStep == 0) {
            printf("increase HALF_MEMORY_OCCUPANCY, memory to write on disk per step: %i Mb", hostMemToDiskPerStep);
            return EXIT_FAILURE;
        }
    }
    nbatch = nstep/batchStep; 
    batchEnd = nstep - batchStep*nbatch;
    int alt = 1;
    hipEvent_t iStart, iStop;
    hipEventCreate(&iStart);
    hipEventCreate(&iStop);
    CUDA_CALL(hipEventRecord(iStart, 0));
    /* Allocate space for results on host */
    //pinned memory
    CUDA_CALL(hipHostMalloc((void**)&gE,          networkSize * ngTypeE * sizeof(double)));
    CUDA_CALL(hipHostMalloc((void**)&gI,          networkSize * ngTypeI *sizeof(double)));
    CUDA_CALL(hipHostMalloc((void**)&spikeTrain,  networkSize * sizeof(double)));
    CUDA_CALL(hipHostMalloc((void**)&nSpike,      networkSize * sizeof(unsigned int)));
    CUDA_CALL(hipHostMalloc((void**)&eventRateE,   networkSize * sizeof(int)));
    CUDA_CALL(hipHostMalloc((void**)&eventRateI,   networkSize * sizeof(int)));

    /* Allocate space for results on device */
	CUDA_CALL(hipMalloc((void **)&d_dVs,		 networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_gE,         networkSize * ngTypeE * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_gI,         networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_hE,         networkSize * ngTypeE * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_hI,         networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_fE,         networkSize * ngTypeE * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_fI,         networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_inputRateE, networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_inputRateI, networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_eventRateE, networkSize * sizeof(int)));
    CUDA_CALL(hipMalloc((void **)&d_eventRateI, networkSize * sizeof(int)));
    CUDA_CALL(hipMalloc((void **)&d_spikeTrain, networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_nSpike,     networkSize * sizeof(unsigned int)));
    CUDA_CALL(hipMalloc((void **)&tBack,        networkSize * sizeof(double)));
    
    /* Create CUDA events */
    hipEvent_t start, stop, gReadyE, gReadyI, vReady, vComputed, spikeRateReady, nSpikeReady, eventRateEReady, eventRateIReady;
    hipEvent_t kStart, kStop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&gReadyE);
    hipEventCreate(&gReadyI);
    hipEventCreate(&vReady);
    hipEventCreate(&kStart);
    hipEventCreate(&kStop);
    hipEventCreate(&vComputed);
    hipEventCreate(&spikeRateReady);
	hipEventCreate(&nSpikeReady);
#ifndef FULL_SPEED
    hipEventCreate(&eventRateEReady);
	hipEventCreate(&eventRateIReady);
#endif
    /* Initialize device arrays */
    // CUDA streams for init
    hipStream_t i1, i2, i3;
    CUDA_CALL(hipStreamCreate(&i1));
    CUDA_CALL(hipStreamCreate(&i2));
    CUDA_CALL(hipStreamCreate(&i3));
    if (!presetInit) {
        for (unsigned int i=0; i<networkSize; i++) {
            for (unsigned int ig=0; ig<ngTypeE; ig++) {
                gE[ig*networkSize + i] = 0.0f;
            }
            for (unsigned int ig=0; ig<ngTypeI; ig++) {
                gI[ig*networkSize + i] = 0.0f;
            }
        }
        // init rand generation for poisson
        init<double><<<init_b1,init_b2,0,i1>>>(d_inputRateE, flatRateE/1000.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i2>>>(d_inputRateI, flatRateI/1000.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i3>>>(tBack, -1.0f); 
        CUDA_CHECK();
		init<double><<<init_b1, init_b2, 0, i1>>>(d_dVs, 0.0f);
		CUDA_CHECK();
        f_init<<<init_b1,init_b2,0,i2>>>(d_fE, networkSize, nE, ngTypeE, EffsE, IffsE);
        CUDA_CHECK();
        f_init<<<init_b1,init_b2,0,i3>>>(d_fI, networkSize, nE, ngTypeI, EffsI, IffsI);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i2>>>(d_gE, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i3>>>(d_gI, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i2>>>(d_hE, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i3>>>(d_hI, 0.0f);
        CUDA_CHECK();

        //CUDA_CALL(hipEventRecord(kStart, 0));
        //CUDA_CALL(hipEventRecord(kStop, 0));
        //CUDA_CALL(hipEventSynchronize(kStop));
        //CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
        //printf("logRand_init<<<%ix%i>>> cost %.1fms\n", init_b1*init_b1*init_b2, init_b2, time);
    }
    CUDA_CALL(hipStreamDestroy(i1));
    CUDA_CALL(hipStreamDestroy(i2));
    CUDA_CALL(hipStreamDestroy(i3));
    CUDA_CALL(hipEventRecord(iStop, 0));
    CUDA_CALL(hipEventSynchronize(iStop));
    CUDA_CALL(hipEventElapsedTime(&time, iStart, iStop));
    printf("initialization cost %fms\n", time);

    /* Create CUDA streams */
    hipStream_t s1, s2, s3, s4, s5, s6, s7;
    CUDA_CALL(hipStreamCreate(&s1));
    CUDA_CALL(hipStreamCreate(&s2));
    CUDA_CALL(hipStreamCreate(&s3));
    CUDA_CALL(hipStreamCreate(&s4));
    CUDA_CALL(hipStreamCreate(&s5));
    CUDA_CALL(hipStreamCreate(&s6));
    CUDA_CALL(hipStreamCreate(&s7));
    #ifndef SKIP_IO
        #ifdef SPIKE_CORRECTION
            p_file.open("p_ushy" + theme + "_ssc.bin", std::ios::out|std::ios::binary);
            v_file.open("v_ictorious" + theme + "_ssc.bin", std::ios::out|std::ios::binary);
            spike_file.open("s_uspicious" + theme + "_ssc.bin", std::ios::out|std::ios::binary);
            nSpike_file.open("n_arcotic" + theme + "_ssc.bin", std::ios::out|std::ios::binary);
            gE_file.open("gE_nerous" + theme + "_ssc.bin", std::ios::out|std::ios::binary);
            gI_file.open("gI_berish" + theme + "_ssc.bin", std::ios::out|std::ios::binary);
        #else
            p_file.open("p_ushy" + theme + ".bin", std::ios::out|std::ios::binary);
            v_file.open("v_ictorious" + theme + ".bin", std::ios::out|std::ios::binary);
            spike_file.open("s_uspicious" + theme + ".bin", std::ios::out|std::ios::binary);
            nSpike_file.open("n_arcotic" + theme + ".bin", std::ios::out|std::ios::binary);
            gE_file.open("gE_nerous" + theme + ".bin", std::ios::out|std::ios::binary);
            gI_file.open("gI_berish" + theme + ".bin", std::ios::out|std::ios::binary);
        #endif
    
        p_file.write((char*)&nE, sizeof(unsigned int));
        p_file.write((char*)&nI, sizeof(unsigned int));
    	unsigned int u_ngTypeE = ngTypeE;
    	unsigned int u_ngTypeI = ngTypeI;
        p_file.write((char*)&u_ngTypeE, sizeof(unsigned int));
        p_file.write((char*)&u_ngTypeI, sizeof(unsigned int));
        double dtmp = vL;
        p_file.write((char*)&dtmp, sizeof(double));
        dtmp = vT;
        p_file.write((char*)&dtmp, sizeof(double));
        dtmp = vE;
        p_file.write((char*)&dtmp, sizeof(double));
        dtmp = vI;
        p_file.write((char*)&dtmp, sizeof(double));
        dtmp = gL_E;
        p_file.write((char*)&dtmp, sizeof(double));
        dtmp = gL_I;
        p_file.write((char*)&dtmp, sizeof(double));
        dtmp = tRef_E;
        p_file.write((char*)&dtmp, sizeof(double));
        dtmp = tRef_I;
        p_file.write((char*)&tmp, sizeof(double));
        p_file.write((char*)&nstep, sizeof(unsigned int));
        p_file.write((char*)&dt, sizeof(double));
        p_file.write((char*)&flatRateE, sizeof(double));
    #endif

    CUDA_CALL(hipEventRecord(start, 0));
    double eventsE = 0.0f;
    double eventsI = 0.0f;
    unsigned int spikesE = 0;
    unsigned int spikesI = 0;
    unsigned int ibatch = 0;
    unsigned int bStep = 0;
    unsigned int batchOffset = 0;
    unsigned int copySize = batchStep;
    unsigned int n = networkSize*copySize;
    CUDA_CALL(hipEventRecord(gReadyE, 0));
    CUDA_CALL(hipEventRecord(gReadyI, 0));
    CUDA_CALL(hipEventRecord(vReady, 0));
    
    double timeV = 0.0f;
    double timeIO = 0.0f;
    double exc_input_ratio = 0.0f;
    double gEavgE = 0.0f;
    double gIavgE = 0.0f;
    double gEavgI = 0.0f;
    double gIavgI = 0.0f;
    for (int i=0; i<nstep; i++) {
        unsigned int offset;
        offset = 0;
        /* Write voltage of last step to disk */
        CUDA_CALL(hipEventSynchronize(vReady));
        #ifndef SKIP_IO
            v_file.write((char*)v, networkSize * sizeof(double));
        #endif
        /* Write conductance of last step to disk */
        CUDA_CALL(hipEventSynchronize(gReadyE));
        #ifndef SKIP_IO
            gE_file.write((char*)gE, networkSize*ngTypeE*sizeof(double));
        #endif
        CUDA_CALL(hipEventSynchronize(gReadyI));
        #ifndef SKIP_IO
            gI_file.write((char*)gI, networkSize*ngTypeI*sizeof(double));
        #endif
        #ifndef FULL_SPEED
            for (unsigned int j=0; j<networkSize; j++) {
                if (j<nE) {
                    for (unsigned int ig=0; ig<ngTypeE; ig++) {
                        gEavgE += gE[ig*networkSize + j];
                    }
                    for (unsigned int ig=0; ig<ngTypeI; ig++) {
                        gIavgE += gI[ig*networkSize + j];
                    }
                } else {
                    for (unsigned int ig=0; ig<ngTypeE; ig++) {
                        gEavgI += gE[ig*networkSize + j];
                    }
                    for (unsigned int ig=0; ig<ngTypeI; ig++) {
                        gIavgI += gI[ig*networkSize + j];
                    }
                }
            }
        #endif
        /* Compute voltage */
        #ifdef KERNEL_PERFORMANCE
            CUDA_CALL(hipEventRecord(kStart, 0));
        #endif
        dim3 grid3(1);
        dim3 block3(1024);
        #ifdef SPIKE_CORRECTION
            unsigned int shared_mem = 1024*sizeof(double)+2*1024*sizeof(unsigned int);
			#if SCHEME < 2
				compute_V<<<grid3, block3, shared_mem, s1>>>(d_v, d_gE, d_gI, d_hE, d_hI, d_preMat, d_inputRateE, d_inputRateI, d_eventRateE, d_eventRateI, d_spikeTrain, d_nSpike, tBack, d_fE, d_fI, leftTimeRateE, leftTimeRateI, lastNegLogRandE, lastNegLogRandI, stateE, stateI, condE, condI, dt, networkSize, nE, seed, dInputE, dInputI, i*dt);
			#else
				compute_V<<<grid3, block3, shared_mem, s1>>>(d_v, d_dVs, d_gE, d_gI, d_hE, d_hI, d_preMat, d_inputRateE, d_inputRateI, d_eventRateE, d_eventRateI, d_spikeTrain, d_nSpike, tBack, d_fE, d_fI, leftTimeRateE, leftTimeRateI, lastNegLogRandE, lastNegLogRandI, stateE, stateI, condE, condI, dt, networkSize, nE, seed, dInputE, dInputI, i*dt);
			#endif
        #else
            unsigned int shared_mem = 1024*sizeof(double)+1024*sizeof(double);
			#if SCHEME < 2
                compute_V_without_ssc<<<grid3, block3, shared_mem, s1>>>(d_v, d_gE, d_gI, d_hE, d_hI, d_preMat, d_inputRateE, d_inputRateI, d_eventRateE, d_eventRateI, d_spikeTrain, d_nSpike, tBack, d_fE, d_fI, leftTimeRateE, leftTimeRateI, lastNegLogRandE, lastNegLogRandI, stateE, stateI, condE, condI, dt, networkSize, nE, seed, dInputE, dInputI, i*dt);
            #else
                compute_V_without_ssc<<<grid3, block3, shared_mem, s1>>>(d_v, d_dVs, d_gE, d_gI, d_hE, d_hI, d_preMat, d_inputRateE, d_inputRateI, d_eventRateE, d_eventRateI, d_spikeTrain, d_nSpike, tBack, d_fE, d_fI, leftTimeRateE, leftTimeRateI, lastNegLogRandE, lastNegLogRandI, stateE, stateI, condE, condI, dt, networkSize, nE, seed, dInputE, dInputI, i*dt);
            #endif
        #endif
        CUDA_CHECK();
        CUDA_CALL(hipEventRecord(vComputed, s1));
        #ifdef KERNEL_PERFORMANCE
            CUDA_CALL(hipEventRecord(kStop, 0));
            CUDA_CALL(hipEventSynchronize(kStop));
            CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
            timeV += time;
        #endif
        #ifndef FULL_SPEED
		    if (printStep) {
		    	printf("A single step of compute_V cost %fms\n", time);
		    }
        #endif
		#ifdef KERNEL_PERFORMANCE
			CUDA_CALL(hipEventRecord(kStart, 0));
		#endif
		CUDA_CALL(hipEventSynchronize(vComputed));
        /* Copy spikeTrain to host */
        CUDA_CALL(hipMemcpyAsync(spikeTrain, d_spikeTrain, networkSize * sizeof(double), hipMemcpyDeviceToHost, s1));
		CUDA_CALL(hipEventRecord(spikeRateReady, s1));
		CUDA_CALL(hipMemcpyAsync(nSpike, d_nSpike, networkSize * sizeof(unsigned int), hipMemcpyDeviceToHost, s2));
		CUDA_CALL(hipEventRecord(nSpikeReady, s2));
        /* Copy input events to host */
        #ifndef FULL_SPEED
            CUDA_CALL(hipMemcpyAsync(eventRateE, d_eventRateE, networkSize * sizeof(int), hipMemcpyDeviceToHost, s3));
		    CUDA_CALL(hipEventRecord(eventRateEReady, s3));
            CUDA_CALL(hipMemcpyAsync(eventRateI, d_eventRateI, networkSize * sizeof(int), hipMemcpyDeviceToHost, s4));
		    CUDA_CALL(hipEventRecord(eventRateIReady, s4));
        #endif
		/* Copy voltage to host */
		CUDA_CALL(hipMemcpyAsync(v, d_v, networkSize * sizeof(double), hipMemcpyDeviceToHost, s5));
		CUDA_CALL(hipEventRecord(vReady, s5));
        // copy exc conductance to host
        CUDA_CALL(hipMemcpyAsync(gE, d_gE, networkSize * ngTypeE * sizeof(double), hipMemcpyDeviceToHost, s6));
		CUDA_CALL(hipEventRecord(gReadyE, s6));
        // copy inh conductance to host
        CUDA_CALL(hipMemcpyAsync(gI, d_gI, networkSize * ngTypeI * sizeof(double), hipMemcpyDeviceToHost, s7));
		CUDA_CALL(hipEventRecord(gReadyI, s7));

        CUDA_CALL(hipEventSynchronize(spikeRateReady));
        /* Write spikeTrain of current step to disk */
        #ifndef SKIP_IO
            spike_file.write((char*)spikeTrain,  networkSize*sizeof(double));
        #endif
		CUDA_CALL(hipEventSynchronize(nSpikeReady));
        #ifndef SKIP_IO
            nSpike_file.write((char*)nSpike,     networkSize*sizeof(unsigned int));
        #endif

        #ifndef FULL_SPEED
            CUDA_CALL(hipEventSynchronize(eventRateEReady));
		    CUDA_CALL(hipEventSynchronize(eventRateIReady));
            #ifndef DEBUG
                printf("\r stepping: %3.1f%%", 100.0f*float(i+1)/nstep);
            #else
                printf("stepping: %3.1f%%, t = %f \n", 100.0f*float(i+1)/nstep, (i+1)*dt);
            #endif
            double _eventsE = 0.0f;
            double _eventsI = 0.0f;
            unsigned int _spikes = 0;
            double sEi = 0.0f;
            for (int j=0; j<networkSize; j++) {
                _eventsE += eventRateE[j];
                _eventsI += eventRateI[j];
                _spikes += nSpike[j];
                if (j<nE) {
                    spikesE += nSpike[j];
                } else {
                    spikesI += nSpike[j];
                }
                for (unsigned int k=0; k<nE; k++) {
                     sEi += preMat[k*networkSize + j] * nSpike[k];
                }
            }
            eventsE += _eventsE;
            eventsI += _eventsI;
            exc_input_ratio += sEi/networkSize;
            if (printStep) {
                printf("instant exc input rate = %fkHz, dt = %f, networkSize = %i\n", _eventsE/(dt*networkSize), dt, networkSize);
                printf("instant inh input rate = %fkHz, dt = %f, networkSize = %i\n", _eventsI/(dt*networkSize), dt, networkSize);
                printf("instant firing rate = %fHz\n", _spikes/(dt*networkSize)*1000.0);
            }
        #endif

		#ifdef KERNEL_PERFORMANCE
			CUDA_CALL(hipEventRecord(kStop, 0));
			CUDA_CALL(hipEventSynchronize(kStop));
			CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
			if (printStep) {
				printf("Memcpy and Disk IO cost %fms\n", time);
			}
			timeIO += time;
		#endif
    }
    /* WHen hit HALF_MEMORY_OCCUPANCY, write half of the array to disk, the other half left to receive from device */

    #ifndef SKIP_IO
        v_file.write((char*)v, networkSize * sizeof(double));
        gE_file.write((char*)gE, networkSize * ngTypeE * sizeof(double));
        gI_file.write((char*)gI, networkSize * ngTypeI * sizeof(double));
    #endif
    printf("\n");
    #ifndef FULL_SPEED
        printf("flatRateE = %fHz, realized mean input rate = %fHz\n", flatRateE, 1000.0*float(eventsE)/(dt*nstep*networkSize));
        printf("flatRateI = %fHz, realized mean input rate = %fHz\n", flatRateI, 1000.0*float(eventsI)/(dt*nstep*networkSize));
        printf("exc firing rate = %eHz\n", float(spikesE)/(dt*nstep*nE)*1000.0);
        printf("inh firing rate = %eHz\n", float(spikesI)/(dt*nstep*nI)*1000.0);
    #endif

    CUDA_CALL(hipEventRecord(stop, 0));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipEventElapsedTime(&time, start, stop));
    printf("CUDA takes %fms, runtime/realtime ratio ms %fms\n", time, time/(dt*nstep));
    printf("compute_V takes %fms, ratio ms %fms\n", timeV, timeV/(dt*nstep));
    printf("IO takes %fms, ratio ms %fms\n", timeIO, timeIO/(dt*nstep));
    #ifndef FULL_SPEED
        printf("input ratio recurrent:feedforward = %f\n", exc_input_ratio/((EffsE*nE+IffsE*nI)/networkSize*dt*nstep/dInputE));
        printf("           exc,        inh\n");
        printf("avg gE = %e, %e\n", gEavgE/nstep/nE, gEavgI/nstep/nI);
        printf("avg gI = %e, %e\n", gIavgE/nstep/nE, gIavgI/nstep/nI);
    #endif
    int nTimer = 1;
    #ifndef SKIP_IO
        p_file.write((char*)&nTimer, sizeof(int));
        p_file.write((char*)&timeV, sizeof(double));
        //p_file.write((char*)&timeIO, sizeof(double));
    #endif

    /* Cleanup */
    printf("Cleaning up:\n");
    CUDA_CALL(hipStreamDestroy(s1));
    CUDA_CALL(hipStreamDestroy(s2));
    CUDA_CALL(hipStreamDestroy(s3));
    CUDA_CALL(hipStreamDestroy(s4));
    CUDA_CALL(hipStreamDestroy(s5));
    CUDA_CALL(hipStreamDestroy(s6));
    CUDA_CALL(hipStreamDestroy(s7));
    printf("    CUDA streams destroyed\n");
    
    #ifndef SKIP_IO
        if (p_file.is_open()) p_file.close();
        if (v_file.is_open()) v_file.close();
        if (spike_file.is_open()) spike_file.close();
        if (nSpike_file.is_open()) nSpike_file.close();
        if (gE_file.is_open()) gE_file.close();
        if (gI_file.is_open()) gI_file.close();
    #endif
    printf("    Output files closed\n");
    
    CUDA_CALL(hipFree(stateE));
    CUDA_CALL(hipFree(stateI));
    CUDA_CALL(hipFree(d_v));
	CUDA_CALL(hipFree(d_dVs));
    CUDA_CALL(hipFree(d_gE));
    CUDA_CALL(hipFree(d_gI));
    CUDA_CALL(hipFree(d_hE));
    CUDA_CALL(hipFree(d_hI));
    CUDA_CALL(hipFree(d_fE));
    CUDA_CALL(hipFree(d_fI));
    CUDA_CALL(hipFree(d_preMat));
    CUDA_CALL(hipFree(leftTimeRateE));
    CUDA_CALL(hipFree(leftTimeRateI));
    CUDA_CALL(hipFree(lastNegLogRandE));
    CUDA_CALL(hipFree(lastNegLogRandI));
    CUDA_CALL(hipFree(d_inputRateE));
    CUDA_CALL(hipFree(d_inputRateI));
    CUDA_CALL(hipFree(d_eventRateE));
    CUDA_CALL(hipFree(d_eventRateI));
    CUDA_CALL(hipFree(d_spikeTrain));
    CUDA_CALL(hipFree(d_nSpike));
    CUDA_CALL(hipFree(tBack));
    printf("    Device memory freed\n");
    CUDA_CALL(hipHostFree(v));
    CUDA_CALL(hipHostFree(gE));
    CUDA_CALL(hipHostFree(gI));
    CUDA_CALL(hipHostFree(eventRateE));
    CUDA_CALL(hipHostFree(eventRateI));
    CUDA_CALL(hipHostFree(spikeTrain));
    CUDA_CALL(hipHostFree(nSpike));
    delete []preMat;
    printf("    Host memory freed\n");
    return EXIT_SUCCESS;
}
