#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check


texture<float, hipTextureType2DLayered> L_retinaProj;
texture<float, hipTextureType2DLayered> M_retinaProj;
texture<float, hipTextureType2DLayered> S_retinaProj;

void next_layer(int ilayer, int width, int height, float* L, float* M, float* S, hipArray *dL, hipArray *dM, hipArray *dS, int nlayer = 1) { 
    hipMemcpy3DParms params = {0};
    params.srcPos = make_hipPos(0,0,0);
    params.dstPos = make_hipPos(0, 0, ilayer);
    params.extent = make_hipExtent(width, height, nlayer);
    params.kind = hipMemcpyHostToDevice;

    params.srcPtr = make_hipPitchedPtr(L, width * sizeof(float), width, height);
    params.dstArray = dL;
    checkCudaErrors(hipMemcpy3D(&params));

    params.srcPtr = make_hipPitchedPtr(M, width * sizeof(float), width, height);
    params.dstArray = dM;
    checkCudaErrors(hipMemcpy3D(&params));

    params.srcPtr = make_hipPitchedPtr(S, width * sizeof(float), width, height);
    params.dstArray = dS;
    checkCudaErrors(hipMemcpy3D(&params));
}

__global__ void plane_to_retina(/*_float* __restrict__ x,
                      _float* __restrict__ y,
                      vInput v*/ int width, int height,
                      int jlayer) 
{
    unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
    float x = static_cast<float>(ix)/width;
    float y = static_cast<float>(iy)/height;

    // x, y as on the retina
    // coord(x,y) 0,0 at fovea positive value pointing towards left and bottom (as in the image would be top and left)

    // 1,-1  0,-1  -1,-1
    // 
    // 1,0   0,0   -1,0
    // 
    // 1,1   0,1   -1,1

    // id 0,0 at top right

    if (x == 0.0 || x == 1.0) {
        float testpt = tex2DLayered(L_retinaProj, x, y, jlayer);
        printf("[%i,(%i-%f,%i-%f)]: %f\n", jlayer, x, y, testpt);
    }
    //x[id] = retina_radius*atan(v.xl + (v.xr -v.xl)*id/v.nx, distance);
    //y[id-v.nx] = retina_radius*atan(v.yb + (v.yt -v.yb)*(id-v.nx)/v.ny, distance);
}


int main(int argc, char **argv) {
    
    hipDeviceProp_t deviceProps;

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, 0));
    printf("CUDA device [%s] has %d Multi-Processors ", deviceProps.name, deviceProps.multiProcessorCount);
    printf("SM %d.%d\n", deviceProps.major, deviceProps.minor);

    if (deviceProps.major < 2)
    {
        printf("Surface requires SM >= 2.0 to support Texture Arrays.  Test will be waived... \n");
        hipDeviceReset();
        exit(EXIT_WAIVED);
    }

// set params for layerd texture
    L_retinaProj.addressMode[0] = hipAddressModeWrap;
    L_retinaProj.addressMode[1] = hipAddressModeWrap;
    L_retinaProj.filterMode = hipFilterModeLinear;
    L_retinaProj.normalized = true;  // access with normalized texture coordinates
    M_retinaProj.addressMode[0] = hipAddressModeWrap;
    M_retinaProj.addressMode[1] = hipAddressModeWrap;
    M_retinaProj.filterMode = hipFilterModeLinear;
    M_retinaProj.normalized = true;  // access with normalized texture coordinates
    S_retinaProj.addressMode[0] = hipAddressModeWrap;
    S_retinaProj.addressMode[1] = hipAddressModeWrap;
    S_retinaProj.filterMode = hipFilterModeLinear;
    S_retinaProj.normalized = true;  // access with normalized texture coordinates
    // readin plane data
    unsigned int width = 16;
    unsigned int height = 16;
    unsigned int num_layers = 16;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *timeLayeredPlaneL;
    hipArray *timeLayeredPlaneM;
    hipArray *timeLayeredPlaneS;
    checkCudaErrors(hipMalloc3DArray(&timeLayeredPlaneL, &channelDesc, make_hipExtent(width, height, num_layers), hipArrayLayered));
    checkCudaErrors(hipMalloc3DArray(&timeLayeredPlaneM, &channelDesc, make_hipExtent(width, height, num_layers), hipArrayLayered));
    checkCudaErrors(hipMalloc3DArray(&timeLayeredPlaneS, &channelDesc, make_hipExtent(width, height, num_layers), hipArrayLayered));

    // fill with initial layers
    size_t size = width*height*num_layers*sizeof(float);
    float* __restrict__ L = new float[size];
    float* __restrict__ M = new float[size];
    float* __restrict__ S = new float[size];
    for (int ilayer = 0; ilayer < num_layers; ilayer++) {
        for (int ih = 0; ih < height; ih++) {
            for (int iw = 0; iw < width; iw++) {
                if (ilayer == 0) {
                    L[width*height*ilayer + ih*width + iw] = iw/width + ih/height + ilayer;
                    M[width*height*ilayer + ih*width + iw] = 2*(iw/width + ih/height + ilayer);
                    S[width*height*ilayer + ih*width + iw] = 3*(iw/width + ih/height + ilayer);
                } else {
                    L[width*height*ilayer + ih*width + iw] = 0;
                    M[width*height*ilayer + ih*width + iw] = 0;
                    S[width*height*ilayer + ih*width + iw] = 0;
                }
            }
        }
    }
    next_layer(0, width, height, L, M, S, timeLayeredPlaneL, timeLayeredPlaneM, timeLayeredPlaneS, num_layers);

    checkCudaErrors(hipBindTextureToArray(L_retinaProj, timeLayeredPlaneL, channelDesc));
    checkCudaErrors(hipBindTextureToArray(M_retinaProj, timeLayeredPlaneM, channelDesc));
    checkCudaErrors(hipBindTextureToArray(S_retinaProj, timeLayeredPlaneS, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
    
    plane_to_retina<<< dimGrid, dimBlock, 0 >>>(width, height, 0);

    getLastCudaError("#0 Kernel execution failed");
    checkCudaErrors(hipDeviceSynchronize());

    for (unsigned int ilayer = 1; ilayer < num_layers; ilayer++) {
        for (int ih = 0; ih < height; ih++) {
            for (int iw = 0; iw < width; iw++) {
                L[ih*width + iw] = iw/width + ih/height;
                M[ih*width + iw] = 2*(iw/width + ih/height);
                S[ih*width + iw] = 3*(iw/width + ih/height);
            }
        }
        int jlayer = ilayer % num_layers;
        next_layer(jlayer, width, height, L, M, S, timeLayeredPlaneL, timeLayeredPlaneM, timeLayeredPlaneS);
        plane_to_retina<<< dimGrid, dimBlock, 0 >>>(width, height, jlayer);
        getLastCudaError("#i>0 Kernel execution failed");
    }
    checkCudaErrors(hipDeviceSynchronize());
    delete []L;
    delete []M;
    delete []S;
    checkCudaErrors(hipFreeArray(timeLayeredPlaneL));
    checkCudaErrors(hipFreeArray(timeLayeredPlaneM));
    checkCudaErrors(hipFreeArray(timeLayeredPlaneS));
    hipDeviceReset();
    return 0;
}

/*
__global__ void load(float *data, int nx, int ny) {
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    surf2Dwrite(data[y * nx + x], outputSurface, x*sizeof(float), y, hipBoundaryModeTrap);
}*/

