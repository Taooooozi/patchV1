#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>
#include <string>
#include <iostream>
#include <cassert>
#include <ctime>
#include <cmath>
#include <fenv.h>
//#include <hip/hip_runtime.h>
//#include <hiprand/hiprand_kernel.h>
//#include "hip/hip_runtime.h"

#include "connect.h"

template <typename T, typename I>
void check_statistics(T* array, I n, T &max, T &min, T &mean, T &std) {

}

int main(int argc, char *argv[])
{
    unsigned long int i = (unsigned long int) -1;
    printf("max unsigned long = %lu\n", i);
    unsigned long long seed = 7548637;
    std::ifstream pos_file;
    std::string dir(argv[1]);
    std::string theme(argv[2]);
    _float scale;
    _float max_radius;
    unsigned int nPotentialNeighbor;
	sscanf(argv[3], "%f", &scale);
	sscanf(argv[4], "%f", &max_radius);
	sscanf(argv[5], "%u", &nPotentialNeighbor);
    std::ofstream mat_file, vec_file;
    std::ofstream blockPos_file, neighborBlock_file;
    std::ofstream stats_file;
    std::ofstream posR_file;
    unsigned int nblock = 48;
    unsigned int networkSize = nblock*blockSize;
    unsigned int neighborSize = 100;
    unsigned int usingPosDim = 2;

    _float radius[NTYPE][2];
    _float neuron_type_acc_count[NTYPE+1];
	_float den_axn[NTYPE];
	_float den_den[NTYPE];
    // E
    radius[0][0] = 0.08f*scale;
    radius[0][1] = 0.15f*scale;
    // I
    radius[1][0] = 0.05f*scale;
    radius[1][1] = 0.1f*scale;

    // row <- column
    _float sTypeMat[NTYPE][NTYPE];
    sTypeMat[0][0] = 1.0f;
    sTypeMat[0][1] = 4.0f;
    sTypeMat[1][0] = 1.0f;
    sTypeMat[1][1] = 4.0f;

    //upper limit of sparsity stricted to NTYPE
    _float pTypeMat[NTYPE][NTYPE];
    pTypeMat[0][0] = 0.15f;
    pTypeMat[0][1] = 0.15f;
    pTypeMat[1][0] = 0.5f;
    pTypeMat[1][1] = 0.5f;

    // 
    unsigned int nTypeMat[NTYPE][NTYPE];
    //nTypeMat[0][0] = 400+20*5; // mean + std*5
    //nTypeMat[0][1] = 100+10*5;
    //nTypeMat[1][0] = 400+20*5;
    //nTypeMat[1][1] = 100+10*5;
    nTypeMat[0][0] = 40; // mean + std*5
    nTypeMat[0][1] = 10;
    nTypeMat[1][0] = 40;
    nTypeMat[1][1] = 10;
    
    // NTYPE
    neuron_type_acc_count[0] = 0;
    neuron_type_acc_count[1] = 768;
    neuron_type_acc_count[2] = 1024;
    assert(neuron_type_acc_count[NTYPE] == blockSize);

    den_axn[0] = 1.0;
    den_axn[1] = 2.0;

    den_den[0] = 1.0;
    den_den[1] = 2.0;

    initialize_package init_pack(radius, neuron_type_acc_count, den_axn, den_den);
    _float speedOfThought = 1.0f; // mm/ms
    
	std::string posfn = dir + theme + "_3d_pos.bin";
    pos_file.open(posfn, std::ios::in|std::ios::binary);
	if (!pos_file.is_open()) {
		std::cout << "failed to open pos file:" << posfn << "\n";
		return EXIT_FAILURE;
	}
    mat_file.open(dir + theme + "_mat.bin", std::ios::out | std::ios::binary);
    vec_file.open(dir + theme + "_vec.bin", std::ios::out | std::ios::binary);
    blockPos_file.open(dir + theme + "_blkPos.bin", std::ios::out | std::ios::binary);
    neighborBlock_file.open(dir + theme + "_neighborBlk.bin", std::ios::out | std::ios::binary);
    stats_file.open(dir + theme + "_stats.bin", std::ios::out | std::ios::binary);
    posR_file.open(dir + theme + "_reshaped_pos.bin", std::ios::out|std::ios::binary);
    size_t d_memorySize, memorySize = 0;

    // read from file hipMemcpy to device
    _float *pos;
        memorySize += usingPosDim*networkSize*sizeof(_float);
	
	// to receive from device
    unsigned long outputSize = 0;

    _float *block_x, *block_y; // nblock
        memorySize += 2*nblock*sizeof(_float);
        outputSize += 2*nblock*sizeof(_float);

    unsigned int *preType;
        memorySize += networkSize*sizeof(unsigned int);
        outputSize += networkSize*sizeof(unsigned int);

    _float *conMat, *delayMat;
        memorySize += 2*blockSize*blockSize*nblock*sizeof(_float);
        outputSize += 2*blockSize*blockSize*nblock*sizeof(_float);

    _float *conVec, *delayVec;
        memorySize += 2*networkSize*neighborSize*sizeof(_float);
        outputSize += 2*networkSize*neighborSize*sizeof(_float);

    unsigned int *vecID;
        memorySize += networkSize*neighborSize*sizeof(unsigned int);
        outputSize += networkSize*neighborSize*sizeof(unsigned int);

    unsigned int *nVec;
        memorySize += networkSize*sizeof(unsigned int);
        outputSize += networkSize*sizeof(unsigned int);

    unsigned int *neighborBlockId, *nNeighborBlock;
        memorySize += (nPotentialNeighbor + 1)*nblock*sizeof(unsigned int);
        outputSize += (nPotentialNeighbor + 1)*nblock*sizeof(unsigned int);

    unsigned int *preTypeConnected, *preTypeAvail; // NTYPE*networkSize
        memorySize += 2*NTYPE*networkSize*sizeof(unsigned int);
        outputSize += 2*NTYPE*networkSize*sizeof(unsigned int);

    _float *preTypeStrSum;
        memorySize += NTYPE*networkSize*sizeof(_float);
        outputSize += NTYPE*networkSize*sizeof(_float);

    
	printf("need to allocate %f MB memory on host\n", static_cast<float>(memorySize)/1024/1024);
	void *cpu_chunk = malloc(memorySize);
	assert(cpu_chunk);

    pos = (_float*) cpu_chunk;
    block_x = pos + usingPosDim*networkSize;
    block_y = block_x + nblock;
    preType = (unsigned int*) (block_y + nblock);
    conMat = (_float*) (preType + networkSize); 
    delayMat = conMat + blockSize*blockSize*nblock;
    conVec = delayMat + blockSize*blockSize*nblock; 
    delayVec = conVec + networkSize*neighborSize;
    vecID = (unsigned int*) (delayVec + networkSize*neighborSize);
    nVec = vecID + networkSize*neighborSize;
    neighborBlockId = nVec + networkSize;
    nNeighborBlock = neighborBlockId + nPotentialNeighbor*nblock;
    preTypeConnected = nNeighborBlock + nblock; 
    preTypeAvail = preTypeConnected + NTYPE*networkSize;
    preTypeStrSum = (_float*) (preTypeAvail + NTYPE*networkSize);

	assert(static_cast<void*>((char*)cpu_chunk + memorySize) == static_cast<void*>(preTypeStrSum + NTYPE * networkSize));

    // ========== GPU mem ============
    d_memorySize = memorySize;
    // init by kernel, reside on device only
    _float *rden, *raxn; // NTYPE
		d_memorySize += 2*networkSize*sizeof(_float);

    _float *dden, *daxn;
        d_memorySize += 2*networkSize*sizeof(_float);

    _float *preTypeS;
        d_memorySize += NTYPE*networkSize*sizeof(_float);

    _float *preTypeP;
        d_memorySize += NTYPE*networkSize*sizeof(_float);

    unsigned int *preTypeN;
        d_memorySize += NTYPE*networkSize*sizeof(unsigned int);

    hiprandStateMRG32k3a* state;
        d_memorySize += networkSize*sizeof(hiprandStateMRG32k3a);

    // init by hipMemcpy for kernel , reside on device only
    _float *d_sTypeMat;
        d_memorySize += NTYPE*NTYPE*sizeof(_float);

    _float *d_pTypeMat;
        d_memorySize += NTYPE*NTYPE*sizeof(_float);

    unsigned int *d_nTypeMat;
        d_memorySize += NTYPE*NTYPE*sizeof(unsigned int);

    // init by hipMemcpy
    _float *d_pos;

    // output to host
    _float *d_block_x, *d_block_y;
    unsigned int *d_preType;
    _float *d_conMat, *d_conVec;
    _float *d_delayMat, *d_delayVec;
    unsigned int *d_vecID, *d_nVec;
    unsigned int *d_neighborBlockId, *d_nNeighborBlock;
    unsigned int *d_preTypeConnected, *d_preTypeAvail;
    _float *d_preTypeStrSum;
    void *gpu_chunk;
	printf("need to allocate %f MB memory on device\n", static_cast<float>(d_memorySize) / 1024 / 1024);
    CUDA_CALL(hipMalloc((void**)&gpu_chunk, d_memorySize));

    rden = (_float*) gpu_chunk; 
    raxn = rden + networkSize;
	dden = raxn + networkSize;
	daxn = dden + networkSize;
    preTypeS = daxn + networkSize;
    preTypeP = preTypeS + NTYPE*networkSize;
    preTypeN = (unsigned int*) preTypeP + NTYPE*networkSize;
    state = (hiprandStateMRG32k3a*) (preTypeN + NTYPE*networkSize);
    d_sTypeMat = (_float*) (state + networkSize);
    d_pTypeMat = d_sTypeMat +NTYPE*NTYPE;
    d_nTypeMat = (unsigned int*) (d_pTypeMat + NTYPE*NTYPE);

    d_pos = (_float*) (d_nTypeMat + NTYPE*NTYPE);

    d_block_x = d_pos + usingPosDim*networkSize; 
    d_block_y = d_block_x + nblock;
    d_preType = (unsigned int*) (d_block_y + nblock);
    d_conMat = (_float*) (d_preType + networkSize); 
    d_delayMat = d_conMat + blockSize*blockSize*nblock;
    d_conVec = d_delayMat + blockSize*blockSize*nblock; 
    d_delayVec = d_conVec + networkSize*neighborSize;
    d_vecID = (unsigned int*) d_delayVec + networkSize*neighborSize;
    d_nVec = d_vecID + networkSize*neighborSize;
    d_neighborBlockId = d_nVec + networkSize;
    d_nNeighborBlock = d_neighborBlockId + nPotentialNeighbor*nblock;
    d_preTypeConnected = d_nNeighborBlock + nblock;
    d_preTypeAvail = d_preTypeConnected + NTYPE*networkSize;
    d_preTypeStrSum = (_float*) (d_preTypeAvail + NTYPE*networkSize);

	assert(static_cast<void*>((char*)gpu_chunk + d_memorySize) == static_cast<void*>(d_preTypeStrSum + NTYPE * networkSize));

    double* tmp = new double[networkSize*usingPosDim];
    pos_file.read(reinterpret_cast<char*>(tmp), usingPosDim*networkSize*sizeof(double));
	for (unsigned int i = 0; i < networkSize * usingPosDim; i++) {
		pos[i] = static_cast<_float>(reinterpret_cast<double*>(tmp)[i]);
	}
	delete[]tmp;
    unsigned int localHeapSize = sizeof(_float)*networkSize*nPotentialNeighbor*blockSize;
    hipDeviceSetLimit(hipLimitMallocHeapSize, localHeapSize*1.5);
    printf("heap size preserved %f Mb\n", localHeapSize*1.5/1024/1024);
    hipStream_t s0, s1, s2;
    hipEvent_t i0, i1, i2;
    hipEventCreate(&i0);
    hipEventCreate(&i1);
    hipEventCreate(&i2);
    CUDA_CALL(hipStreamCreate(&s0));
    CUDA_CALL(hipStreamCreate(&s1));
    CUDA_CALL(hipStreamCreate(&s2));
    CUDA_CALL(hipMemcpy(d_pos, pos, usingPosDim*networkSize*sizeof(_float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_sTypeMat, sTypeMat, NTYPE*NTYPE*sizeof(_float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_pTypeMat, pTypeMat, NTYPE*NTYPE*sizeof(_float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_nTypeMat, nTypeMat, NTYPE*NTYPE*sizeof(_float), hipMemcpyHostToDevice));
    initialize<<<nblock, blockSize, 0, s0>>>(state, 
											 d_preType, 
											 rden, 
											 raxn, 
											 dden, 
											 daxn, 
											 d_sTypeMat,
											 d_pTypeMat,
											 d_nTypeMat,
											 preTypeS, 
											 preTypeP, 
											 preTypeN, 
											 init_pack, seed, networkSize);
	CUDA_CHECK();
	//CUDA_CALL(hipEventRecord(i1, s1));
	//CUDA_CALL(hipEventSynchronize(i1));
    printf("initialzied\n");
    unsigned int shared_mem;
    shared_mem = 2*warpSize*sizeof(_float);
    cal_blockPos<<<nblock, blockSize, shared_mem, s1>>>(d_pos, 
														d_block_x, 
														d_block_y, 
														networkSize);
	CUDA_CHECK();
	CUDA_CALL(hipEventRecord(i1, s1));
	CUDA_CALL(hipEventSynchronize(i1));
    printf("block centers calculated\n");
	shared_mem = sizeof(unsigned int);
    get_neighbor_blockId<<<nblock, blockSize, shared_mem, s0>>>(d_block_x, 
																d_block_y, 
																d_neighborBlockId, 
																d_nNeighborBlock, 
																max_radius, nPotentialNeighbor);
	CUDA_CHECK();
	CUDA_CALL(hipEventRecord(i1, s1));
	CUDA_CALL(hipEventSynchronize(i1));
    printf("neighbor blocks acquired\n");
	//CUDA_CALL(hipEventRecord(i0, s0));
	//CUDA_CALL(hipEventSynchronize(i0));
	//CUDA_CALL(hipEventSynchronize(i1));
	//CUDA_CALL(hipEventSynchronize(i2));
    shared_mem = blockSize*sizeof(_float) + blockSize*sizeof(_float) + blockSize*sizeof(unsigned int);
    generate_connections<<<nblock, blockSize, shared_mem, s0>>>(d_pos, 
																preTypeS,
																preTypeP,
																preTypeN,
																d_neighborBlockId, 
																d_nNeighborBlock, 
																rden, 
																raxn, 
																d_conMat, 
																d_delayMat, 
																d_conVec, 
																d_delayVec, 
																d_vecID,
                                                                d_nVec,
																d_preTypeConnected, 
																d_preTypeAvail, 
																d_preTypeStrSum, 
																d_preType, 
																dden, 
																daxn, 
																state, 
																networkSize, neighborSize, nPotentialNeighbor, speedOfThought);
	CUDA_CHECK();
	CUDA_CALL(hipEventRecord(i0, s0));
	CUDA_CALL(hipEventSynchronize(i0));
    printf("connectivity constructed\n");
	CUDA_CALL(hipMemcpy(block_x, d_block_x, outputSize, hipMemcpyDeviceToHost)); // the whole chunk of output
	//CUDA_CALL(hipMemcpy(preType, d_preType, 1, hipMemcpyDeviceToHost));
	CUDA_CALL(hipStreamDestroy(s0));
    CUDA_CALL(hipStreamDestroy(s1));
    CUDA_CALL(hipStreamDestroy(s2));
    // output to binary data files
    mat_file.write((char*)conMat, nblock*blockSize*blockSize*sizeof(_float));
    mat_file.write((char*)delayMat, nblock*blockSize*blockSize*sizeof(_float));
    
    vec_file.write((char*)nVec, networkSize*sizeof(unsigned int));
    for (unsigned int i=0; i<networkSize; i++) {
        vec_file.write((char*)&(vecID[i*neighborSize]), nVec[i]*sizeof(unsigned int));
        vec_file.write((char*)&(conVec[i*neighborSize]), nVec[i]*sizeof(_float));
        vec_file.write((char*)&(delayVec[i*neighborSize]), nVec[i]*sizeof(_float));
    }

    blockPos_file.write((char*)block_x, nblock*sizeof(_float));
    blockPos_file.write((char*)block_y, nblock*sizeof(_float));

    neighborBlock_file.write((char*)nNeighborBlock, nblock*sizeof(unsigned int));
    for (unsigned int i=0; i<nblock; i++) {
        neighborBlock_file.write((char*)&(neighborBlockId[i*nPotentialNeighbor]), nNeighborBlock[i]*sizeof(unsigned int));
    }

    stats_file.write((char*)preTypeConnected, NTYPE*networkSize*sizeof(unsigned int));
    stats_file.write((char*)preTypeAvail, NTYPE*networkSize*sizeof(unsigned int));
    stats_file.write((char*)preTypeStrSum, NTYPE*networkSize*sizeof(_float));
    
    posR_file.write((char*)pos, networkSize*usingPosDim*sizeof(_float));

    CUDA_CALL(hipFree(gpu_chunk));
	free(cpu_chunk);
    return 0;
}
