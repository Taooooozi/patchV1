#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>
#include <string>
#include <iostream>
#include <cassert>
#include <ctime>
#include <cmath>
#include <fenv.h>
//#include <hip/hip_runtime.h>
//#include <hiprand/hiprand_kernel.h>
//#include "hip/hip_runtime.h"

#include "connect.h"

template <typename T, typename I>
void check_statistics(T* array, I n, T &max, T &min, T &mean, T &std) {

}

int main(int argc, char *argv[])
{
    unsigned long int i = (unsigned long int) -1;
    printf("max unsigned long = %lu\n", i);
    unsigned long long seed = 7548637;
    std::ifstream pos_file;
    std::string dir(argv[1]);
    std::string theme(argv[2]);
    std::ofstream conMat_file, conVec_file, stats_file;
    unsigned int nblock = 48;
    unsigned int nPotentialNeighbor = 8;
    unsigned int networkSize = nblock*blockSize;
    unsigned int neighborSize = 400;
    unsigned int usingPosDim = 2;

    _float radius[NTYPE][2];
    _float neuron_type_acc_count[NTYPE+1];
	_float den_axn[NTYPE];
	_float den_den[NTYPE];
    // E
    _float scale = 5.0f;
    radius[0][0] = 0.08f*scale;
    radius[0][1] = 0.15f*scale;
    // I
    radius[1][0] = 0.05f*scale;
    radius[1][1] = 0.1f*scale;

    // row <- column
    _float sTypeMat[NTYPE][NTYPE];
    sTypeMat[0][0] = 1.0f;
    sTypeMat[0][1] = 4.0f;
    sTypeMat[1][0] = 1.0f;
    sTypeMat[1][1] = 4.0f;

    //upper limit of sparsity stricted to NTYPE
    _float pTypeMat[NTYPE][NTYPE];
    pTypeMat[0][0] = 0.15f;
    pTypeMat[0][1] = 0.15f;
    pTypeMat[1][0] = 0.5f;
    pTypeMat[1][1] = 0.5f;

    // 
    unsigned int nTypeMat[NTYPE][NTYPE];
    nTypeMat[0][0] = 400+20*5; // mean + std*5
    nTypeMat[0][1] = 100+10*5;
    nTypeMat[1][0] = 400+20*5;
    nTypeMat[1][1] = 100+10*5;
    
    // NTYPE
    neuron_type_acc_count[0] = 0;
    neuron_type_acc_count[1] = 768;
    neuron_type_acc_count[2] = 1024;
    assert(neuron_type_acc_count[NTYPE] == blockSize);

    den_axn[0] = 1.0;
    den_axn[1] = 2.0;

    den_den[0] = 1.0;
    den_den[1] = 2.0;

    initialize_package init_pack(radius, neuron_type_acc_count, den_axn, den_den);
    _float speedOfThought = 1.0f; // mm/ms
    _float max_radius = 0.4f*scale;
    
	std::string posfn = dir + theme + "_3d_pos.bin";
    pos_file.open(posfn, std::ios::in|std::ios::binary);
	if (!pos_file.is_open()) {
		std::cout << "failed to open pos file:" << posfn << "\n";
		return EXIT_FAILURE;
	}
    conMat_file.open(theme + "_conMat.bin", std::ios::out | std::ios::binary);
    conVec_file.open(theme + "_conVec.bin", std::ios::out | std::ios::binary);
    size_t d_memorySize, memorySize = 0;

    // read from file hipMemcpy to device
    _float *pos;
        memorySize += usingPosDim*networkSize*sizeof(_float);
	
	// to receive from device
    unsigned long outputSize = 0;

    _float *block_x, *block_y; // nblock
        memorySize += 2*nblock*sizeof(_float);
        outputSize += 2*nblock*sizeof(_float);

    unsigned int *preType;
        memorySize += networkSize*sizeof(unsigned int);
        outputSize += networkSize*sizeof(unsigned int);

    _float *conMat, *delayMat;
        memorySize += 2*blockSize*blockSize*nblock*sizeof(_float);
        outputSize += 2*blockSize*blockSize*nblock*sizeof(_float);

    _float *conVec, *delayVec;
        memorySize += 2*networkSize*neighborSize*sizeof(_float);
        outputSize += 2*networkSize*neighborSize*sizeof(_float);

    unsigned int *vecID;
        memorySize += networkSize*neighborSize*sizeof(unsigned int);
        outputSize += networkSize*neighborSize*sizeof(unsigned int);

    unsigned int *neighborBlockId, *nNeighborBlock;
        memorySize += (nPotentialNeighbor + 1)*nblock*sizeof(unsigned int);
        outputSize += (nPotentialNeighbor + 1)*nblock*sizeof(unsigned int);

    unsigned int *preTypeConnected, *preTypeAvail; // NTYPE*networkSize
        memorySize += 2*NTYPE*networkSize*sizeof(unsigned int);
        outputSize += 2*NTYPE*networkSize*sizeof(unsigned int);

    _float *preTypeStrSum;
        memorySize += NTYPE*networkSize*sizeof(_float);
        outputSize += NTYPE*networkSize*sizeof(_float);

    
	printf("need to allocate %f MB memory on host\n", static_cast<float>(memorySize)/1024/1024);
	void *cpu_chunk = malloc(memorySize);
	assert(cpu_chunk);

    pos = (_float*) cpu_chunk;
    block_x = pos + usingPosDim*networkSize;
    block_y = block_x + nblock;
    preType = (unsigned int*) (block_y + nblock);
    conMat = (_float*) (preType + networkSize); 
    delayMat = conMat + blockSize*blockSize*nblock;
    conVec = delayMat + blockSize*blockSize*nblock; 
    delayVec = conVec + networkSize*neighborSize;
    vecID = (unsigned int*) (delayVec + networkSize*neighborSize);
    neighborBlockId = vecID + networkSize*neighborSize;
    nNeighborBlock = neighborBlockId + nPotentialNeighbor*nblock;
    preTypeConnected = nNeighborBlock + nblock; 
    preTypeAvail = preTypeConnected + NTYPE*networkSize;
    preTypeStrSum = (_float*) (preTypeAvail + NTYPE*networkSize);

	assert(static_cast<void*>((char*)cpu_chunk + memorySize) == static_cast<void*>(preTypeStrSum + NTYPE * networkSize));

    // ========== GPU mem ============
    d_memorySize = memorySize;
    // init by kernel, reside on device only
    _float *rden, *raxn; // NTYPE
		d_memorySize += 2*networkSize*sizeof(_float);

    _float *dden, *daxn;
        d_memorySize += 2*networkSize*sizeof(_float);

    _float *preTypeS;
        d_memorySize += NTYPE*networkSize*sizeof(_float);

    _float *preTypeP;
        d_memorySize += NTYPE*networkSize*sizeof(_float);

    unsigned int *preTypeN;
        d_memorySize += NTYPE*networkSize*sizeof(unsigned int);

    hiprandStateMRG32k3a* state;
        d_memorySize += networkSize*sizeof(hiprandStateMRG32k3a);

    // init by hipMemcpy for kernel , reside on device only
    _float *d_sTypeMat;
        d_memorySize += NTYPE*NTYPE*sizeof(_float);

    _float *d_pTypeMat;
        d_memorySize += NTYPE*NTYPE*sizeof(_float);

    unsigned int *d_nTypeMat;
        d_memorySize += NTYPE*NTYPE*sizeof(unsigned int);

    // init by hipMemcpy
    _float *d_pos;

    // output to host
    _float *d_block_x, *d_block_y;
    unsigned int *d_preType;
    _float *d_conMat, *d_conVec;
    _float *d_delayMat, *d_delayVec;
    unsigned int *d_vecID;
    unsigned int *d_neighborBlockId, *d_nNeighborBlock;
    unsigned int *d_preTypeConnected, *d_preTypeAvail;
    _float *d_preTypeStrSum;
    void *gpu_chunk;
	printf("need to allocate %f MB memory on device\n", static_cast<float>(d_memorySize) / 1024 / 1024);
    CUDA_CALL(hipMalloc((void**)&gpu_chunk, d_memorySize));

    rden = (_float*) gpu_chunk; 
    raxn = rden + networkSize;
	dden = raxn + networkSize;
	daxn = dden + networkSize;
    preTypeS = daxn + networkSize;
    preTypeP = preTypeS + NTYPE*networkSize;
    preTypeN = (unsigned int*) preTypeP + NTYPE*networkSize;
    state = (hiprandStateMRG32k3a*) (preTypeN + NTYPE*networkSize);
    d_sTypeMat = (_float*) (state + networkSize);
    d_pTypeMat = d_sTypeMat +NTYPE*NTYPE;
    d_nTypeMat = (unsigned int*) (d_pTypeMat + NTYPE*NTYPE);

    d_pos = (_float*) (d_nTypeMat + NTYPE*NTYPE);

    d_block_x = d_pos + usingPosDim*networkSize; 
    d_block_y = d_block_x + nblock;
    d_preType = (unsigned int*) (d_block_y + nblock);
    d_conMat = (_float*) (d_preType + networkSize); 
    d_delayMat = d_conMat + blockSize*blockSize*nblock;
    d_conVec = d_delayMat + blockSize*blockSize*nblock; 
    d_delayVec = d_conVec + networkSize*neighborSize;
    d_vecID = (unsigned int*) d_delayVec + networkSize*neighborSize;
    d_neighborBlockId = d_vecID + networkSize*neighborSize;
    d_nNeighborBlock = d_neighborBlockId + nPotentialNeighbor*nblock;
    d_preTypeConnected = d_nNeighborBlock + nblock;
    d_preTypeAvail = d_preTypeConnected + NTYPE*networkSize;
    d_preTypeStrSum = (_float*) (d_preTypeAvail + NTYPE*networkSize);

	assert(static_cast<void*>((char*)gpu_chunk + d_memorySize) == static_cast<void*>(d_preTypeStrSum + NTYPE * networkSize));

    double* tmp = new double[networkSize*usingPosDim];
    pos_file.read(reinterpret_cast<char*>(tmp), usingPosDim*networkSize*sizeof(double));
	for (unsigned int i = 0; i < networkSize * usingPosDim; i++) {
		pos[i] = static_cast<_float>(reinterpret_cast<double*>(tmp)[i]);
	}
	delete[]tmp;
    unsigned int localHeapSize = sizeof(_float)*networkSize*nPotentialNeighbor*blockSize;
    hipDeviceSetLimit(hipLimitMallocHeapSize, localHeapSize*1.5);
    printf("heap size preserved %f Mb\n", localHeapSize*1.5/1024/1024);
    hipStream_t s0, s1, s2;
    hipEvent_t i0, i1, i2;
    hipEventCreate(&i0);
    hipEventCreate(&i1);
    hipEventCreate(&i2);
    CUDA_CALL(hipStreamCreate(&s0));
    CUDA_CALL(hipStreamCreate(&s1));
    CUDA_CALL(hipStreamCreate(&s2));
    CUDA_CALL(hipMemcpy(d_pos, pos, usingPosDim*networkSize*sizeof(_float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_sTypeMat, sTypeMat, NTYPE*NTYPE*sizeof(_float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_pTypeMat, pTypeMat, NTYPE*NTYPE*sizeof(_float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_nTypeMat, nTypeMat, NTYPE*NTYPE*sizeof(_float), hipMemcpyHostToDevice));
    initialize<<<nblock, blockSize, 0, s0>>>(state, 
											 d_preType, 
											 rden, 
											 raxn, 
											 dden, 
											 daxn, 
											 d_sTypeMat,
											 d_pTypeMat,
											 d_nTypeMat,
											 preTypeS, 
											 preTypeP, 
											 preTypeN, 
											 init_pack, seed, networkSize);
	CUDA_CHECK();
	//CUDA_CALL(hipEventRecord(i1, s1));
	//CUDA_CALL(hipEventSynchronize(i1));
    printf("initialzied\n");
    unsigned int shared_mem;
    shared_mem = 2*warpSize*sizeof(_float);
    cal_blockPos<<<nblock, blockSize, shared_mem, s1>>>(d_pos, 
														d_block_x, 
														d_block_y, 
														networkSize);
	CUDA_CHECK();
	CUDA_CALL(hipEventRecord(i1, s1));
	CUDA_CALL(hipEventSynchronize(i1));
    printf("block centers calculated\n");
	shared_mem = sizeof(unsigned int);
    get_neighbor_blockId<<<nblock, blockSize, shared_mem, s0>>>(d_block_x, 
																d_block_y, 
																d_neighborBlockId, 
																d_nNeighborBlock, 
																max_radius, nPotentialNeighbor);
	CUDA_CHECK();
	CUDA_CALL(hipEventRecord(i1, s1));
	CUDA_CALL(hipEventSynchronize(i1));
    printf("neighbor blocks acquired\n");
	//CUDA_CALL(hipEventRecord(i0, s0));
	//CUDA_CALL(hipEventSynchronize(i0));
	//CUDA_CALL(hipEventSynchronize(i1));
	//CUDA_CALL(hipEventSynchronize(i2));
    shared_mem = blockSize*sizeof(_float) + blockSize*sizeof(_float) + blockSize*sizeof(unsigned int);
    generate_connections<<<nblock, blockSize, shared_mem, s0>>>(d_pos, 
																preTypeS,
																preTypeP,
																preTypeN,
																d_neighborBlockId, 
																d_nNeighborBlock, 
																rden, 
																raxn, 
																d_conMat, 
																d_delayMat, 
																d_conVec, 
																d_delayVec, 
																d_vecID,
																d_preTypeConnected, 
																d_preTypeAvail, 
																d_preTypeStrSum, 
																d_preType, 
																dden, 
																daxn, 
																state, 
																networkSize, neighborSize, nPotentialNeighbor, speedOfThought);
	CUDA_CHECK();
	CUDA_CALL(hipEventRecord(i0, s0));
	CUDA_CALL(hipEventSynchronize(i0));
    printf("connectivity constructed\n");
	CUDA_CALL(hipMemcpy(block_x, d_block_x, outputSize, hipMemcpyDeviceToHost)); // the whole chunk of output
	//CUDA_CALL(hipMemcpy(preType, d_preType, 1, hipMemcpyDeviceToHost));
	CUDA_CALL(hipStreamDestroy(s0));
    CUDA_CALL(hipStreamDestroy(s1));
    CUDA_CALL(hipStreamDestroy(s2));
    /*unsigned long preSumN[NTYPE][NTYPE];
	unsigned long preSumStr[NTYPE][NTYPE];
    for (unsigned int i=0; i<networkSize; i++) {
        for (unsigned int j=0; j<NTYPE; j++) {
        }
    }*/
    CUDA_CALL(hipFree(gpu_chunk));
	free(cpu_chunk);
    return 0;
}
