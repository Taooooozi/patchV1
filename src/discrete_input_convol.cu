#include "hip/hip_runtime.h"
#include "discrete_input_convol.cuh"

extern texture<float, hipTextureType2DLayered> L_retinaConSig;
extern texture<float, hipTextureType2DLayered> M_retinaConSig;
extern texture<float, hipTextureType2DLayered> S_retinaConSig;
extern __device__ __constant__ float sqrt2;

__device__ 
__forceinline__ 
Float spatialKernel(Float x, Float y, Float rx, Float ry) {
    return exponential(-x*x/(rx*rx) - y*y/(ry*ry));
}

__device__ 
__forceinline__ 
Float temporalKernel(Float tau, Zip_temporal &temp, Float lfac1, Float lfac2, Size lid, Size tid) {
    Float tau1 = tau/temp.tauR;
    Float tau2 = tau/temp.tauD;
    //Float A1 = power(tau1, temp.nR-1)/temp.tauR;
    //Float A2 = power(tau2, temp.nD-1)/temp.tauD;

    Float A1 = (temp.nR-1) * logrithm(tau1);
    Float A2 = (temp.nD-1) * logrithm(tau2);

    //if ((lid ==0 || lid == gridDim.x) && tid == 0) {
    //  printf("lid:%d, tid:%d\n A1 = %f, tau1 = %f\n A2 = %f, tau2 = %f\n", lid, tid, A1, tau1, A2, tau2);
    //}

    //Float tp = AexpTau(A1, tau1 + lfac1) - temp.ratio*AexpTau(A2, tau2 + lfac2);

    Float tpR = exponential(A1 - tau1 - lfac1)/temp.tauR;
    Float tpD = exponential(A2 - tau2 - lfac2)/temp.tauD;
    Float tp = tpR - temp.ratio*tpD;
    //if ((lid ==0 || lid == gridDim.x) && tid == 0) {
    //  printf("lid:%d, id:%d, tpR = %f, tpD = %f\n", lid, tid, tpR, tpD);
    //}
    return tp;
}

__device__ 
__forceinline__
Float get_intensity(SmallSize coneType, float x, float y, unsigned int iLayer) {
    Float contrast;
    switch (coneType) {
        case 0:
            contrast = static_cast<Float>(tex2DLayered(L_retinaConSig, x, y, iLayer));
            break;
        case 1:
            contrast = static_cast<Float>(tex2DLayered(M_retinaConSig, x, y, iLayer));
            break;
        case 2:
            contrast = static_cast<Float>(tex2DLayered(S_retinaConSig, x, y, iLayer));
            break;
        case 3:
            contrast = static_cast<Float>(tex2DLayered(L_retinaConSig, x, y, iLayer) 
                                        + tex2DLayered(M_retinaConSig, x, y, iLayer))/2.0; 
            break;
        case 4:
            contrast = static_cast<Float>(tex2DLayered(L_retinaConSig, x, y, iLayer) 
                                        + tex2DLayered(M_retinaConSig, x, y, iLayer) 
                                        + tex2DLayered(S_retinaConSig, x, y, iLayer))/3.0;
            break;
        default:
            printf("unrecognized cone type");
        /*
        case 4:
            break;
        case 5:
            break;
        case 6:
            break;
        */
    }
    if (contrast <= 0) {
        printf("(%f,%f) = %f, L:%f, M:%f, S: %f\n", x, y, contrast
                , static_cast<Float>(tex2DLayered(L_retinaConSig, x, y, iLayer))
                , static_cast<Float>(tex2DLayered(M_retinaConSig, x, y, iLayer))
                , static_cast<Float>(tex2DLayered(S_retinaConSig, x, y, iLayer)));
        assert(contrast > 0);
    }
    return contrast;
}

// for 2 cone-types LGN only, can be generalized for more cone-types
// gridSize: (nLGN, nType) blocks for store 1-D nLGN for convol
// blockSize: spatialSample1D x spatialSample1D (npixel_1D)

/* TODO: speed comparison, all thread load from global memory vs load shared memory then to thread
    __device__
    __inline__
    void store_spatialWeight0(
            Spatial_component &spatial,
            Float* __restrict__ SW_storage
            Float* __restrict__ SC_storage
            Float xhspan, 
            Float yhspan,
            Float dx,
            Float dy,
            Float nsig, // span of spatialRF sample in units of std
            Size id,
            Size lid,
            Size iType,
            Size nType,
    ) {
        // parameter are stored as (nType, nLGN), weights are stored as (nLGN, nType, weight)
        Zip_spatial spat;
        spat.load(spatial, lid);
        SmallSize nSample = blockDim.x * blockDim.y;
        Size offset = (id*nType + iType)*nSample;
        Size tid = threadIdx.y*blockDim.x + threadIdx.x;
        Size storeID = offset + tid;
        Size xID = storeID + offset;
        Size yID = xID + nSample;
    
        // coord to center
        Float x = (threadIdx.x + 0.5)*dx - xhspan;
        Float y = (threadIdx.y + 0.5)*dy - yhspan;
        Float sample_vol = dx * dy;
    
        SW_storage[storeID] = spatialKernel(x, y, k, spat.rx, spat.ry)*sample_vol;
        float x_plane, y_plane;
        retina_to_plane(cx + x, cy + y, x_plane, y_plane);
        SC_storage[xID] = x_plane;
        SC_storage[yID] = y_plane;
    }
*/

// iType is for center surround (or multiple surroud) in a single LGN
// not to be confused with coneTypes
// weights are stored in shapes of (nLGN, nType, nKernelSample)
__device__
__forceinline__
void store_temporalWeight(
        Temporal_component &temporal,
        Float* __restrict__ TW_storage,
        Float* __restrict__ reduced, //__shared__
        Float &temporalWeight,
        SmallSize nKernelSample,
        Float kernelSampleDt,
        Float kernelSampleT0,
        Size id,
        Size tid,
        Size lid,
        SmallSize iType,
        SmallSize nType
) {
    Zip_temporal temp;
    // load temporal parameters
    temp.load(temporal, lid);
    SmallSize patchSize = blockDim.x*blockDim.y;
    Size nPatch = nKernelSample/patchSize;
    Size remain = nKernelSample%patchSize;
    /*DEBUG
    if ((lid ==0 || lid == gridDim.x) && tid == 0) {
        printf("%f, %f, %f, %f, %f, %f\n", temp.nR, temp.nD, temp.tauR, temp.tauD, temp.delay, temp.ratio);
        printf("patchSize = %u, nPatch = %u, remain = %u\n", patchSize, nPatch, remain);
    }
    __syncthreads();
    */
    
    Float lfac1, lfac2;
    lfac1 = log_gamma(temp.nR);
    lfac2 = log_gamma(temp.nD);

    /*DEBUG
    if ((lid ==0 || lid == gridDim.x) && tid == 0) {
        printf("lid:%d, tid:%d\n%f: %f, %f, %f\n %f: %f, %f, %f\n", lid, tid, temp.nR, lfac1, tgamma(temp.nR), exp(lfac1), temp.nD, lfac2, tgamma(temp.nD), exp(lfac2));
    }
	__syncthreads(); 
    */
    // account for the delay into T0
    kernelSampleT0 -= temp.delay;
    // initialize the sum of temporalWeights to 0
    temporalWeight = 0;
    for (Size iPatch = 0; iPatch < nPatch+1; iPatch++) {
        Float tw;
        if (iPatch < nPatch || tid < remain) {
            // temporalKernel takes abs(now-t)
            // but we store weight in time-reverse (tau -> 0)
            // i.e., last in storage-> 0 -^v-- tau <-first in storage,   
            SmallSize twid = nKernelSample-1 - iPatch*patchSize - tid;
            Size storeID = (id*nType + iType)*nKernelSample + iPatch*patchSize + tid;

			Float t = twid * kernelSampleDt + kernelSampleT0;

            /*DEBUG
			if ((lid ==0 || lid == gridDim.x) && tid == 0) {
              printf("lid = %d, tid = %d, t = %f\n", lid, tid, t);
            }
            */
            if (t < 0) {
                tw = 0.0;
            } else {
                tw = temporalKernel(t, temp, lfac1, lfac2, lid, tid);
            }
            /*DEBUG
			if ((lid ==0 || lid == gridDim.x) && tid == 0) {
              printf("lid = %d, tid = %d, tw = %f\n", lid, tid, tw);
            }
            */
			
            TW_storage[storeID] = tw;
            // get absolute values ready for max_convol
            tw = abs(tw);
        } else {
            tw = 0.0;
        }
        assert(!isnan(tw));
        __syncthreads();
        block_reduce<Float>(reduced, tw);
        if (tid == 0) {
            temporalWeight += reduced[0];
        }
    }
}

// coordinates are stored as (2, nLGN, nType, nSample), 
// weights are stored as (nLGN, nType, nSample)
__device__
__forceinline__
void store_spatialWeight(
        Float* reduced,
		Float centerPolar,
		Float centerEcc,
		Float coso,
		Float sino,
		Float wSpan,
		Float hSpan,
		Float dw,
		Float dh,
		Float wSigSqrt2,
		Float hSigSqrt2,
		Float normViewDistance,
		Float LR_x0,
		Float LR_y0,
        bool LR,
        Float* __restrict__ SW_storage,
        float* __restrict__ SC_storage,
        Size storeID, // (id*nType + iType) * nSample + tid;
        Size nSample
) {
    // rads relative to center
    Float w = (threadIdx.x + 0.5)*dw - wSpan;
    Float h = (threadIdx.y + 0.5)*dh - hSpan;

    Float spatialWeight = spatialKernel(w, h, wSigSqrt2, hSigSqrt2);
    
	block_reduce<Float>(reduced, spatialWeight);
    // TODO: gaussian spatialWeight with fixed sample point in the unit of sigma is the same across neuorns, can be passed from host directly
    SW_storage[storeID] = spatialWeight/reduced[0];
	Float cosp, sinp; 
    Float cosEcc, sinEcc;
	orthPhiRotate3D(centerPolar, centerEcc + h, w, cosp, sinp, cosEcc, sinEcc);

    Float tanEcc;
	axisRotate3D(centerPolar, centerEcc, coso, sino, cosp, sinp, cosEcc, sinEcc, tanEcc);

    float x, y;
    retina_to_plane(cosp, sinp, tanEcc, x, y, normViewDistance, LR_x0, LR_y0);
    { // DEBUG visual field and stimulus field not matching
        if (LR) {
            if (x < 0 || x > 0.5) {
                printf("x\n");
                assert(x>=0);
                assert(x<=0.5);
            }
        } else {
            if (x < 0.5 || x > 1) {
                printf("x\n");
                assert(x>=0.5);
                assert(x<=1);
            }
        }
        if (y<0 || y>1) {
            printf("y\n");
            assert(y>=0);
            assert(y<=1);
        }
    }
    
    // store coords for retrieve data from texture
    SC_storage[storeID] = x; // x
               //nLGN * nType * nSample (all the x)
    SC_storage[gridDim.x*gridDim.y*nSample + storeID] = y; // y
}

//__launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)
__launch_bounds__(1024, 2)
__global__
void store(
        Float* __restrict__ max_convol,

        Temporal_component &temporal,
        Float* __restrict__ TW_storage,
        SmallSize nKernelSample,
        Float kernelSampleDt,
        Float kernelSampleT0,

        Spatial_component &spatial,
        Float* __restrict__ SW_storage,
        float* __restrict__ SC_storage,
		Size nLGN_L,
		Float L_x0,
		Float L_y0,
		Float R_x0,
		Float R_y0,
		Float normViewDistance,
        Float nsig // span of spatialRF sample in units of std
) {
    __shared__ Float reduced[warpSize];
    __shared__ Float shared_spat[10]; // centerPolar, centerEcc, coso, sino, wSpan, hSpan, dw, dh, wSigSqrt2, hSigSqrt2
    Size id = blockIdx.x;
    SmallSize iType = blockIdx.y;
    Size lid = iType*gridDim.x + id;
    SmallSize nType = gridDim.y;
    Size tid = threadIdx.y*blockDim.x + threadIdx.x;
    SmallSize nSample = blockDim.x * blockDim.y;

    Float temporalWeight;
    store_temporalWeight(temporal, TW_storage, reduced, temporalWeight, nKernelSample, kernelSampleDt, kernelSampleT0, id, tid, lid, iType, nType);
    /* DEBUG
	if ((lid ==0 || lid == gridDim.x) && tid == 0) {
        printf("temporalWeights stored\n");
        assert(!isnan(temporalWeight));
    }*/
    __syncthreads();
    //
	Float LR_x0, LR_y0;
    bool LR = id < nLGN_L;
	if (LR) {
		LR_x0 = L_x0;
		LR_y0 = L_y0;
	} else {
		LR_x0 = R_x0;
		LR_y0 = R_y0;
	}

    bool use_shared = true;
    Float k;
    if (use_shared) { // TODO: *compare with global broadcast
        Size offset = id*nType + iType;
        if (tid == 0) {
            Zip_spatial spat;
            spat.load(spatial, lid);
            Float wSpan = nsig * spat.rx / sqrt2;
            Float dw = 2*wSpan/blockDim.x;

            Float hSpan = nsig * spat.ry / sqrt2;
            Float dh = 2*hSpan/blockDim.y;
			k = spat.k;
			{
            	shared_spat[0] = spat.x;
				shared_spat[1] = spat.y;
				shared_spat[2] = cosine(spat.orient); 
				shared_spat[3] = sine(spat.orient); 
				shared_spat[4] = wSpan;
				shared_spat[5] = hSpan;
				shared_spat[6] = dw;
				shared_spat[7] = dh;
				shared_spat[8] = spat.rx; 
				shared_spat[9] = spat.ry;
			}
        }
        __syncthreads();
        // load from shared mem
        store_spatialWeight(reduced, shared_spat[0], shared_spat[1], shared_spat[2], shared_spat[3], shared_spat[4], shared_spat[5], shared_spat[6], shared_spat[7], shared_spat[8], shared_spat[9], normViewDistance, LR_x0, LR_y0, LR, SW_storage, SC_storage, offset*nSample+tid, nSample);
    } 
    //spatialWeight = abs(spatialWeight); // get absolute values ready for max_convol, always positive, not necessary
    /* DEBUG
	if ((lid ==0 || lid == gridDim.x) && tid == 0) {
        printf("spatialWeights stored\n");
        assert(!isnan(spatialWeight));
	    if (lid ==0 && tid == 0) {
            assert(max_convol[id] == 0.0);
        }
    }
    */
    // k is now integrated amplitude over space 
    if (tid == 0) { // add center surround together, iType = 0, 1
        atomicAdd(max_convol+id, temporalWeight * abs(k) * kernelSampleDt);
    }
}

// grid: [nLGN, 2, 1]
// block: [nSpatialSample1D, nSpatialSample1D, 1]
__launch_bounds__(1024, 2)
__global__ 
void LGN_convol_c1s(
        Float* __restrict__ luminance,
        Float* __restrict__ SW_storage,
        float* __restrict__ SC_storage,
        Float* __restrict__ TW_storage,
        Float* __restrict__ current_convol,
        Float* __restrict__ contrast,
        SmallSize* __restrict__ coneType,
        Spatial_component &spatial,
		Size nLGN_L,
		Float normViewDistance,
        PosInt currentFrame,
        Size maxFrame,
		Size ntPerFrame,
        PosInt iFramePhase,
        Float Itau,
        Size iKernelSampleT0,
        Size kernelSampleInterval,
        Size nKernelSample,
        Float dt,
        Size denorm 
) {
    __shared__ Float reducedS[warpSize];
    __shared__ Float reducedC[warpSize];
    extern __shared__ Float nSampleShared[];

    // weights are stored in shapes of (nLGN, nType, weight)
    Size tid = threadIdx.y*blockDim.x + threadIdx.x;
    Size nSample = blockDim.x * blockDim.y;
	
    //TODO: Itau may take different value for different cone type
    // convolve center and update luminance
    Float convolS, convolC;
    Float contrastS, contrastC;
	if (tid == 0) {
		convolS = 0.0;
		convolC = 0.0;
	}
    /* kernel sampling diagram with frames
                      [,)
        frame:       curr            next              next+1
        framePhase|tPerFrame-framePhase                  ^        
        time:  <--|   ^   |-------> tPerFrame <------|   ^    |
                now-tau                                       now
        sample:           1       2       3       4       5     
                  |...|...:---|-------|-------|------:|---|---|
        dt:     -4   0    4   8       16      24  v   32    v 40
                  v                               Dt        T0 
         lastDecay|nextDecay
        e.g., tau = 40*dt
    */
    Size lidS = 1*gridDim.x + blockIdx.x;
	Size offsetS = blockIdx.x*2 + 1;
    Size storeIDS = offsetS*nSample + tid;
    Size lidC = 0*gridDim.x + blockIdx.x;
	Size offsetC = blockIdx.x*2 + 0;
    Size storeIDC = offsetC*nSample + tid;

    // coord on the stimulus plane
    float x0C = SC_storage[storeIDC];
    float y0C = SC_storage[gridDim.x*gridDim.y*nSample + storeIDC];
    float x0S = SC_storage[storeIDS];
    float y0S = SC_storage[gridDim.x*gridDim.y*nSample + storeIDS];
    assert(x0S <= 1.0);
    assert(y0S <= 1.0);
    assert(x0C <= 1.0);
    assert(y0C <= 1.0);
    assert(x0S >= 0.0);
    assert(y0S >= 0.0);
    assert(x0C >= 0.0);
    assert(y0C >= 0.0);

    Float kS, kC;
    if (tid == 0) {
        kS = spatial.k[lidS];
        kC = spatial.k[lidC];
    }
    /* Light adaptation process:
        tau*dI/dt = -I + F(t);
        F(t) = piecewise F(t_{i}), for t in [t_{i}, t_{i+1}), t_{i} is the onset time of the i-th frame
        Float lastDecayIn = luminance[lid]; // = [sum_i2n(F_i[exp(-t_{i+1}/Itau) - exp(-t_{i}/Itau)]) - F_{n+1}*exp(-t_{i+1}/Itau)]*exp(-t/Itau)
        Float F_1 = lastF[lid]; // = F_{n+1}
        //F_i is the mean of all sampled pixel value of the ith frame in the LGN's RF.
    */
    
    SmallSize typeS = coneType[blockIdx.x + 1*gridDim.x];
    SmallSize typeC = coneType[blockIdx.x + 0*gridDim.x];

    Float spatialWeightS = SW_storage[storeIDS];
    Float spatialWeightC = SW_storage[storeIDC];
    //initialize return value
    /* looping the following over (nPatch+1) patches on nKernelSample samples points:
        p - parallelized by all threads;
        n - needed by all threads;
        s - single thread 
    */
    // non-dimensionalized decay time-scale unit  for intensity
    Float I_unit = dt/denorm/Itau; // denorm is the co-divisor to compare frame with dt
    Size nPatch = nKernelSample/nSample;
    for (Size iPatch=0; iPatch<nPatch+1; iPatch++) {
        Size nActive;
        // for p in time, active (for temporal samples) threads only,
        if (iPatch == nPatch) { // no divergent branch
            nActive = nKernelSample % nSample;
        } else {
            nActive = nSample;
        }
        //1. Load temporal weights: p in time
        // forward in time, stored reversed in TW_storage
        // i.e., last in storage-> t-0 -^v-- t-tau <-first in storage,   
        // convolution time start at t-tau; first sample point: t-tau+kernelSampleT0;
        Size temporalWeightS = TW_storage[offsetS*nKernelSample + iPatch*nSample + tid] * (tid<nActive);  // tid < nActive, 0 otherwise
        Size temporalWeightC = TW_storage[offsetC*nKernelSample + iPatch*nSample + tid] * (tid<nActive);  // tid < nActive, 0 otherwise
        //2. Find new frames - n, usually just 1
        PosInt itFrames = (nActive-1)*kernelSampleInterval;
        if (iPatch == 0) {
            itFrames += iKernelSampleT0; // iKernelSampleT0 = 0 or kernelSampleInterval/2
        }
        // number of frames in one patch
        //Size nFrame = (itFrames*denorm + iFramePhase + (ntPerFrame-1)) / ntPerFrame - 1; // exclude the currentFrame within the framePhase, already in F_1
        Size nFrame = (itFrames*denorm + iFramePhase) / ntPerFrame + 1;
        if (blockIdx.x==0 && tid == 0) {
            if (iPatch == 0) {
                printf("nFrame: %u, maxFrame = %u\n", nFrame, maxFrame);
            }
            printf("framePhase = %u/%u\n", iFramePhase, ntPerFrame);
            //printf("itFrames = %u, nActive = %u, denorm = %u, iKernelSampleT0 = %u, kernelSampleInterval\n", iFramePhase, nActive, denorm, iKernelSampleT0, kernelSampleInterval);
            //assert(nFrame == maxFrame);
        }
        //3. For all the new frames
        for (Size iFrame = 0; iFrame < nFrame; iFrame++) {
            //Get F_i by reduce - p: in space
            Float local_I = get_intensity(3, x0S, y0S, (currentFrame + iFrame) % maxFrame);
            block_reduce<Float>(reducedS, local_I);
            if (tid == 0) {
                // __shared__ to (register/local) to __shared__
                nSampleShared[iFrame] = reducedS[0]/nSample;  // shared memory now used for spatial mean luminance, F_i
            }
        }
        __syncthreads();
        /*
            //!!! Update light adapation variables here to hide latency: p in space 
            if (iPatch == 0 && tid == 0) { // first dt must be in the first patch
                PosInt itf0 = denorm + iFramePhase;
                Float luminance_tmp = lastDecayIn*exponential(-dt);
                if ((itf0 + ntPerFrame-1)/ntPerFrame > 1) { // check if a new frame is introduced within a dt
                    // here we ASSUME tPerframe > dt, i.e., at most one change of frame happen within a single dt
                    Float F_2 = nSampleShared[0]; // the first frame in the patch
                    lastF[lid] = F_2;
                    Float exponent = (itf0%ntPerFrame)*I_unit; // needs this extra variable otherwise exponential returns inf (no idea why)
                    luminance_tmp += (F_1 - F_2) * exponential(-exponent); // == exp(-(dt+framePhase-tPerFrame)*I_unit)
                } // else lastF is not changed
                // *** if (ift0 + denorm == ntPerFrame) lastF is not stored and the first frame of next convolution should be recalculated
                luminance[lid] = luminance_tmp;
                assert(abs(luminance_tmp) < abs(lastDecayIn));
            }
            // TODO: old_frame contrast (lastDecayIn, F1) data can be stored(310mb for tau 250ms dt 0.1ms) not to be calculated every time, then we can fully parallelize this step
            //4. Calculate mean_I: p in time 
            if (tid < nActive && nFrame > 0) {
                PosInt itf = tid * kernelSampleInterval;
                if (iPatch == 0) {
                    itf += iKernelSampleT0;
                }
                itf *= denorm;
                Float lastDecayIn_tmp = lastDecayIn;
                if (isnan(lastDecayIn_tmp)) {
                    printf("oldlastDecayIn = %f\n", lastDecayIn_tmp);
                    assert(!isnan(lastDecayIn_tmp));
                }
                Float exponent = itf*I_unit; // needs this extra variable otherwise exponential returns inf (no idea why)
                lastDecayIn *= exponential(-exponent); // = [sum_i2n(F_i[exp(-t_{i+1}/Itau) - exp(-t_{i}/Itau)]) - F_{n+1}*exp(-t_{i+1}/Itau)]*exp(-t/Itau)
                itf += iFramePhase;
                Size local_nFrame = (itf + ntPerFrame-1) / ntPerFrame - 1; // exclude the currentFrame within the framePhase, already in F_1
                // this loop is actually where the most of the branch divergence happen, local_nFrame can be different for each thread;
                itf = itf % ntPerFrame;
                for (Size iFrame = 0; iFrame < local_nFrame; iFrame++) {
                    // number of active threads decreases for each loop
                    Float F_2 = nSampleShared[iFrame];
                    // F_{n+1} decayed with exp(-((t+t0) - t_{i+2})/Itau); time to current kernelSampleT from currentFrame input time
                    Float exponent = (itf+(local_nFrame-1-iFrame)*ntPerFrame)*I_unit; // needs this extra variable otherwise exponential returns inf (no idea why)
                    lastDecayIn += (F_1 - F_2) * exponential(-exponent);
                    F_1 = F_2;
                }
            }
            __syncthreads(); // make sure shared memory reads complete before reuse for other data
            if (tid < nActive) {
                nSampleShared[tid] = lastDecayIn + F_1; //shared memory now used as spatiotemporal mean luminance
            }
            // broadcast [lastDecayIn, F_1] for the next convolution step if not the final patch: TODO consider move to the end
            if (iPatch <= nPatch) {
                // nSample == nActive;
                if (tid == nActive-1) {
                    reduced[0] = lastDecayIn;
                    reduced[1] = F_1;
                }
                __syncthreads();
                // broadcast [lastDecayIn, F_1]
                lastDecayIn = reduced[0];
                F_1 = reduced[1];
            }
            __syncthreads();
        */
        //5. For each sample point in time: 
        //  Get contrast from local_I(ntensity) and mean_I(ntensity): p in space 
        Float filteredC, filteredS;
        // initialize with the first frame in the patch
        PosInt it = 0;
        if (iPatch == 0) {
            it = iKernelSampleT0;
        }
        PosInt iFrame = 0;
        Int preFrame = currentFrame-1;
        for (PosInt iSample = 0; iSample < nActive; iSample++) {
            PosInt frameNow = currentFrame + (it*denorm + iFramePhase)/ntPerFrame;
            if (frameNow > preFrame) { // advance frame
                //Load mean luminance from shared memory first
                Float mean_I = nSampleShared[iFrame];
                // surround 
                Float local_I = get_intensity(typeS, x0S, y0S, frameNow % maxFrame);
                assert(local_I > 0);
                if (tid == 0 && blockIdx.x == 0) {
                    printf("frameNow = %u\n", frameNow);
                }
                Float local_contrast;
                if (mean_I > 0) {
                    local_contrast = local_I/mean_I - 1.0;
                } else {
                    local_contrast = local_I;
                }
                if (abs(local_contrast) > 1.0) {
                    local_contrast = copyms(1.0, local_contrast); // copyms is copysign(value, sign);
                }
                block_reduce<Float>(reducedS, spatialWeightS*local_contrast);
                if (iPatch == nPatch && iFrame == nFrame && tid ==0) {
                    contrast[gridDim.x*1+blockIdx.x] = reducedS[0];
                    luminance[lidC] = mean_I;
                }
                // center
                local_I = get_intensity(typeC, x0C, y0C, frameNow % maxFrame);
                if (mean_I > 0) {
                    local_contrast = local_I/mean_I - 1.0;
                } else {
                    local_contrast = local_I;
                }
                if (abs(local_contrast) > 1.0) {
                    local_contrast = copyms(1.0, local_contrast); // copyms is copysign(value, sign);
                }
                block_reduce<Float>(reducedC, spatialWeightC*local_contrast);
                if (iPatch == nPatch && iFrame == nFrame && tid ==0) {
                    contrast[gridDim.x*0+blockIdx.x] = reducedC[0];
                    if (blockIdx.x == 0) {
                        printf("contrast0 = \n", reducedC[0]);
                    }
                }
                preFrame = frameNow;
                iFrame++;
            }
            if (tid == iSample) {
                filteredC = reducedC[0]*temporalWeightC; // shared memory have spatially convolved values 
                filteredS = reducedS[0]*temporalWeightS; // shared memory have spatially convolved values 
            }
            // advance time
            it += kernelSampleInterval;
        }
        //__syncthreads();
        //6. reduce sum with temporal weights: p in time
        if (tid >= nActive) {
            filteredS = 0.0;
            filteredC = 0.0;
        }
        block_reduce<Float>(reducedS, filteredS);
        //7. add to convol: s
        if (tid == 0) {
            convolS += reducedS[0];
        }
        block_reduce<Float>(reducedC, filteredC);
        if (tid == 0) {
            convolC += reducedC[0];
        }
        //9. advance [currentFrame, framePhase] if not the final patch: n
        if (iPatch < nPatch) {
            Size iPhaseIncrement = (nSample-1)*kernelSampleInterval;
            if (iPatch == 0) {
                iPhaseIncrement += iKernelSampleT0;
            }
            currentFrame += nFrame-1;
            iFramePhase = (iPhaseIncrement*denorm + iFramePhase) % ntPerFrame;
        }
    }
    if (tid == 0) {
        // times amplitude and space-time volume, k is amplitude*dwdh
        convolC *= kernelSampleInterval*dt*kC;
        convolS *= kernelSampleInterval*dt*kS;
    }

    Float convol = convolC + convolS;
    // update convolution data, initialized in LGN_nonlinear
    if (threadIdx.y*blockDim.x + threadIdx.x == 0) {
        atomicAdd(current_convol+blockIdx.x, convol);
    }
}

__launch_bounds__(1024, 2)
__global__ 
void LGN_nonlinear(
        Size nLGN,
        Static_nonlinear &logistic,
        Float* __restrict__ max_convol,
        Float* __restrict__ current_convol,
        Float* __restrict__ LGN_fr
) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    Float max, current;
    Float C50, K, A, B;
    bool engaging = id<nLGN;
    unsigned int MASK = __ballot_sync(FULL_MASK, static_cast<int>(engaging));
    if (engaging) {
        current = current_convol[id];
		// initialize for next time step
		current_convol[id] = 0.0;
		max = max_convol[id];
        logistic.load_first(id, C50, K, A, B);
        Float convol = current;
        if (current < 0) {
            current = 0;
        }
        __syncwarp(MASK);
        Float fr = max * transform(C50, K, A, B, current/max);
        if (fr < 0) {
            printf("convol = %f, fr = %f, K=%f, A= %f, B= %f, C50 =%f, max = %f\n", convol, fr, K, A, B, C50, max);
            assert(fr >= 0);
        }
        LGN_fr[id] = fr;
        //LGN_fr[id] = max * transform(C50, K, A, B, current/max);
    }
}
