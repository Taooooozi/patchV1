#include "hip/hip_runtime.h"
#include "discrete_input_convol.h"

// TODO-10:
// block-processing without input-output delay
// http://www.cs.ust.hk/mjg_lib/bibs/DPSu/DPSu.Files/Ga95.PDF
// block processing file:///C:/Users/gueux/Desktop/FFTConvolution.pdf

// DIRECT FORM only:

__device__ 
__inline__ 
_float AexpTau(_float a, _float tau) {
    return a * expp(-tau);
}

__device__ 
__inline__ 
_float spatialKernel(Float x, Float y, Float rx, Float ry) {
    return expp(-x*x/(rx*rx) - y*y/(ry*ry));
}

__device__ 
__inline__ 
_float temporalKernel(Float tau, Temporal_component &temp, Float fac1, Float fac2) {
    Float tau1 = tau/temp.tauR;
    Float tau2 = tau/temp.tauD;
    Float A1 = power(tau1, temp.nR-1)/(temp.tauR * fac1);
    Float A2 = power(tau2, temp.nD-1)/(temp.tauD * fac2);

    Float tp = AexpTau(A1, tau1) - temp.ratio*AexpTau(A2, tau2);
    return tp;
}


__device__
__inline__
void get_coord_in_plane(Float xsig, Float ysig, Float cx, Float cy, Float nsig, SmallSize nx, SmallSize ny, float &x0, float &y0, Float &dxdy, bool mainThread) {
    // make change consistent with the same part in store_spatialWeight
    Float xhspan = nsig * xsig / sqrt2;
    Float dx = 2*xhspan/nx;

    Float yhspan = nsig * ysig / sqrt2;
    Float dy = 2*yhspan/ny;

    x = (threadIdx.x + 0.5)*dx - xhspan;
    y = (threadIdx.y + 0.5)*dy - yhspan;

    // texture coords have to be float
    retina_to_plane(cx+x, cy+y, x0, y0);
    if (mainThread) {
        dxdy = dx*dy;
    }
}

__device__
__inline__
void retina_to_plane(Float x0, Float y0, float &x, float &y) {
    Float r = sqrt(x0*x0 + y0*y0);
    Float tanr = -tan(r);
    Float xr = x0/r;
    Float yr = x0/r;
    x = static_cast<float>(xr*tanr);
    y = static_cast<float>(yr*tanr);
}

__device__ 
__inline__
Float get_intensity(unsigned int coneType, float x, float y, unsigned int iLayer) {
    Float contrast;
    switch (coneType) {
        case 0:
            contrast = static_cast<Float>(tex2DLayered(L_retinaConSig, x, y, iLayer));
            break;
        case 1:
            contrast = static_cast<Float>(tex2DLayered(M_retinaConSig, x, y, iLayer));
            break;
        case 2:
            contrast = static_cast<Float>(tex2DLayered(S_retinaConSig, x, y, iLayer));
            break;
        case 3:
            contrast = static_cast<Float>(tex2DLayered(L_retinaConSig, x, y, iLayer) 
                                        + tex2DLayered(M_retinaConSig, x, y, iLayer) 
                                        + tex2DLayered(S_retinaConSig, x, y, iLayer))/3.0;
            break;
        default:
            printf("unrecognized cone type");
        /*
        case 4:
            break;
        case 5:
            break;
        case 6:
            break;
        */
    }
    return contrast;
}

// for 2 cone-types LGN only, can be generalized for more cone-types
// gridSize: (nLGN, nType) blocks for store 1-D nLGN for convol
// blockSize: spatialSample1D x spatialSample1D (npixel_1D)

/* TODO: compare all thread load from global memory vs shared
    __device__
    __inline__
    void store_spatialWeight0(
            Spatial_component &spatial,
            Float* __restrict__ SW_storage
            Float* __restrict__ SC_storage
            Float xhspan, 
            Float yhspan,
            Float dx,
            Float dy,
            Float nsig, // span of spatialRF sample in units of std
            Size id,
            Size lid,
            Size iType,
            Size nType,
    ) {
        // parameter are stored as (nType, nLGN), weights are stored as (nLGN, nType, weight)
        Zip_spatial spat;
        spat.load(spatial, lid);
        SmallSize nSample = blockDim.x * blockDim.y;
        Size offset = (id*nType + iType)*nSample;
        Size tid = threadIdx.y*blockDim.x + threadIdx.x;
        Size storeID = offset + tid;
        Size xID = storeID + offset;
        Size yID = xID + nSample;
    
        // coord to center
        Float x = (threadIdx.x + 0.5)*dx - xhspan;
        Float y = (threadIdx.y + 0.5)*dy - yhspan;
        Float sample_vol = dx * dy;
    
        SW_storage[storeID] = spatialKernel(x, y, k, spat.rx, spat.ry)*sample_vol;
        Float x_plane, y_plane;
        retina_to_plane(cx + x, cy + y, x_plane, y_plane);
        SC_storage[xID] = x_plane;
        SC_storage[yID] = y_plane;
    }
*/

// iType is for center surround (or multiple surroud) in a single LGN
// not to be confused with coneTypes

struct shared_spat {
    Float xhspan, yhspan, dx, dy, cx, cy, rx, ry;
    shared_spat(Float *spat) {
        xhspan = spat[0];
        yhspan = spat[1];
        dx = spat[2];
        dy = spat[3];
        cx = spat[4];
        cy = spat[5];
        rx = spat[6];
        ry = spat[7];
    }
}

__device__
__inline__
void store_temporalWeight(
        Temporal_component &temporal,
        Float* __restrict__ TW_storage,
        Float* __restrict__ reduced, //__shared__
        Float &temporalWeight,
        SmallSize nKernelSample,
        Float kernelSampleDt,
        Float kernelSampleT0,
        Size id,
        Size tid,
        Size lid,
        SmallSize iType,
        SmallSize nType
) {
    Zip_temporal temp;
    // load temporal parameters
    temp.load(temporal, lid);
    SmallSize patchSize = blockDim.x*blockDim.y;
    SmallSize nPatch = nKernelSample/patchSize;
    SmallSize remain = nKernelSample%patchSize;
    
    Float fac1, fac2;
    fac1 = tgamma(temp.nR);
    fac2 = tgamma(temp.nD);
    

    for (SmallSize iPatch = 0; iPatch < nPatch+1; iPatch++) {
        Float tw;
        if (iPatch < nPatch || tid < remain) {
            // temporalKernel takes abs(now-t)
            // but we store weight in time-reverse (tau -> 0)
            SmallSize twid = nKernelSample-1 - iPatch*patchSize - tid;
            Size storeID = (id*nType + iType)*nKernelSample + iPatch*patchSize + tid;

            Float t = twid*kernelSampleDt + kernelSampleT0
            tw = temporalKernel(t, temp, fac1, fac2);
            TW_storage[storeID] = tw;
            if (tw < 0) tw = -tw;
        } else {
            tw = 0.0;
        }
        block_reduce<Float>(reduced, tw);
        if (tid == 0) {
            temporalWeight += reduced[0];
        }
    }
}

__device__
__inline__
Float store_spatialWeight(
        shared_spat &ss,
        Float* __restrict__ SW_storage,
        Float* __restrict__ SC_storage,
        Float nsig, // span of spatialRF sample in units of std
        Size id,
        Size tid,
        Size offset, // (id*nType + iType) * nSample
        Size nSample,
        bool storeSpatial
) {
    // parameter are stored as (nType, nLGN), weights are stored as (nLGN, nType, weight)
    Size storeID = offset + tid;
    Size xID = offset + storeID;
    Size yID = xID + nSample;

    // coord to center
    Float x = (threadIdx.x + 0.5)*ss.dx - ss.xhspan;
    Float y = (threadIdx.y + 0.5)*ss.dy - ss.yhspan;

    spatialWeight = spatialKernel(x, y, ss.rx, ss.ry);
    
    if (storeSpatial) {
        SW_storage[storeID] = spatialWeight;
        Float x_plane, y_plane;
        retina_to_plane(ss.cx + x, ss.cy + y, x_plane, y_plane);
        SC_storage[xID] = x_plane;
        SC_storage[yID] = y_plane;
    }
    return spatialWeight;
}

// weights are stored in shapes of (nLGN, nType, weight)
__global__
void store(
        Float* __restrict__ max_convol,

        Temporal_component* __restrict__ temporal,
        Float* __restrict__ TW_storage,
        SmallSize nKernelSample,
        Float kernelSampleDt,
        Float kernelSampleT0,

        Spatial_component* __restrict__ spatial,
        Float* __restrict__ SW_storage,
        Float* __restrict__ SC_storage,
        Float* __restrict__ dxdy_storage,
        Float nsig, // span of spatialRF sample in units of std
        bool storeSpatial
) {
    __shared__ Float reduced[warpSize]; 
    __shared__ Float spat[8]; // xhspan, yhspan, dx, dy, cx, cy, rx, ry
    Size id = blockIdx.x;
    SmallSize iType = blockIdx.y;
    Size lid = iType*gridDim.x + id;
    SmallSize nType = gridDim.y;
    Size tid = threadIdx.y*blockDim.x + threadIdx.x;
    SmallSize nSample = blockDim.x * blockDim.y;

    Float temporalWeight, spatialWeight, dxdy, k;
    store_temporalWeight(temporal, TW_storage, reduced, temporalWeight, nKernelSample, kernelSampleDt, kernelSampleT0, id, tid, lid, iType, nType);

    bool use_shared = true;
    if (use_shared) { // TODO: compare with global broadcast
        Size offset = id*nType + iType;
        if (tid == 0) {
            Zip_spatial spat0;
            spat0.load(spatial, lid);
            Float xhspan = nsig * spat0.rx / sqrt2;
            Float dx = 2*xhspan/blockDim.x;

            Float yhspan = nsig * spat0.ry / sqrt2;
            Float dy = 2*yhspan/blockDim.y;
            dxdy = dx*dy;
            k = spat0.k
            // store dxdy
            dxdy_storage[offset] = dxdy;

            spat[0] = xhspan; spat[1] = yhspan; spat[2] = dx; spat[3] = dy; spat[4] = spat0.x; spat[5] = spat0.y; spat[6] = spat0.rx; spat[7] = spat0.ry;
        }
        __syncthreads();
        // load from shared mem
        shared_spat ss(spat);
        spatialWeight = store_spatialWeight(ss, SW_storage, SC_storage, nsig, id, tid, offset*nSample, nSample, storeSpatial);
    } 
    block_reduce<Float>(reduced, spatialWeight)

    if (tid == 0) { // iType = 0, 1
        atomicAdd(max_convol+id, reduced[0] * temporalWeight * k * dxdy * kernelSampleDt);
    }
}

__device__
void sub_convol(
        SmallSize type,
        PosInt nsig,
        SmallSize currentFrame,
        SmallSize maxFrame,
        Float framePhase,
        Float Itau,
        Float kernelSampleDt,
        Size nKernelSample,
        Float &convol, // for returning the convol value
        Spatial_component &spatial,
        Float* __restrict__ decayIn,
        Float* __restrict__ lastF,
        Float* __restrict__ reduced, // shared mem ptr for block_reduce
        Float* __restrict__ nSampleShared, // shared mem ptr for block_reduce
        Float* __restrict__ SW_storage,
        Float* __restrict__ SC_storage,
        Float* __restrict__ dxdy_storage,
        Float* __restrict__ TW_storage,
        Size lid,
        Size tid,
        SmallSize iType,
        SmallSize nType,
        Float dt,
        bool spatialStored
) {
    /* kernel sampling diagram with frames
                      [,)
        frame:       curr            next              next+1
        framePhase|tPerFrame-framePhase                  ^        
        time:  <--|   ^   |-------> tPerFrame <------|   ^    |
                now-tau                                       now
        sample:           1       2       3       4       5     
                  |...|...:---|-------|-------|------:|---|---|
        dt:     -4   0    4   8       16      24  v   32    v 40
                  v                               Dt        T0 
         lastDecay|nextDecay
        e.g., tau = 40*dt
    */
    Size id = blockIdx.x;
    SmallSize nSample = blockDim.x * blockDim.y;

    Float dxdy, k;
    Float spatialWeight;
    float x0, y0; // coord on the stimulus plane
    Size offset0 = (id*nType + iType);
    if (spatialStored) {
        Size offset = offset0*nSample;
        Size storeID = offset + threadIdx.y*blockDim.x + threadIdx.x;
        Size xID = storeID + offset;
        Size yID = xID + nSample;
        spatialWeight = SW_storage[storeID];
        x0 = SC_storage[xID];
        y0 = SC_storage[yID];
        if (tid == 0) {
            dxdy = dxdy_storage[offset0];
            k = spatial.k[lid];
        }
    } else {
        Zip_spatial spat;
        spat.load(spatial, lid);
        // texture coords have to be float
        get_coord_in_plane(spat.rx, spat.ry, spat.x, spat.y, nsig, blockDim.x, blockDim.y, x0, y0, dxdy, tid == 0); // dxdy is only given to the tid == 0
        spatialWeight = spatialKernel(x, y, spat.rx, spat.ry);
        if (tid == 0) {
            k = spat.k;
        }
    }
    /* Light adaptation process:
        tau*dI/dt = -I + F(t);
        F(t) = piecewise F(t_{i}), for t in [t_{i}, t_{i+1}), t_{i} is the onset time of the i-th frame
    */
    Size offset = gridDim.x*iType + id;
    Float lastDecayIn = decayIn[offset]; // = sum_i2n(F_i[exp(-t_{i+1}/Itau) - exp(-t_{i}/Itau)] - F_{n+1})*exp(-(t-tau)/Itau)
    Float F_1 = lastF[offset]; // = F_{n+1}
    //F_i is the mean of all sampled pixel value of the ith frame in the LGN's RF.

    /* looping the following over (nPatch + 1) patches on nKernelSample samples points:
        p - parallelized by all threads;
        n - needed by all threads;
        s - single thread 
     */
    SmallSize nPatch = nKernelSample/nSample;
    patchSize
    for (SmallSize iPatch=0; iPatch<nPatch+1; iPatch++) {
        Float temporalWeight;
        SmallSize nActive;
        // for p in time, active threads only,
        if (iPatch == nPatch) {
            nActive = nKernelSample - iPatch*nSample;
        } else {
            nActive = nSample;
        }
        //1. Load temporal weights: p in time
        if (tid < nActive) {
            //convolution time start at t-tau, forward in time, reversed in temporalKernel; first sample point: t-tau+kernelSampleT0
            temporalWeight = TW_storage[iPatch*nSample + tid]; 
        }
        //2. Find new frames - n
        Float tFrames = nActive*kernelSampleDt + framePhase;
        SmallSize nFrame = static_cast<SmallSize>(tFrames/tPerFrame);
        //3. For all the new frames
        for (SmallSize iFrame = 0; iFrame < nFrame; iFrame++) {
            //Get F_i by reduce - p: in space
            Float local_I = get_intensity(type, x0, y0, (currentFrame + iFrame + 1) % maxFrame);
            block_reduce<Float>(reduced, local_I);
            if (tid == 0) {
                // __shared__ to (register/local) to __shared__
                nSampleShared[iFrame] = reduced[0]/nSample;  // shared memory now used for spatial luminance sum
            }
            __syncthreads();
        }
        //!!! Update light adapation variables here to hide latency: p in space 
        if (iPatch == 0 && tid == 0) { // dt < kernelSampleDt, must be in the first patch
            Float tf0 = dt + framePhase;
            if (tf0 < tPerFrame) {
                //lastF is not changed
                decayIn[offset] = lastDecayIn*expp(-dt/Itau);
            } else {
                // here we ASSUME tPerframe > dt, i.e., at most one change of frame happen within a single dt
                Float F_2 = nSampleShared[0];
                decayIn[offset] = lastDecayIn*expp(-dt/Itau) + (F_1 - F_2) * expp(-(tf0- tPerFrame)/Itau);
                lastF[id] = F_2;
            }
        }
        //4. Calculate mean_I: p in time 
        if (tid < nActive) {
            Float t = (tid+1)*kernelSampleDt;
            Float tf0 = t+framePhase;
            Size local_nFrame = static_cast<Size>(tf0/tPerFrame);
            // if nFrame == 0 then F_{n+1} (F_1) is not changed
            lastDecayIn *= expp(-t/Itau); // sum_i2n(F_i[exp(-t_{i+1}/Itau) - exp(-t_{i}/Itau)] - F_{n+1})*exp(-(t0-tau)/Itau) decayed to t = t0 + t
            for (SmallSize iFrame = 0; iFrame < local_nFrame; iFrame++) {
                // number of active threads decreases for each loop
                Float F_2 = nSampleShared[iFrame]; // load from shared memory to register first // TODO: check register usage here 
                tf0 -= tPerFrame;
                Float decay = expp(-tf0/Itau);
                lastDecayIn += (F_1 - F_2) * decay; // F_{n+1} decayed with exp(-((t+t0) - t_{i+2})/Itau);
                F_1 = F_2;
            }
            nSampleShared[tid] = lastDecayIn + F_1; //shared memory now used as mean luminance
        }
        __syncthreads();
        //5. For each sample point in time: 
        //  Get contrast from local_I(ntensity) and mean_I(ntensity): p in space 
        SmallSize iFrame = static_cast<SmallSize>((kernelSampleDt + framePhase)/tPerFrame);
        Float local_I = get_intensity(type, x0, y0, (currentFrame + iFrame) % maxFrame);
        for (SmallSize iSample = 0; iSample < nSample; iSample++) {
            //Load mean luminance from shared memory first
            Float mean_I = nSampleShared[iSample];
            SmallSize frameNow = static_cast<SmallSize>((iSample*kernelSampleDt + framePhase)/tPerFrame);
            if (frameNow > iFrame) {
                // new frame, in case frame rate > sample rate, we don't increase iFrame in single units
                iFrame = frameNow;
                local_I = get_intensity(type, x0, y0, (currentFrame + iFrame) % maxFrame);
            }
            Float local_contrast = (local_I-mean_I)/mean_I;
            if (abs(local_contrast) > 1.0) {
                local_contrast = copy(1.0, local_contrast); // copy is copysign(value, sign);
            }
            block_reduce<Float>(reduced, spatialWeight*local_contrast);
            if (tid == 0) {
                // __shared__ to (register/local) to __shared__
                nSampleShared[iSample] = reduced[0]; // shared memory now stores spatially convolved values
            }
        }
        //6. reduce sum with temporal weights: p in time
        Float filtered;
        if (tid < nActive) {
            filtered = nSampleShared[tid]*temporalWeight; // shared memory have spatially convolved values 
        } else {
            filtered = 0.0f;
        }
        block_reduced<Float>(reduced, filtered);
        //7. add to convol: s
        if (tid == 0) {
            convol += reduced[0];
        }
        //9 .advance currentFrame and framePhase if not the final patch: n
        if (iPatch < nPatch) {
            currentFrame += nFrame;
            framePhase = fmod(nActive*kernelSampleDt + framePhase, tPerFrame);
        }
    }
    if (tid == 0) {
        // times amplitude and space-time volume
        convol *= kernelSampleDt*dxdy*k;
    }
}

// grid: [nLGN, 1, 1]
// block: [nSpatialSample1D, nSpatialSample1D, 1]
__global__ 
void LGN_convol_c1s(
        Float* __restrict__ decayIn,
        Float* __restrict__ lastF,
        Float* __restrict__ SW_storage,
        Float* __restrict__ SC_storage,
        Float* __restrict__ dxdy_storage,
        Float* __restrict__ TW_storage,
        Float* __restrict__ LGNfr,
        SmallSize* __restrict__ coneType,
        Spatial_component &spatial,
        Float nsig,
        SmallSize currentFrame,
        SmallSize maxFrame,
        Float framePhase,
        Float Itau,
        Float kernelSampleDt,
        Size nKernelSample,
        Float dt,
        bool spatialStored
) {
    __shared__ Float reduced[warpSize];
    SmallSize nSample = blockDim.x*blockDim.y;
    extern __shared__ Float nSampleShared[];
    unsigned int id = blockIdx.x;
    SmallSize type = coneType[id];
    unsigned int tid = threadIdx.y*blockDim.x + threadIdx.x;

    // weights are stored in shapes of (nLGN, nType, weight)

    Float convol;
    if (tid == 0) {
        convol = 0.0f;
    }

    //TODO: Itau may take different value for different cone type
    // convolve center and update decayIn, lastF
    sub_convol(type, nsig, currentFrame, maxFrame, framePhase, Itau, kernelSampleDt, nKernelSample, convol, spatial, decayIn, lastF, reduced, nSampleShared, SW_storage, SC_storage, dxdy_storage, TW_storage, lid, tid, 0, 2, dt, spatialStored);

    type = coneType[id + gridDim.x];

    // convolve surround and add to convol and update decayIn, lastF
    sub_convol(type, nsig, currentFrame, maxFrame, framePhase, Itau, kernelSampleDt, nKernelSample, convol, spatial, decayIn, lastF, reduced, nSampleShared, SW_storage, SC_storage, dxdy_storage, TW_storage, lid, tid, 1, 2, dt, spatialStored);

    // update convolution data 
    if (tid == 0) {
        LGNfr[id] = convol;
    }
}

__global__ 
void LGN_nonlinear(
        Static_nonlinear &logistic,
        Float* __restrict__ max_convol,
        Float* __restrict__ LGN_fr
) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	Float _max_convol = max_convol[id];
	// min = -max;
	_float current_convol = LGN_fr[id];
    if (current_convol < 0) {
        current_convol = 0;
    }
    __syncwarp(); // check necessity

    Float ratio = logistic.transform(id, current_convol/_max_convol);
    LGN_fr[id] = current_convol * ratio;
}

