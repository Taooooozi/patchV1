#include "hip/hip_runtime.h"
#include "coredynamics.cuh"

__forceinline__  __device__ double get_a(double gE, double gI, double gL) {
    return gE + gI + gL;
}

__forceinline__  __device__ double get_b(double gE, double gI, double gL) {
    return gE * vE + gI * vI + gL * vL;
}

template <typename T>
__device__ __forceinline__ void warpReduce(volatile T* data, int id, int halfLen) {
    #pragma unroll
    for (unsigned int i=halfLen; i>0; i>>=1) { 
        data[id] += data[id + i];
    }
}

template <typename T>
__device__ __forceinline__ void reduce2(volatile T *g, volatile T *h, int id, int halfLen) {
    if (halfLen >= 32) {
        #pragma unroll 
        for (unsigned int i=halfLen; i>32; i>>=1) { 
            if (id < i) {
                // keep data stored sequentially
                g[id] += g[id + i];
                h[id] += h[id + i];
            }
            __syncthreads();
        }
        // warp size no need to __syncthreads
        if (id < 32) {
            warpReduce(g, id, halfLen);
            warpReduce(h, id, halfLen);
        }
    } else {
        if (id < halfLen) {
            warpReduce(g, id, halfLen);
            warpReduce(h, id, halfLen);
        }
    }
}

template <typename T>
__global__ void partial_dot1d(T* x, T* y1, T* y2, T* g, T* h, int size) {
    extern __shared__ T product[];
    T *product_g = product;
    T *product_h = &(product_g[size]) ;
    unsigned int blockLen = blockDim.x;
    unsigned int blockLen_half = blockDim.x/2;
    // thread index
    unsigned int block_id = threadIdx.x;
    unsigned int grid_id = blockIdx.x;
    unsigned int global_id = grid_id*(2*blockLen) + block_id;
    // elmenent-wise product to shared memory
    product_g[block_id] = x[global_id]*y1[global_id] + x[global_id + blockLen] * y1[global_id+blockLen];
    product_h[block_id] = x[global_id]*y2[global_id] + x[global_id + blockLen] * y2[global_id+blockLen];
    __syncthreads();
    // reduction within block

    //if (block_id == 0 ) {
    //    printf("blockLen/2 %i, dataSize %i \n",  blockLen_half, size);
    //}
    reduce2<T>(product_g, product_h, block_id, blockLen_half);

    if (block_id == 0) {
        g[grid_id] = product_g[0];
        h[grid_id] = product_h[0];
    }
}

template <typename T>
__global__ void final_reduce(T* pg, T* ph, T* g, T* h, int size) {
    extern __shared__ T partials[];
    T* partial_g = partials;
    T* partial_h = &(partial_g[size]);
    unsigned int blockLen = blockDim.x;
    unsigned int blockLen_half = blockDim.x/2;
    // thread index
    unsigned int block_id = threadIdx.x;
    unsigned int global_id = blockIdx.x*(2*blockLen) + block_id;
    // elmenent-wise product to shared memory
    partial_g[block_id] = pg[global_id] + pg[global_id + blockLen];
    partial_h[block_id] = ph[global_id] + ph[global_id + blockLen];
    __syncthreads();
    // reduction within block
    reduce2<T>(partial_g, partial_h, block_id, blockLen_half);

    if (block_id == 0) {
        (*g) += partial_g[0];
        (*h) += partial_h[0];
        //printf("r1 = %f, r2 = %f \n", partial_g[0], partial_h[0]);
    }
}

__global__ void recal_G(double* __restrict__ gE,
                        double* __restrict__ gI,
                        double* __restrict__ hE,
                        double* __restrict__ hI,
                        double* __restrict__ preMat,
                        double* __restrict__ gactVecE,
                        double* __restrict__ hactVecE,
                        double* __restrict__ gactVecI,
                        double* __restrict__ hactVecI,
                        double* __restrict__ gEproduct_b1,
                        double* __restrict__ hEproduct_b1,
                        double* __restrict__ gIproduct_b1,
                        double* __restrict__ hIproduct_b1,
                        unsigned int networkSize, unsigned int ngTypeE, unsigned int ngTypeI, unsigned int b1, unsigned int b2
                        ) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        unsigned int aid = networkSize*ig;
        unsigned int bid = b1*ngTypeE*id + b1*ig;
        unsigned int gid = aid+id;
        partial_dot1d<double><<<b1, b2, b2*2*sizeof(double)>>>(&(preMat[id]), &(gactVecE[aid]), &(hactVecE[aid]), &(gEproduct_b1[bid]), &(hEproduct_b1[bid]),b2);
        //d_CUDA_CHECK();
        final_reduce<double><<<1, b1/2, b1*sizeof(double)>>>(&(gEproduct_b1[bid]), &(hEproduct_b1[bid]), &(gE[gid]), &(hE[gid]), b1/2);
        //d_CUDA_CHECK();
    }
    //printf("id-%i: %f -> %f\n", id, bgE, gE[id]);
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        unsigned int aid = networkSize*ig;
        unsigned int bid = b1*ngTypeI*id + b1*ig;
        unsigned int gid = aid+id;
        partial_dot1d<double><<<b1, b2, b2*2*sizeof(double)>>>(&(preMat[id]), &(gactVecI[aid]), &(hactVecI[aid]), &(gIproduct_b1[bid]), &(hIproduct_b1[bid]), b2);
        //d_CUDA_CHECK();
        final_reduce<double><<<1, b1/2, b1*sizeof(double)>>>(&(gIproduct_b1[bid]), &(hIproduct_b1[bid]), &(gI[gid]), &(hI[gid]), b1/2);
        //d_CUDA_CHECK();
    }
}

__global__ void logRand_init(double *logRand, hiprandStateMRG32k3a *state, unsigned long long seed) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed, id, 0, &localState);
    logRand[id] = -log(hiprand_uniform_double(&localState));
	cuPrintf("%f\n", logRand[id]);
    //logRand[id] = 1.0f;
    state[id] = localState;
}

__device__ int set_input_time(double inputTime[],
                              double dt,
                              double rate,
                              double *leftTimeRate,
                              double *lastNegLogRand,
                              hiprandStateMRG32k3a* __restrict__ state) {
    int i = 0;
    double tau, dTau, negLogRand;
    tau = (*lastNegLogRand - (*leftTimeRate))/rate;
    if (tau > dt) {
        *leftTimeRate += (dt * rate);
        return i;
    } else do {
        inputTime[i] = tau;
        negLogRand = -log(hiprand_uniform_double(state));
        dTau = negLogRand/rate;
        tau += dTau;
        i++;
        if (i == MAX_FFINPUT_PER_DT) {
            printf("exceeding max input per dt %i\n", MAX_FFINPUT_PER_DT);
            //printf("inputTime: %f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", inputTime[0], inputTime[1], inputTime[2], inputTime[3], inputTime[4], inputTime[5], inputTime[6], inputTime[7], inputTime[8], inputTime[9]);
            break;
        }
    } while (tau <= dt);
    *lastNegLogRand = negLogRand;
    *leftTimeRate = (dt - tau + dTau) * rate;
    return i;
}

__host__ __device__ void evolve_g(ConductanceShape &cond,
                                  double* __restrict__ g, 
                                  double* __restrict__ h, 
                                  double* __restrict__ f,
                                  double inputTime[],
                                  unsigned int nInput, double dt, unsigned int ig
                                  ) {

    cond.decay_conductance(g, h, dt, ig); 
    for (int i=0; i<nInput; i++) {
        cond.compute_single_input_conductance(g, h, *f, dt-inputTime[i], ig);
    }
}

 __device__  double step(Func_RK2* lif, double dt, double tRef) {
    lif->tsp = -1.0f;
    if (lif->tBack <= 0.0f) {
        // not in refractory period
        lif->runge_kutta_2(dt);
        if (lif->v > vT) {
            // crossed threshold
            lif->tsp = lif->compute_spike_time(dt); 
            lif->tBack = lif->tsp + tRef;
            //printf("neuron #%i fired initially\n", id);
        }
    } 
    // return from refractory period
    if (lif->tBack > 0.0f && lif->tBack < dt) {
        lif->compute_pseudo_v0(dt);
        lif->runge_kutta_2(dt);
        lif->tBack = -1.0f;
    } 
    // during refractory period
    if (lif->tBack > dt) {
        lif->reset_v(); 
        lif->tBack -= dt;
    }
    return lif->tsp;
}

__global__ void compute_V(double* __restrict__ v,
                          double* __restrict__ gE,
                          double* __restrict__ gI,
                          double* __restrict__ hE,
                          double* __restrict__ hI,
                          double* __restrict__ a,
                          double* __restrict__ b,
                          double* __restrict__ preMat,
                          double* __restrict__ inputRate,
                          int* __restrict__ eventRate,
                          double* __restrict__ spikeTrain,
                          double* __restrict__ tBack,
                          double* __restrict__ gactVecE,
                          double* __restrict__ hactVecE,
                          double* __restrict__ gactVecI,
                          double* __restrict__ hactVecI,
                          double* __restrict__ fE,
                          double* __restrict__ fI,
                          double* __restrict__ leftTimeRate,
                          double* __restrict__ lastNegLogRand,
                          hiprandStateMRG32k3a* __restrict__ state,
                          unsigned int ngTypeE, unsigned int ngTypeI, ConductanceShape condE, ConductanceShape condI, double dt, unsigned int networkSize, unsigned int nE, unsigned long long seed) {

    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    // if #E neurons comes in warps (size of 32) then there is no branch divergence.
    double gL, tRef;
    if (id < nE) {
        tRef = tRef_E;
        gL = gL_E;
    } else {
        tRef = tRef_I;
        gL = gL_I;
    }
    LIF lif(v[id], tBack[id]);
    /* set a0 b0 for the first step */
    double gI_t;
    double gE_t;
    // init cond E 
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gE_t += gE[networkSize*ig + id];
    }
    //  cond I 
    gI_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gI_t += gI[networkSize*ig + id];
    }
    lif.set_p0(gE_t, gI_t, gL);

    /* Get feedforward input */
    // consider use shared memory for dynamic allocation
    double inputTime[MAX_FFINPUT_PER_DT];
    hiprandStateMRG32k3a localState = state[id];
    int nInput;
    //if (init) {
    //    nInput = 1;
    //    inputTime[0] = dt*0.9f;
    //    lastNegLogRand[id] = 1.0f;
    //    leftTimeRate[id] = 0.0f;
    //} else {
        nInput = set_input_time(inputTime, dt, inputRate[id], &(leftTimeRate[id]), &(lastNegLogRand[id]), &localState);
    //}
    // return a realization of Poisson input rate
    eventRate[id] = nInput;
    // update rng state 
    state[id] = localState;
    /* evolve g to t+dt with ff input only */
    unsigned int gid;
    gE_t = 0.0f;
    #pragma unroll
    for (int ig=0; ig<ngTypeE; ig++) {
        gid = networkSize*ig + id;
        double g_i = gE[gid];
        double h_i = hE[gid];
        double f_i = fE[gid];
        evolve_g(condE, &g_i, &h_i, &f_i, inputTime, nInput, dt, ig);
        gE_t += g_i;
        gE[gid] = g_i;
        hE[gid] = h_i;
        // for learning
        //fE[gid] = f_i;
    }
    //printf("id %i, exc cond ready.\n",id);
    gI_t = 0.0f;
    /* no feed-forward inhibitory input (setting nInput = 0) */
    #pragma unroll
    for (int ig=0; ig<ngTypeI; ig++) {
        gid = networkSize*ig + id;
        double g_i = gI[gid];
        double h_i = hI[gid];
        double f_i = fI[gid];
        evolve_g(condI, &g_i, &h_i, &f_i, inputTime, 0, dt, ig);
        gI_t += g_i;
        gI[gid] = g_i;
        hI[gid] = h_i;
        // for learning
        //fI[gid] = f_i;
    }
    lif.set_p1(gE_t, gI_t, gL);
    // rk2 step

    spikeTrain[id] = step(&lif, dt, tRef);
	v[id] = lif.v;
    tBack[id] = lif.tBack;

    //setup acting vectors
    double g_end, h_end;
    int spiked = 0;
    if (id < nE) {
        #pragma unroll
        for (int ig=0; ig<ngTypeE; ig++) {
            g_end = 0.0;
            h_end = 0.0;
            if (spikeTrain[id]>0.0f) {
                condE.compute_single_input_conductance(&g_end, &h_end, 1.0f, dt-lif.tsp, ig);
                spiked = 1;
            }
            gid = networkSize*ig+id;
            gactVecE[gid] = spiked*g_end;
            hactVecE[gid] = spiked*h_end;
        }
    } else {
        #pragma unroll
        for (int ig=0; ig<ngTypeI; ig++) {
            g_end = 0.0;
            h_end = 0.0;
            if (spikeTrain[id]>0.0f) {
                condI.compute_single_input_conductance(&g_end, &h_end, 1.0f, dt-lif.tsp, ig);
                spiked = 1;
            }
            gid = networkSize*ig+id;
            gactVecI[gid] = spiked*g_end;
            hactVecI[gid] = spiked*h_end;
        }
    }
    //printf("id-%i, gend %f, hend %f, spiked %i \n",id, g_end, h_end, spiked);
    //if (id == 0) {
    //    printf("fml\n");
    //}
}

__device__ void Func_RK2::runge_kutta_2(double dt) {
    double fk0 = eval0(v0);
    double fk1 = eval1(v0 + dt*fk0);
    v = v0 + dt*(fk0+fk1)/2.0f;
}


__device__ double LIF:: compute_spike_time(double dt) {
    return (vT-v0)/(v-v0)*dt;
}

__device__ void LIF:: compute_pseudo_v0(double dt) {
    v0 = (vL-tBack*(b0 + b1 - a1*b0*dt)/2.0f)/(1.0f+tBack*(-a0 - a1 + a1*a0*dt)/2.0f);
    runge_kutta_2(dt);
}


__device__ void LIF::set_p0(double gE, double gI, double gL) {
    a0 = get_a(gE, gI, gL);
    b0 = get_b(gE, gI, gL); 
}

__device__ void LIF::set_p1(double gE, double gI, double gL) {
    a1 = get_a(gE, gI, gL);
    b1 = get_b(gE, gI, gL); 
}

inline  __host__ __device__ double eval_LIF(double a, double b, double v) {
    return -a * v + b;
}

__device__ double LIF:: eval0(double _v) {
    return eval_LIF(a0,b0,_v);
}
__device__ double LIF:: eval1(double _v) {
    return eval_LIF(a1,b1,_v);
}

__device__ void LIF:: reset_v() {
    v = vL;
}
