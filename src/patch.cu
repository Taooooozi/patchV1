#include "hip/hip_runtime.h"
#include "patch.h"

int main(int argc, char *argv[])
{
    std::ofstream v_file, spike_file, gE_file, gI_file;
    float time;
    //hipEventCreateWithFlags(&gReady, hipEventDisableTiming);
    hiprandStateMRG32k3a *state;
    unsigned long long seed;
    //seed = 183765712;
    seed = std::time(0);
    int device;
    int b1,b2;
	int ms = 1;
    b1 = 160;
    b2 = 128;
    bool printStep = false;
    bool moreSharedMemThanBlocks = true;
    unsigned int nstep = 200;
    /* Overwrite parameters */
    for (int i = 0; i<argc; i++) {
        printf(argv[i]);
        printf(" ");
    }
    printf("\n");
    if (argc == 2) {
        sscanf(argv[argc-1],"%u",&seed); 
    }
    if (argc == 3) {
        sscanf(argv[argc-1],"%d",&b2); 
        sscanf(argv[argc-2],"%d",&b1); 
    }
    if (argc == 4) {
        sscanf(argv[argc-1],"%d",&b2); 
        sscanf(argv[argc-2],"%d",&b1); 
        sscanf(argv[argc-3],"%d",&nstep);
    }
    if (argc == 5) {
        sscanf(argv[argc-1],"%u",&seed);
        sscanf(argv[argc-2],"%d",&b2);
        sscanf(argv[argc-3],"%d",&b1);
        sscanf(argv[argc-4],"%d",&nstep);
    }
	if (argc == 6) {
		sscanf(argv[argc - 1], "%d", &ms);
		sscanf(argv[argc - 2], "%u", &seed);
		sscanf(argv[argc - 3], "%d", &b2);
		sscanf(argv[argc - 4], "%d", &b1);
		sscanf(argv[argc - 5], "%d", &nstep);
	}
    printf("%i x %i, %i steps, seed = %u\n", b1, b2, nstep, seed);
	unsigned int networkSize = b1*b2;
    int warpSize = 32;
    if (networkSize/float(warpSize) != float(networkSize/warpSize)) {
        printf("please make networkSize multiples of %i\n", warpSize);
        return EXIT_FAILURE;
    }
    if (networkSize/10.0 != float(networkSize/10)) {
        printf("To have higher computation occupancy make a factor of 10 in networkSize\n");
    }
	int init_b2 = warpSize;
	int init_b1 = networkSize / init_b2;
    double eiRatio = 3.0f/4.0f;
    int b1E = b1*eiRatio;
    int b2E = b2*eiRatio;
    printf("prepare_cond <<< %i x %i >>> %i, %i, %f \n", b1E, b2E, b1, b2, eiRatio);
    int b1I = b1*(1-eiRatio);
    int b2I = b2*(1-eiRatio);
    unsigned int nE = networkSize*eiRatio;
    unsigned int nI = networkSize-nE;
    double t = 25.0f;
    double dt = t/float(nstep); // ms
    double flatRate = 10000.0f; // Hz
    //double flatRate = 0.0f; // Hz
    double ffsE = 2e-3;
    double s = 1e-2*ffsE/(networkSize);
    double ffsI = 2e-2;
    /* to be extended */
    bool presetInit = false;
    unsigned int ngTypeE = 2;
    unsigned int ngTypeI = 1;
    double riseTimeE[2] = {1.0f, 5.0f}; // ms
    double riseTimeI[1] = {1.0f};
    double decayTimeE[2] = {3.0f, 80.0f};
    double decayTimeI[1] = {5.0f};

    ConductanceShape condE(riseTimeE, decayTimeE, ngTypeE);
    ConductanceShape condI(riseTimeI, decayTimeI, ngTypeI);
    printf("designated input rate = %3.1fHz\n", flatRate);
	printf("dt = %f ms\n", dt);
    printf("nE = %i, nI = %i\n", nE, networkSize-nE);
    printf("t = %f x %i = %f\n", dt, nstep, t);
	int nInput = ceil(flatRate / 1000.0f * dt);
	#ifdef TEST_WITH_MANUAL_FFINPUT
		printf("for testing purpose, feedforward input is set to %i per %fms\n", nInput, dt);
		printf("for manual testing, please change the inputTime manually in source and recompile\n");
        cpu_version(networkSize, nInput, nstep, dt, nE, s, ffsE, ffsI);
    #endif
    struct hipDeviceProp_t properties;  
    double *v, *gE, *gI, *preMat; 
    int *eventRate, *d_eventRate;
    double *d_v1, *d_v2, *dv, *d_gE, *d_gI, *d_hE, *d_hI, *d_fE, *d_fI, *d_preMat, *d_inputRate;
    double *d_a0, *d_b0, *d_a1, *d_b1, *v_current, *v_old, *d_v_hlf;
    double *gactVec, *hactVec;
    double *leftTimeRate, *lastNegLogRand;
    double *spikeTrain, *d_spikeTrain, *tBack;
    bool *not_matched;
    bool *d_not_matched;

    while (init_b2 < 256 && init_b1 > 1) {
        init_b2 = init_b2*2;
        init_b1 = init_b1/2;
    }
    printf("init size %i, %i\n", init_b1, init_b2);

	/* check for double precision support */
	CUDA_CALL(hipGetDevice(&device));
	CUDA_CALL(hipGetDeviceProperties(&properties, device));
	if (!(properties.major >= 2 || (properties.major == 1 && properties.minor >= 3))) {
		printf(" double precision not supported\n");
		return EXIT_FAILURE;
	}

    unsigned int nbatch, batchEnd, batchStep;
    unsigned int ngType;
    if (ngTypeE > ngTypeI) {
        ngType = ngTypeE;
    } else {
        ngType = ngTypeI;
    }
    // v, gE, gI, spikeTrain
    unsigned int hostMemToDiskPerStep = ceil(networkSize * (sizeof(double) + ngTypeE*sizeof(double) + ngTypeI*sizeof(double) + sizeof(int) )/(1024*1024));
    //batchStep = floor(HALF_MEMORY_OCCUPANCY/hostMemToDiskPerStep);
    batchStep = 1;
    if (batchStep < 10) {
        printf("consider increase HALF_MEMORY_OCCUPANCY, batch step = %i\n", batchStep);
    } else {
        if (batchStep == 0) {
            printf("increase HALF_MEMORY_OCCUPANCY, memory to write on disk per step: %i Mb", hostMemToDiskPerStep);
            return EXIT_FAILURE;
        }
    }
    nbatch = nstep/batchStep; 
    batchEnd = nstep - batchStep*nbatch;
    int alt = 1;
    hipEvent_t iStart, iStop;
    hipEventCreate(&iStart);
    hipEventCreate(&iStop);
    CUDA_CALL(hipEventRecord(iStart, 0));
    /* Allocate space for results on host */
    //pinned memory
    CUDA_CALL(hipHostMalloc((void**)&v,           networkSize * sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&gE,          networkSize * ngTypeE * sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&gI,          networkSize * ngTypeI *sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&spikeTrain,  networkSize * sizeof(double) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&eventRate,   networkSize * sizeof(int) * batchStep * alt));
    CUDA_CALL(hipHostMalloc((void**)&not_matched, networkSize * sizeof(bool)));
    preMat = (double *)calloc(networkSize, sizeof(double));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&d_v1,           networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_v2,           networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&dv,             networkSize * sizeof(double)));
	CUDA_CALL(hipMalloc((void **)&d_v_hlf,        networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_gE,           networkSize * ngTypeE *sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_gI,           networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_hE,           networkSize * ngTypeE *sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_hI,           networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_fE,           networkSize * ngTypeE *sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_fI,           networkSize * ngTypeI * sizeof(double))); 
    CUDA_CALL(hipMalloc((void **)&d_a0,            networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_b0,            networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_a1,            networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_b1,            networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_inputRate,    networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_eventRate,    networkSize * sizeof(int)));
    CUDA_CALL(hipMalloc((void **)&d_spikeTrain,   networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&tBack,          networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&gactVec,        networkSize * ngType * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&hactVec,        networkSize * ngType * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_preMat,       networkSize * networkSize * sizeof(double)));
    /* Allocate space for rng on device */
    CUDA_CALL(hipMalloc((void **)&state,          networkSize * sizeof(hiprandStateMRG32k3a)));
    CUDA_CALL(hipMalloc((void **)&leftTimeRate,   networkSize * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&lastNegLogRand, networkSize * sizeof(double)));
    /* Allocate space for partial reduce results on device */
    /* Allocate variables that allow write-write conflict for global-OR operation on device*/
    CUDA_CALL(hipMalloc((void **)&d_not_matched, networkSize * sizeof(bool)));
    
    int maxTPB = properties.maxThreadsPerBlock/ms;
    int EmaxTPB, ImaxTPB;
    int mE, mI; 
    if (maxTPB < nE) {
        EmaxTPB = maxTPB;
        mE = (nE+EmaxTPB-1)/EmaxTPB;
        EmaxTPB = nE/mE;
    } else {
        mE = 1;
        EmaxTPB = nE;
    }
    while (EmaxTPB*mE != nE && EmaxTPB > EmaxTPB/2) {
        mE = mE + 1;
        EmaxTPB = nE/mE;
    }

    if (maxTPB < nI) {
        ImaxTPB = maxTPB;
        mI = (nI+ImaxTPB-1)/ImaxTPB;
        ImaxTPB = nI/mI;
    } else {
        mI = 1;
        ImaxTPB = nI;
    }
    while (ImaxTPB*mI != nI && ImaxTPB > ImaxTPB/2) {
        mI = mI + 1;
        ImaxTPB = nI/mI;
    }

    dim3 rgE_b1, rgI_b1;
    int EnTPB = networkSize/(networkSize/EmaxTPB);
    int InTPB = networkSize/(networkSize/ImaxTPB);
    if (EnTPB > maxTPB) {
        EnTPB = maxTPB;
    }
    if (InTPB > maxTPB) {
        InTPB = maxTPB;
    }
    dim3 rgE_b2(EnTPB,1);
    dim3 rgI_b2(InTPB,1);
    printf("mE = %i, mI = %i\n", mE, mI);
    //dim3 rgE_b2(EmaxTPB,1);
    //dim3 rgI_b2(ImaxTPB,1);
    int msE = 1; // multiple shared actVec load per thread
    int msI = 1;
    int s_actVec_lE; // length of shared actVec
    int s_actVec_lI;
    unsigned int rgE_shared;
    unsigned int rgI_shared;

    s_actVec_lE = EmaxTPB;
    rgE_shared = 2*ngTypeE*s_actVec_lE*sizeof(double);
    if (rgE_shared > properties.sharedMemPerBlock) {
        printf("E: The size of the requested shared memory %iKb by recal_G is not available\n", rgE_shared/1024);
        return EXIT_FAILURE;
    } else {
        if (moreSharedMemThanBlocks) {
            while (rgE_shared*2  < properties.sharedMemPerBlock && mE/float(msE*2) == float(mE/(msE*2))) {
                msE = msE * 2;
                rgE_shared = rgE_shared * 2;
            }
        }
    }
    s_actVec_lE = msE*s_actVec_lE; // number of actVec each chunk dump into shared mem, msE multiples of maxTPB
    rgE_b1.x = nE/s_actVec_lE; // chunks of maxTPB neurons
    rgE_b1.y = networkSize/EnTPB; // total number of presynaptic neurons divided by the the shared actVec
    printf("E: recal_G<<<(%i,%i,%i)x(%i,%i,%i), %iKb>>>, msE = %i\n", rgE_b1.x, rgE_b1.y, rgE_b1.z, rgE_b2.x, rgE_b2.y, rgE_b2.z, rgE_shared/1024, msE);

    s_actVec_lI = ImaxTPB;
    rgI_shared = 2*ngTypeI*s_actVec_lI*sizeof(double);
    if (rgI_shared > properties.sharedMemPerBlock) {
        printf("I: The size of the requested shared memory %iKb by recal_G is not available\n", rgI_shared/1024);
        return EXIT_FAILURE;
    } else {
        if (moreSharedMemThanBlocks) {
            while (rgI_shared*2  < properties.sharedMemPerBlock && mI/float(msI*2) == float(mI/(msI*2))) {
                msI = msI * 2;
                rgI_shared = rgI_shared * 2;
            }
        }
    }
    s_actVec_lI = msI*s_actVec_lI;
    rgI_b1.x = nI/s_actVec_lI;
    rgI_b1.y = networkSize/InTPB;
    printf("I: recal_G<<<(%i,%i,%i)x(%i,%i,%i), %iKb>>>, msI = %i\n", rgI_b1.x, rgI_b1.y, rgI_b1.z, rgI_b2.x, rgI_b2.y, rgI_b2.z, rgI_shared/1024, msI);

    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    int rE_b2, rI_b2;
    double *gE_b1y, *gI_b1y, *hE_b1y, *hI_b1y;
    if (rgE_b1.x >= 32) {
        int e = 5;
        while (rgE_b1.x > 1<<e) e++;
        rE_b2 = 1<<e;
        printf("blockdims for reduction of %i per thread : %i x %i \n", rgE_b1.x, networkSize, rE_b2);
        CUDA_CALL(hipMalloc((void **)&gE_b1y,  networkSize * rE_b2 * ngTypeE * sizeof(double)));
        CUDA_CALL(hipMalloc((void **)&hE_b1y,  networkSize * rE_b2 * ngTypeE * sizeof(double)));
    }
    if (rgI_b1.x >= 32) {
        int e = 5;
        while (rgI_b1.x > 1<<e) e++;
        rI_b2 = 1<<e;
        printf("blockdims for reduction of %i per thread : %i x %i \n", rgI_b1.x, networkSize, rE_b2);
        CUDA_CALL(hipMalloc((void **)&gI_b1y,  networkSize * rI_b2 * ngTypeI * sizeof(double)));
        CUDA_CALL(hipMalloc((void **)&hI_b1y,  networkSize * rI_b2 * ngTypeI * sizeof(double)));
    }




    /* Create CUDA events */
    hipEvent_t start, stop, spikeCorrected, gReadyE, gReadyI, vReady, initialSpikesObtained, spikeRateReady, eventRateReady;
    hipEvent_t kStart, kStop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&gReadyE);
    hipEventCreate(&gReadyI);
    hipEventCreate(&vReady);
    hipEventCreate(&spikeCorrected);
    hipEventCreate(&initialSpikesObtained);
    hipEventCreate(&kStart);
    hipEventCreate(&kStop);
    hipEventCreate(&spikeRateReady);
    hipEventCreate(&eventRateReady);
    /* Initialize device arrays */
    // CUDA streams for init
    hipStream_t i1, i2, i3, i4, i5, i6, i7;
    CUDA_CALL(hipStreamCreate(&i1));
    CUDA_CALL(hipStreamCreate(&i2));
    CUDA_CALL(hipStreamCreate(&i3));
    CUDA_CALL(hipStreamCreate(&i4));
    CUDA_CALL(hipStreamCreate(&i5));
    CUDA_CALL(hipStreamCreate(&i6));
    CUDA_CALL(hipStreamCreate(&i7));
    if (presetInit) {
    } else {
        for (unsigned int i=0; i<networkSize; i++) {
            v[i] = 0.0f;
            gE[i] = 0.0f;
            gI[i] = 0.0f;
            spikeTrain[i] = dt;
        }
        // init rand generation for poisson
        logRand_init<<<init_b1,init_b2,0,i1>>>(lastNegLogRand, state, seed);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i2>>>(d_inputRate, flatRate/1000.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i3>>>(d_v1, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i4>>>(d_v2, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i5>>>(leftTimeRate, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1,init_b2,0,i6>>>(tBack, -1.0f); 
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i7>>>(d_fE, ffsE);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i1>>>(d_fI, ffsI);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i2>>>(d_gE, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i3>>>(d_gI, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeE,init_b2,0,i4>>>(d_hE, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngTypeI,init_b2,0,i5>>>(d_hI, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngType,init_b2,0,i6>>>(gactVec, 0.0f);
        CUDA_CHECK();
        init<double><<<init_b1*ngType,init_b2,0,i7>>>(hactVec, 0.0f);
        CUDA_CHECK();
        //CUDA_CALL(hipEventRecord(kStart, 0));
        printf("storage size of preMat %.1fMb\n", float(networkSize*networkSize*sizeof(double))/1024.0/1024.0);
        init<<<init_b1*init_b1*init_b2,init_b2,0,i1>>>(d_preMat, s);
        CUDA_CHECK();
        //CUDA_CALL(hipEventRecord(kStop, 0));
        //CUDA_CALL(hipEventSynchronize(kStop));
        //CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
        //printf("logRand_init<<<%ix%i>>> cost %.1fms\n", init_b1*init_b1*init_b2, init_b2, time);
    }
    CUDA_CALL(hipStreamDestroy(i1));
    CUDA_CALL(hipStreamDestroy(i2));
    CUDA_CALL(hipStreamDestroy(i3));
    CUDA_CALL(hipStreamDestroy(i4));
    CUDA_CALL(hipStreamDestroy(i5));
    CUDA_CALL(hipStreamDestroy(i6));
    CUDA_CALL(hipStreamDestroy(i7));
    CUDA_CALL(hipEventRecord(iStop, 0));
    CUDA_CALL(hipEventSynchronize(iStop));
    CUDA_CALL(hipEventElapsedTime(&time, iStart, iStop));
    printf("initialization cost %fms\n", time);

    /* Create CUDA streams */
    hipStream_t s1, s2, s3, s4, s5;
    CUDA_CALL(hipStreamCreate(&s1));
    CUDA_CALL(hipStreamCreate(&s2));
    CUDA_CALL(hipStreamCreate(&s3));
    CUDA_CALL(hipStreamCreate(&s4));
    CUDA_CALL(hipStreamCreate(&s5));
    unsigned int shared_mem = 0;
    v_file.open("v_ictorious.bin", std::ios::out|std::ios::binary);
    spike_file.open("s_uspicious.bin", std::ios::out|std::ios::binary);
    gE_file.open("gE_nerous.bin", std::ios::out|std::ios::binary);
    gI_file.open("gI_berish.bin", std::ios::out|std::ios::binary);
    CUDA_CALL(hipEventRecord(start, 0));
    double events = 0.0f;
    int spikes = 0;
    unsigned int ibatch = 0;
    unsigned int bStep = 0;
    unsigned int batchOffset = 0;
    unsigned int copySize = batchStep;
    unsigned int n = networkSize*copySize;
    CUDA_CALL(hipEventRecord(gReadyE, 0));
    CUDA_CALL(hipEventRecord(gReadyI, 0));
    CUDA_CALL(hipEventRecord(vReady, 0));
    
    //for (int ibatch=0; i<nbatch; ibatch++) {
    //    if(ibatch == nbatch-1) {
    //        copySize = batchEnd;
    //    }
        bool it = true;
        double timeV = 0.0f;
        double timeG = 0.0f;
        double timeS = 0.0f;
        double timeP = 0.0f;
        for (int i=0; i<nstep; i++) {
            unsigned int offset; 
            //offset = networkSize*(batchOffset + i);
            offset = 0;
            /* Write voltage to file */
            CUDA_CALL(hipEventSynchronize(vReady));
            v_file.write((char*)v, networkSize * sizeof(double));
            /* Write conductance of last step to disk */
            CUDA_CALL(hipEventSynchronize(gReadyE));
            gE_file.write((char*)&(gE[n*ngTypeE*batchOffset]),     n*ngTypeE*sizeof(double));
            CUDA_CALL(hipEventSynchronize(gReadyI));
            gI_file.write((char*)&(gI[n*ngTypeI*batchOffset]),     n*ngTypeI*sizeof(double));
            /* Compute voltage (acquire initial spikes) */
            if (i%2 == 0) {
                v_current = d_v2;
                v_old = d_v1;
            } else {
                v_current = d_v1;
                v_old = d_v2;
            }
            #ifdef KERNEL_PERFORMANCE
                CUDA_CALL(hipEventRecord(kStart, 0));
            #endif
            compute_dV <<<b1, b2, shared_mem, s1>>> (v_old, dv, d_gE, d_gI, d_hE, d_hI, d_a0, d_b0, d_a1, d_b1, d_preMat, d_inputRate, d_eventRate, d_spikeTrain, tBack, gactVec, hactVec, d_fE, d_fI, leftTimeRate, lastNegLogRand, d_v_hlf, state, ngTypeE, ngTypeI, ngType, condE, condI, dt, networkSize, nE, seed, nInput, it);
            CUDA_CHECK();
            CUDA_CALL(hipEventRecord(initialSpikesObtained, s1));
            CUDA_CALL(hipEventRecord(kStop, 0));
            CUDA_CALL(hipEventSynchronize(kStop));
            CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
            timeV += time;
            if (printStep) {
                printf("A single step of compute_V cost %fms\n", time);
            }
            // Copy feedforward eventRate to file
            CUDA_CALL(hipStreamWaitEvent(s1, initialSpikesObtained, 0));
            CUDA_CALL(hipMemcpyAsync(eventRate, d_eventRate, networkSize * sizeof(int), hipMemcpyDeviceToHost, s1));
            CUDA_CALL(hipEventRecord(eventRateReady, s1));
            /* Spike correction */
            bool no_match = true;
            unsigned int imatch = 0;
            CUDA_CALL(hipEventRecord(kStart, 0));
            while (no_match && imatch < networkSize) {
                printf("correct_spike <<< %i x %i >>>\n", b1, b2);
                correct_spike <<<b1, b2, 0, s1>>> (d_not_matched, d_spikeTrain, d_v_hlf, v_old, dv, d_a0, d_b0, d_a1, d_b1, v_current, d_preMat, tBack, ngTypeE, ngTypeI, condE, condI, dt, nE, networkSize);
                CUDA_CHECK();
                printf("correct_spike <<< %i x %i >>> finished\n", b1, b2);
                CUDA_CALL(hipMemcpyAsync(&not_matched, d_not_matched, networkSize * sizeof(bool), hipMemcpyDeviceToHost,s1));
                CUDA_CALL(hipEventRecord(spikeCorrected, s1));
                no_match = false;
                CUDA_CALL(hipEventSynchronize(spikeCorrected)); 
                printf("synchronized\n");
				//CUDA_CALL(hipDeviceSynchronize());
                for (int iw = 0; iw < warpSize; iw ++) {
					printf("iw = %i\n", iw);
                    bool iw_no_match = false;
                    for (unsigned int j=0; j<networkSize/warpSize; j++) {
                        unsigned int wid = j*warpSize + iw;
						printf("%i = %i*%i + %i\n", wid, j, warpSize, iw);
                        if (wid > networkSize) {
                            break;
                        }
                        if (not_matched[j*warpSize + iw]) {
                            iw_no_match = true;
                            break;
                        }
                    }
                    if (iw_no_match) {
                        no_match = true;
                        break;
                    }
                }
                imatch++;
            }
            printf("%i matching iterations\n", imatch);
            #ifdef KERNEL_PERFORMANCE
                CUDA_CALL(hipEventRecord(kStop, 0));
                CUDA_CALL(hipEventSynchronize(kStop));
                CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
                if (printStep) {
                    printf("spike correction of step %u cost %fms\n", i, time);
                }
                timeS += time;
            #endif
            // Finalize current step
            #ifdef KERNEL_PERFORMANCE
                CUDA_CALL(hipEventRecord(kStart, 0));
            #endif
            // Copy voltage and spikeTrain of the current step to host (first E then I)
            CUDA_CALL(hipMemcpyAsync(&spikeTrain, d_spikeTrain, networkSize * sizeof(double), hipMemcpyDeviceToHost, s1));
            CUDA_CALL(hipEventRecord(spikeRateReady, s1));
            CUDA_CALL(hipMemcpyAsync(&v, v_current, networkSize * sizeof(double), hipMemcpyDeviceToHost, s1));
            CUDA_CALL(hipEventRecord(vReady, s1));
            // Get presynaptic conductance ready
            printf("prepare_cond <<< %i x %i >>> %i, %i, %f \n", b1E, b2E, b1, b2, eiRatio);
            prepare_cond <<<b1E, b2E, 0, s2>>> (tBack, d_spikeTrain, gactVec, hactVec, condE, dt, ngTypeE, 0, networkSize);
            CUDA_CHECK();
            CUDA_CALL(hipEventRecord(gReadyE, s2));
            prepare_cond <<<b1I, b2I, 0, s3>>> (tBack, d_spikeTrain, gactVec, hactVec, condI, dt, ngTypeE, nE, networkSize);
            CUDA_CHECK();
            CUDA_CALL(hipEventRecord(gReadyI, s3));
            #ifdef KERNEL_PERFORMANCE
                CUDA_CALL(hipEventRecord(kStop, 0));
                CUDA_CALL(hipEventSynchronize(kStop));
                CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
                if (printStep) {
                    printf("establish the current step cost %fms\n", time);
                }
                timeP += time;
            #endif

            /* Recalibrate conductance to postsynaptic neurons, for the next step*/
            // recal E
            #ifdef KERNEL_PERFORMANCE
                CUDA_CALL(hipEventRecord(kStart, 0));
            #endif
            CUDA_CALL(hipStreamWaitEvent(s2, gReadyI, 0));
            recal_G <<<rgE_b1,rgE_b2,rgE_shared,s2>>> (d_gE, d_hE, d_preMat,
                                                     gactVec, hactVec,
                                                     gE_b1y, hE_b1y,
                                                     networkSize, 0, ngTypeE, s_actVec_lE, msE);
            CUDA_CHECK();
            // recal I
            CUDA_CALL(hipStreamWaitEvent(s3, gReadyE, 0));
            recal_G<<<rgI_b1,rgI_b2,rgI_shared,s3>>>(d_gI, d_hI, d_preMat,
                                                     gactVec, hactVec,
                                                     gI_b1y, hI_b1y,
                                                     networkSize, nE, ngTypeI, s_actVec_lI, msI);
            CUDA_CHECK();
            if (rgE_b1.x >= 32) {
                //  reduce sum
                reduce_G<<<networkSize, rE_b2, sizeof(double)*2*rE_b2, s2>>>(d_gE, d_hE, gE_b1y, hE_b1y, ngTypeE, rgE_b1.x);
                CUDA_CHECK();
            }
            // copy exc conductance to host
            CUDA_CALL(hipMemcpyAsync(&gE, d_gE, networkSize * ngTypeE * sizeof(double), hipMemcpyDeviceToHost, s2));
            if (rgI_b1.x >= 32) {
                reduce_G<<<networkSize, rI_b2, sizeof(double)*2*rI_b2, s3>>>(d_gI, d_hI, gI_b1y, hI_b1y, ngTypeI, rgI_b1.x);
                CUDA_CHECK();
            }
            // copy inh conductance to host
            CUDA_CALL(hipMemcpyAsync(&gI, d_gI, networkSize * ngTypeI * sizeof(double), hipMemcpyDeviceToHost, s3));
            #ifdef KERNEL_PERFORMANCE
                CUDA_CALL(hipEventRecord(kStop, 0));
                CUDA_CALL(hipEventSynchronize(kStop));
                CUDA_CALL(hipEventElapsedTime(&time, kStart, kStop));
                if (printStep) {
                    printf("A single step of recal_G cost %fms\n", time);
                }
                timeG += time;
            #endif
            CUDA_CALL(hipEventRecord(gReadyE, s2));
            CUDA_CALL(hipEventRecord(gReadyI, s3));
            //printf("\r total: %3.1f, batch: %3.1f", 100.0f*float(ibatch+1)/nbatch, float(i)/copySize);
            CUDA_CALL(hipEventSynchronize(eventRateReady));
            CUDA_CALL(hipEventSynchronize(spikeRateReady));
            /* Copy spikeTrain to host */
            spike_file.write((char*)&(spikeTrain[n*batchOffset]),  n*sizeof(double));
            printf("\r stepping: %3.1f%%", 100.0f*float(i+1)/nstep);
            fflush(stdout);
            it = false;
            double _events = 0.0f;
            int _spikes = 0;
            for (int j=0; j<networkSize; j++) {
                _events += eventRate[j];
                if (spikeTrain[j] < dt) {
                    _spikes++;
                }
            }
            events += _events;
            spikes += _spikes;
            if (printStep) {
                printf("instant input rate = %fkHz, dt = %f, networkSize = %i\n", _events/(dt*networkSize), dt, networkSize);
                printf("instant firing rate = %fHz\n", _spikes/(dt*networkSize)*1000.0);
            }
            if (batchOffset == 0) {
                batchOffset = 0;
                //batchOffset = batchStep;
            } else {
                batchOffset = 0;
            }
        }
        /* WHen hit HALF_MEMORY_OCCUPANCY, write half of the array to disk, the other half left to receive from device */
        // Alternating
        // switch batchOffset
    //}

    v_file.write((char*)v, networkSize * sizeof(double));
    spike_file.write((char*)spikeTrain, networkSize * sizeof(int));
    gE_file.write((char*)gE, networkSize * ngTypeE * sizeof(double));
    gI_file.write((char*)gI, networkSize * ngTypeI * sizeof(double));
    printf("\n");

    printf("flatRate = %fkHz, realized mean input rate = %fkHz\n", flatRate/1000.0, float(events)/(dt*nstep*networkSize));
    printf("mean firing rate = %fHz\n", float(spikes)/(dt*nstep*networkSize)*1000.0);

    CUDA_CALL(hipEventRecord(stop, 0));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipEventElapsedTime(&time, start, stop));
    printf("CUDA takes %fms, runtime/realtime ratio ms %fms\n", time, time/(dt*nstep));
    printf("compute_V takes %fms, ratio ms %fms\n", timeV, timeV/(dt*nstep));
    printf("recal_G takes %fms, ratio ms %fms\n", timeG, timeG/(dt*nstep));
    printf("correct_spike takes %fms, ratio ms %fms\n", timeS, timeS/(dt*nstep));
    printf("prepare_cond and establish_v takes %fms, ratio ms %fms\n", timeP, timeP/(dt*nstep));

    /* Cleanup */
    printf("Cleaning up:\n");
    CUDA_CALL(hipStreamDestroy(s1));
    CUDA_CALL(hipStreamDestroy(s2));
    CUDA_CALL(hipStreamDestroy(s3));
    printf("    CUDA streams destroyed\n");
    if (v_file.is_open()) v_file.close();
    if (spike_file.is_open()) spike_file.close();
    if (gE_file.is_open()) gE_file.close();
    if (gI_file.is_open()) gI_file.close();
    printf("    Output files closed\n");
    
    CUDA_CALL(hipFree(state));
    CUDA_CALL(hipFree(d_v1));
    CUDA_CALL(hipFree(d_v2));
    CUDA_CALL(hipFree(d_gE));
    CUDA_CALL(hipFree(d_gI));
    CUDA_CALL(hipFree(d_hE));
    CUDA_CALL(hipFree(d_hI));
    CUDA_CALL(hipFree(d_fE));
    CUDA_CALL(hipFree(d_fI));
    CUDA_CALL(hipFree(dv));
    CUDA_CALL(hipFree(gactVec));
    CUDA_CALL(hipFree(hactVec));
    CUDA_CALL(hipFree(d_preMat));
    CUDA_CALL(hipFree(d_a0));
    CUDA_CALL(hipFree(d_b0));
    CUDA_CALL(hipFree(d_a1));
    CUDA_CALL(hipFree(d_b1));
    if (rgE_b1.x >= 32) {
        CUDA_CALL(hipFree(gE_b1y));
        CUDA_CALL(hipFree(hE_b1y));
    }
    if (rgI_b1.x >= 32) {
        CUDA_CALL(hipFree(gI_b1y));
        CUDA_CALL(hipFree(hI_b1y));
    }
    CUDA_CALL(hipFree(leftTimeRate));
    CUDA_CALL(hipFree(lastNegLogRand));
    CUDA_CALL(hipFree(d_inputRate));
    CUDA_CALL(hipFree(d_eventRate));
    CUDA_CALL(hipFree(d_spikeTrain));
	CUDA_CALL(hipFree(d_not_matched));
    CUDA_CALL(hipFree(tBack));
    printf("    Device memory freed\n");
    CUDA_CALL(hipHostFree(v));
    CUDA_CALL(hipHostFree(gE));
    CUDA_CALL(hipHostFree(gI));
    CUDA_CALL(hipHostFree(eventRate));
    CUDA_CALL(hipHostFree(spikeTrain));
	CUDA_CALL(hipHostFree(not_matched));
    free(preMat);
    printf("    Host memory freed\n");
    return EXIT_SUCCESS;
}
