#include "hip/hip_runtime.h"
#include "connect.h"

extern __device__ __constant__ pFeature pref[];

// TODO: randomize neuronal attributes by using distribution, strength x number of con. should be controlled
__global__ 
void initialize(hiprandStateMRG32k3a* __restrict__ state,
                Size*  __restrict__ preType, // 
                Float* __restrict__ rden,
                Float* __restrict__ raxn,
                Float* __restrict__ dden,
                Float* __restrict__ daxn,
                Float* __restrict__ preF_type,
                Float* __restrict__ preS_type,
                Size*  __restrict__ preN_type,
                Float* __restrict__ LGN_V1_sSum,
                Float* __restrict__ ExcRatio,
                Float extExcRatio, Float min_FB_ratio, initialize_package init_pack, unsigned long long seed, Size networkSize, Size nType, Size nArchtype, Size nFeature, bool CmoreN, Float p_n_LGNeff) 
{
    //__shared__ reduced[warpSize];
    Size id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandStateMRG32k3a localState = state[id];
    hiprand_init(seed, id, 0, &localState);
	state[id] = localState;
   	Size type;
	// determine the arch neuronal type and its properties
	#pragma unroll
    for (Size i=0; i<nType; i++) {
        if (threadIdx.x < init_pack.typeAccCount[i]) {
            rden[id] = init_pack.rden[i];
            raxn[id] = init_pack.raxn[i];
            dden[id] = init_pack.dden[i];
            daxn[id] = init_pack.daxn[i];
            preType[id] = i;
            type = i;
            break;
        }
	}

	Float LGN_sSum = LGN_V1_sSum[id];
	Float presetConstExc = p_n_LGNeff + init_pack.sumType[type]*(1+extExcRatio);
    Float ratio;
	if (init_pack.sumType[type] == 0) {
		ratio = 1.0;
	} else {
		ratio = (presetConstExc - LGN_sSum)/(init_pack.sumType[type]*(1+extExcRatio));
	}
    if (ratio < min_FB_ratio) ratio = min_FB_ratio;
	ExcRatio[id] = ratio;
	for (PosInt i=0; i<nType; i++) {
		PosInt tid = i*networkSize+id;
		PosInt ttid = i*nType + type;
        if (CmoreN) {
    	    preS_type[tid] = init_pack.sTypeMat[ttid];
            if (i < init_pack.iArchType[0]) {
                preN_type[tid] = static_cast<Size>(rounding(ratio*init_pack.nTypeMat[ttid]));
            } else {
    	        //preN_type[tid] = static_cast<Size>(rounding((ratio + LGN_sSum/init_pack.sumType[i*nType+0])*init_pack.nTypeMat[ttid]));
    	        preN_type[tid] = init_pack.nTypeMat[ttid];
            }
        } else {
    	    preN_type[tid] = init_pack.nTypeMat[ttid];
            if (i < init_pack.iArchType[0]) {
                preS_type[tid] = ratio*init_pack.sTypeMat[ttid];
            } else {
    	        //preS_type[tid] = (ratio+ LGN_sSum/init_pack.sumType[i*nType+0])*init_pack.sTypeMat[ttid];
    	        preS_type[tid] = init_pack.sTypeMat[ttid];
            }
        }
		//printf("%u-%u-%u: LGN_sSum = %f,  %u*%f = %f\n", blockIdx.x, threadIdx.x, i, LGN_sSum, preN_type[tid], preS_type[tid], preN_type[tid] * preS_type[tid]);
        for (PosInt j=0; j<nFeature; j++) {
            PosInt fid = (j*nType + i)*networkSize + id;
            preF_type[fid] = init_pack.typeFeatureMat[j*nType*nType+ttid];
        }
	}
}

__device__ 
__forceinline__
Float tri_cos(Float a, Float b, Float c) {
    return (a*a + b*b - c*c)/(2*a*b);
}

//__device__ Float seg(Float cosine, Float radius) {
//    return arccos(cosine)/(radius*radius);
//}

//__device__ Float chord(Float radius, Float cosine) {
//    Float r2 = radius*radius;
//    Float cos2 = cosine*cosine;
//    return square_root(r2- cos2*r2) * radius*cosine;
//}

__device__ 
__forceinline__
Float area(Float raxn, Float rden, Float d) {
    Float cos_theta_axn = tri_cos(raxn, d, rden);
	Float cos_theta_den = tri_cos(rden, d, raxn);

    Float theta_axn = arccos(cos_theta_axn);
    Float theta_den = arccos(cos_theta_den);

    Float sin_theta_axn = sine(theta_axn);
    Float sin_theta_den = sine(theta_den);

    return (theta_axn-sin_theta_axn*cos_theta_axn)*raxn*raxn 
         + (theta_den-sin_theta_den*cos_theta_den)*rden*rden;
}

// co-occupied area of the presynaptic axons / dendritic area
__device__ 
__forceinline__
Float connect(Float distance, Float raxn, Float rden, bool gaussian_profile) {
	Float weight;
	if (gaussian_profile) {
		Float spread = raxn*raxn + rden*rden;
        if (distance < 3*square_root(spread/2)) {
		    weight = exponential(-distance*distance/spread)/(M_PI*spread);
        } else {
            weight = 0.0;
        }
	} else {
    	weight = 0.0;
    	if (raxn + rden > distance && distance > abs(raxn - rden)) {
    	    weight = area(raxn, rden, distance)/(M_PI*rden*rden); // conn. prob. is defined by the presynaptic point of view
    	} else if (distance <= abs(raxn - rden)) {
    	    weight = 1.0;
    	}
	}
    return weight;
}

__global__ 
void cal_blockPos(double* __restrict__ pos,
                  Float* __restrict__ block_x,
                  Float* __restrict__ block_y,
                  Size networkSize) 
{
    __shared__ double reduced[warpSize];
    Size id = (2*blockDim.x)*blockIdx.x + threadIdx.x;
    double x = pos[id];
    double y = pos[id + blockDim.x];
    block_reduce<double>(reduced, x);
    if (threadIdx.x == 0) {
        block_x[blockIdx.x] = static_cast<Float>(reduced[0]/blockDim.x);
    }
    block_reduce<double>(reduced, y);
    if (threadIdx.x == 0) {
        block_y[blockIdx.x] = static_cast<Float>(reduced[0]/blockDim.x);
    }
}

__global__ 
void get_neighbor_blockId(Float* __restrict__ block_x,
                          Float* __restrict__ block_y,
                          PosInt* __restrict__ neighborBlockId,
                          Size* __restrict__ nNeighborBlock,
						  Size nblock, Float max_radius, Size maxNeighborBlock) 
{
    __shared__ PosInt id[warpSize];
    __shared__ Float min[warpSize];
    __shared__ Int bid[blockSize];
	__shared__ Float distance[blockSize];

	extern __shared__ Float final_distance[];
	PosInt* final_bid = (PosInt*) (final_distance + maxNeighborBlock);

    Float bx = block_x[blockIdx.x]; // center of the target block
    Float by = block_y[blockIdx.x];
    Size tid = threadIdx.y*blockDim.x + threadIdx.x;

    Size nPatch = (nblock + blockDim.x-1)/blockDim.x - 1;
    Size remain = nblock%blockDim.x;
	if (remain == 0) {
		remain = blockDim.x;
	}

    Size offset = 0;
    if (tid == 0) {
        id[0] = 0;
    }
    bid[tid] = -1;
    __syncthreads();
    for (Size iPatch = 0; iPatch < nPatch+1; iPatch++) {
        if (iPatch < nPatch || tid < remain) {
            PosInt blockId = offset + threadIdx.x;
            Float x = block_x[blockId] - bx;
            Float y = block_y[blockId] - by;
            Float dis = square_root(x*x + y*y);
            if (dis < max_radius) {
                distance[tid] = dis;
                bid[tid] = blockId;
            }
        }
        __syncthreads();
        if (tid == 0) { // rearrange
            PosInt current_id = id[0];
            for (PosInt i=0; i<blockDim.x; i++) {
                if (bid[i] != -1) {
                    final_distance[current_id] = distance[i]; 
                    final_bid[current_id] = bid[i]; 
                    bid[i] = -1; 
                    current_id++;
                    if (current_id > maxNeighborBlock) {
                        printf("actual nNeighbor = %d > %d (preserved)\n", current_id, maxNeighborBlock);
                        assert(current_id <= maxNeighborBlock);
                    }
                }
            }
            id[0] = current_id;
        }
        __syncthreads();
        if (iPatch < nPatch) {
            offset += blockDim.x;
        }
    }
    Size nb = id[0];
    if (tid == 0) {
        nNeighborBlock[blockIdx.x] = nb;
	    //printf("%u: %u blocks in total\n", blockIdx.x, nb);
    }
    Float dis;
    PosInt local_bid;
    if (tid < nb) {
        dis = final_distance[tid];
        /*DEBUG
            if (blockIdx.x == 2) {
                printf("preSort#%u:%u, %e\n", tid, final_bid[tid], dis);
            }
        */
    }
    // sorting
    for (Size i=0; i<nb;  i++) {
        find_min(min, dis, id, nb);
        PosInt min_id = id[0];
        if (tid == min_id) {
            dis = max_radius + 1; // to be excluded for next min
        }
        if (tid == i) { // get read for global store
            local_bid = final_bid[min_id];
        }
		/* DEBUG
            if (blockIdx.x == 2 && tid < nb) {
                printf("%u#%u:%u, %e\n",i, tid, final_bid[tid], dis);
            }
            __syncthreads();
        */
    }
    //TODO: if nb > blockSize
    if (tid < nb) {
        neighborBlockId[maxNeighborBlock*blockIdx.x + tid] = local_bid;
    }
	/* DEBUG
        if (blockIdx.x == 2) {
	        for (PosInt j = 0; j<nb; j++) {
	        	if (tid == j) {
	        		if (j == 0) {
	        			printf("block %u, %u neighbors in total\n", blockIdx.x, nb);
	        		}
	        		printf("#%u, %u: %f -> %f,", j, local_bid, final_distance[j], dis);
	        		if (j == nb-1) {
	        			printf("\n");
	        		}
	        	}
	        	__syncwarp();
	        }
        }
	*/
}

__launch_bounds__(1024,1)
__global__ 
void generate_connections(double* __restrict__ pos,
                          Float* __restrict__ preF_type,
                          Float* __restrict__ preS_type,
                          Size* __restrict__ preN_type,
                          PosInt* __restrict__ neighborBlockId,
                          Size* __restrict__ nNeighborBlock,
                          Float* __restrict__ rden,
                          Float* __restrict__ raxn,
                          Float* __restrict__ conMat, //within block connections
                          Float* __restrict__ delayMat,
                          Float* __restrict__ conVec, //for neighbor block connections
                          Float* __restrict__ delayVec, //for neighbor block connections
                          Size* __restrict__ vecID,
                          Size* __restrict__ nVec,
                          Size* __restrict__ preTypeConnected,
                          Size* __restrict__ preTypeAvail,
                          Float* __restrict__ preTypeStrSum,
                          Size* __restrict__ preType,
                          Float* __restrict__ feature,
                          Float* __restrict__ dden,
                          Float* __restrict__ daxn,
                          Size* __restrict__ typeAcc0,
                          hiprandStateMRG32k3a* __restrict__ state,
                          PosInt block_offset, Size networkSize, Size maxDistantNeighbor, Size nearNeighborBlock, Size maxNeighborBlock, Size nType, Size nFeature, bool gaussian_profile, bool strictStrength, Float tol) 
{
    // TODO: load with warps but more, e.g., raxn, daxn, preType
    __shared__ double x1[blockSize];
    __shared__ double y1[blockSize];
    //__shared__ Float ra[blockDim.x];
    Size blockId = blockIdx.x + block_offset;
    Size nn = nNeighborBlock[blockId];
    Size offset = blockId*blockDim.x;
    double x0 = pos[offset*2 + threadIdx.x];
    double y0 = pos[offset*2 + threadIdx.x + blockDim.x];
    Size id = offset + threadIdx.x;
    // number of potential presynaptic connections outsied nearNeighbors, to be stored in vector.
    Size nb = 0; 
    if (nn > nearNeighborBlock) {
        nb = nn - nearNeighborBlock;
        nn = nearNeighborBlock;// nearNeighbors 
    } 
    Float* tempNeighbor;
    if (nb > 0) {
        nb *= blockDim.x;
        tempNeighbor = new Float[nb];
    }
    Float rd = rden[id];
    Float dd = dden[id];

    Size* sumType = new Size[nType]; // avail
    Float* sumP = new Float[nType];
    Float* pF = new Float[nFeature*nType];
    Float* fV = new Float[nFeature];
    #pragma unroll
    for (PosInt i=0; i<nType; i++) {
        sumType[i] = 0;
        sumP[i] = 0.0;
	}
    for (PosInt i=0; i<nFeature; i++) {
        fV[i] = feature[i*networkSize + id];
        for (PosInt j=0; j<nType; j++) {
            pF[i*nType + j] = preF_type[(i*nType+j)*networkSize + id];
        }
    }
    //============= collect p of all ==========
    // withhin block and nearNeighbor
    for (PosInt in=0; in<nn; in++) {
        PosInt bid = neighborBlockId[maxNeighborBlock*blockId + in] * blockDim.x; // # neurons in all past blocks 
        x1[threadIdx.x] = pos[bid*2 + threadIdx.x];
        y1[threadIdx.x] = pos[bid*2 + blockDim.x + threadIdx.x];
        __syncthreads();
        #pragma unroll
        for (Size i=0; i<blockDim.x; i++) {
            // blockwise load from gmem the (potential) presynaptic neurons' properties
            Size ipre = bid + i; // pre-id in network
            Float ra = raxn[ipre];
            Float x = static_cast<Float>(x1[i] - x0);
            Float y = static_cast<Float>(y1[i] - y0);
            //type vector, indexed across the network
            Float distance = square_root(x*x + y*y);
	    	// weight from area
            Float p = connect(distance, ra, rd, gaussian_profile);
            PosIntL mid = (static_cast<PosIntL>(blockIdx.x*nearNeighborBlock + in)*blockDim.x + i)*blockDim.x + threadIdx.x; // defined outside, so delayMat has access to it
                Size ip = preType[ipre];
            if (p > 0 && id != ipre) { // not self-connected
                //Size ip = preType[ipre];
                // id in the conMat [nblock,nearNeighborBlock,blockDim.x,blockDim.x] loop in the second axis, (last dim is the post-id: threadIdx.x, pre-id in the chunk: i)
                //DEBUG
                BigSize matSize = static_cast<BigSize>(blockDim.x*blockDim.x)*nearNeighborBlock*gridDim.x;
                if (mid >= matSize) {
                    printf("(%ux%ux%ux%u) = %u\n", gridDim.x, nearNeighborBlock, blockDim.x, blockDim.x, matSize);
                    assert(mid < matSize);
                }
                //
	    		// update weight with density of axon dendrites and preference over type
                //p *= daxn[ipre] * dd;// * preP_type[ip*networkSize + id];
				p *= daxn[ipre] * dd;
                for (Size iFeature = 0; iFeature < nFeature; iFeature++) {
                    //p *= pref[iFeature](fV[iFeature], feature[iFeature*networkSize + ipre], pF[iFeature*nType +ip]);
					p *= pref[iFeature](fV[iFeature], feature[iFeature*networkSize + ipre], pF[iFeature*nType +ip]);
                }
                assert(p >= 0);
				if (p > 0) {
                	sumType[ip]++;
                	sumP[ip] += p;
                	conMat[mid] = p;
				}
            }
            delayMat[mid] = distance; // record even if not connected, for LFP
        }
		__syncthreads();
    }
    // the remaining neighbors
    if (nb > 0) {
        for (Size in=nn; in<nNeighborBlock[blockIdx.x]; in++) {
            Size bid = neighborBlockId[maxNeighborBlock*blockId + in] * blockDim.x;
            x1[threadIdx.x] = pos[bid*2 + threadIdx.x];
            y1[threadIdx.x] = pos[bid*2 + blockDim.x + threadIdx.x];
            __syncthreads();
            #pragma unroll
            for (Size i=0; i<blockDim.x; i++) {
                // blockwise load from gmem the (potential) presynaptic neurons' properties
                Size ipre = bid + i; // pre-id in the network
                Float ra = raxn[ipre];
                double x = x1[i] - x0;
                double y = y1[i] - y0;
                Float distance = static_cast<Float>(square_root(x*x + y*y));
                Float p = connect(distance, ra, rd, gaussian_profile);
                Size tid = (nn-in)*blockDim.x + i; // only ofr tempNeighbor, which is local, no need to coalease memory
                tempNeighbor[tid] = 0;
                if (p > 0) {
                    Size ip = preType[ipre];
                    p *= daxn[ipre] * dd; //* preP_type[ip*networkSize+id];
                    for (Size iFeature = 0; iFeature < nFeature; iFeature ++) {
                        p *= pref[iFeature](fV[iFeature], feature[iFeature*networkSize + ipre], pF[iFeature*nType + ip]);
                    }
                    assert(p>=0);
					if (p > 0) {
                    	sumType[ip]++;
                    	sumP[ip] += p;
                    	tempNeighbor[tid] = p;
					}
                }
            }
            __syncthreads();
        }
    }
    __syncwarp();
    Size* pN = new Size[nType];
    #pragma unroll
    for (Size i=0; i<nType; i++) {
        pN[i] = preN_type[i*networkSize + id];
        preTypeAvail[i*networkSize + id] = sumType[i];
		if (sumType[i] < ceiling(pN[i]*tol)) {
			printf("neuron %u-%u dont have enough type %u neurons to connect to\n", blockIdx.x, threadIdx.x, i);
			assert(sumType[i] >= ceiling(pN[i]*tol));
		}
    }
	__syncthreads();
    delete []pF;
    delete []fV;
    //============= redistribute p of all ==========
    Float* sumStrType = new Float[nType];
    Float* sumType0 = new Float[nType];
    Float* pS = new Float[nType];
    Size* nid = new Size[nType];
	Size max_N = 0;
    #pragma unroll
    for (Size i=0; i<nType; i++) {
		if (pN[i] > max_N) max_N = pN[i];
        pS[i] = preS_type[i*networkSize + id];
    	sumType[i] = 0;
		nid[i] = 0;
    }
    PosInt* _vecID = new PosInt[nType*max_N];

    hiprandStateMRG32k3a localState = state[id];
	Size count = 0;
	Size connected = false;
	while (!connected) {
    	for (Size i=0; i<nType; i++) {
			sumType0[i] = sumType[i];
			if (sumType0[i] < ceiling(tol*pN[i])) {
    	    	sumType[i] = 0;
    	    	sumStrType[i] = 0;
			}
		}
    	for (Size in=0; in<nn; in++) {
    	    PosInt bid = neighborBlockId[maxNeighborBlock*blockId + in] * blockDim.x;
    	    #pragma unroll
    	    for (Size i=0; i<blockDim.x; i++) {
    	        PosIntL mid = (static_cast<PosIntL>(blockIdx.x*nearNeighborBlock + in)*blockDim.x + i)*blockDim.x + threadIdx.x;
    	        Size ipre = bid + i;
    	        Size ip = preType[ipre];
				if (sumType0[ip] < ceiling(tol*pN[ip])) {
    	        	Float p = abs(conMat[mid]);
    	        	if (p > 0) {
						if (count == 0) p *= pN[ip]/sumP[ip];
    	        	    Float xrand = uniform(&localState);
    	        	    if (xrand < p) {
    	        	        Float str = pS[ip];
    	        	        if (p > 1) {
    	        	            str = str*p;
    	        	        }
    	        	        sumType[ip] ++;
    	        	        sumStrType[ip] += str;
							conMat[mid] = p;
    	        	    } else {
							conMat[mid] = -p;
						}
    	        	} 
				}
    	    }
    	}
    	if (nb > 0) {
    	    for (Size in=nn; in<nNeighborBlock[blockIdx.x]; in++) {
    	        Size bid = neighborBlockId[maxNeighborBlock*blockId + in];
    	        #pragma unroll
    	        for (Size i=0; i<blockDim.x; i++) {
    	            Size tid = (nn-in)*blockDim.x + i;
    	            Size ipre = bid + i;
    	            Size ip = preType[ipre];
					if (sumType0[ip] < ceiling(tol*pN[ip])) {
    	            	Float p = tempNeighbor[tid];
    	            	if (p > 0) {
							if (count == 0) {
    	            	    	p *= pN[ip]/sumP[ip];
								tempNeighbor[tid] = p;
							}
    	            	    Float xrand = uniform(&localState);
    	            	    if (xrand < p) {
    	            	        Float str = pS[ip];
    	            	        if (p > 1) {
    	            	            str = str*p;
    	            	        }
    	            	        sumType[ip] ++;
    	            	        sumStrType[ip] += str;
    	            	        _vecID[ip*max_N + nid[ip]] = tid;
    	            	        nid[ip]++;
                				if (nid[ip] > max_N) {
                				    printf("set bigger max_N, currently %u\n", max_N);
                				    assert(nid[ip] <= max_N);
                				}
    	            	    }
    	            	}
					}
    	        }
				__syncthreads();
    	    }
    	}
		connected = true;
		for (PosInt i=0;i<nType;i++) {
			if (sumType[i] < ceiling(pN[i]*tol)) {
				connected = false;
			}
		}
		count++;
		if (count > 1 && !connected) {
			printf("neuron %u-%u need to make another round(%u) of connection, because of %u/%u, %u/%u\n", blockIdx.x, threadIdx.x, count, sumType[0],pN[0], sumType[1],pN[1]);
		}
		if (count >= 20) {
			printf("neuron %u-%u don't have one (or any) of the types of neurons to connect to\n", blockIdx.x, threadIdx.x);
			assert(count < 20);
			//connected = true;
		}
		if (count == 1) {
    		delete []sumP;
		}
	}
	delete []sumType0;
	Size total_nid = 0;
	for (PosInt i=0; i<nType; i++) {
		total_nid += nid[i];
	}
    nVec[id] = total_nid;
    
    Float *ratio = new Float[nType];
    if (strictStrength) {
        for (Size i=0; i<nType; i++) {
            if (sumStrType[i] > 0) {
                ratio[i] = pS[i]*pN[i]/sumStrType[i];
            } else {
                ratio[i] = 0;
            }
        }
    }
    #pragma unroll
    for (Size i=0; i<nType; i++) {
        preTypeConnected[i*networkSize + id] = sumType[i];
        sumStrType[i] = 0;
    }
    delete []sumType;

    // ======== strictly normalize the strengths ==========
    for (Size in=0; in<nn; in++) {
        Size bid = neighborBlockId[maxNeighborBlock*blockId + in] * blockDim.x;
        #pragma unroll
        for (Size i=0; i<blockDim.x; i++) {
            PosIntL mid = (static_cast<PosIntL>(blockIdx.x*nearNeighborBlock + in)*blockDim.x + i)*blockDim.x + threadIdx.x;
            Size ip = preType[bid + i];
			Float str = conMat[mid];
			if (str <= 0) str = 0;
			else {
				if (str > 1) str *= pS[ip];
				else str = pS[ip];
			}
    		if (strictStrength) {
            	 str *= ratio[ip];
			}
			conMat[mid] = str;
			if (str > 0) {
            	sumStrType[ip] += str;
			}
        }
    }
    if (total_nid > 0) {
		PosInt* qid = new PosInt[nType];
		for (PosInt i=0; i<nType; i++) {
			qid[i] = 0;
		}
    	Size iid = 0;
        for (Size in=nn; in<nNeighborBlock[blockIdx.x]; in++) {
            Size bid = neighborBlockId[maxNeighborBlock*blockId + in];
            x1[threadIdx.x] = pos[bid*2 + threadIdx.x];
            y1[threadIdx.x] = pos[bid*2 + blockDim.x + threadIdx.x];
            __syncthreads();
			if (iid >= total_nid) {
				break;
			}
            #pragma unroll
            for (Size i=0; i<blockDim.x; i++) {
                Size tid = (nn-in)*blockDim.x + i;
				PosInt ipre = bid + i;
                Size ip = preType[ipre];
				if (qid[ip] >= nid[ip]) {
					i = typeAcc0[ip+1]-1;
					continue;
				}
				if (_vecID[ip*max_N + qid[ip]] == tid) {
					Float p = tempNeighbor[tid];
					Float str = pS[ip];
					if (p > 1) str *= p;
    				if (strictStrength) {
						str *= ratio[ip];
					}
					vecID[maxDistantNeighbor*id + iid] = ipre;
					conVec[maxDistantNeighbor*id + iid] = str;
                	Float x = static_cast<Float>(x1[i] - x0);
                	Float y = static_cast<Float>(y1[i] - y0);
	    			Float distance = square_root(x*x + y*y);
                	delayVec[maxDistantNeighbor*id + iid] = distance;
                	sumStrType[ip] += str;
                	iid ++;
					qid[ip]++;
                	if (iid > maxDistantNeighbor) {
                	    printf("set bigger maxDistantNeighbor, currently %u\n", maxDistantNeighbor);
                	    assert(iid <= maxDistantNeighbor);
                	}
				}
				if (iid >= total_nid) {
					break;
				}
            }
            __syncthreads();
        }
    }
    if (nb > 0) {
        delete []tempNeighbor;
    }

    delete []ratio;
    #pragma unroll
    for (Size i=0; i<nType; i++) {
        preTypeStrSum[i*networkSize + id] = sumStrType[i];
		if (strictStrength) {
			if (abs(sumStrType[i] - pN[i]*pS[i])/(pN[i]*pS[i]) > 1e-3) {
				printf("%u-%u-%u: sumStrType[i] = %f,  pN[i]*pS[i] = %f, count = %u, nid = %u\n", blockIdx.x, threadIdx.x, i, sumStrType[i], pN[i]*pS[i], count, nVec[id]);
				//assert(abs(sumStrType[i] - pN[i]*pS[i])/(pN[i]*pS[i]) <= 1e-3);
			}
		}
    }
    delete []sumStrType;
    delete []pN;
    delete []pS;
}
